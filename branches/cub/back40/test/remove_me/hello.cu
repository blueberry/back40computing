#include "hip/hip_runtime.h"


#include <stdio.h>

#include <hipcub/hipcub.hpp>


using namespace hipcub;

template <
	LoadModifier LOAD_MODIFIER,
	StoreModifier STORE_MODIFIER,
	typename T>
__global__ void Kernel(T *d_in, T *d_out)
{
	T datum = Load<LOAD_MODIFIER>(d_in + threadIdx.x);
	Store<STORE_MODIFIER>(d_out + threadIdx.x, datum);

	datum = Load<LOAD_MODIFIER>(d_in + threadIdx.x + 1);
	Store<STORE_MODIFIER>(d_out + threadIdx.x + 1, datum);

}

struct Foo { int a; double b; } *d_struct = NULL;

__global__ void Kernel2(Foo *d_in, Foo *d_out)
{
	Foo datum = *(d_in + threadIdx.x);
	*(d_out + threadIdx.x) = datum;

	datum = *(d_in + threadIdx.x + 1);
	*(d_out + threadIdx.x + 1) = datum;
}


/**
 * Main
 */
int main(int argc, const int**argv)
{
/*
	double* d_double = NULL;
	Kernel<LOAD_CS, STORE_WB><<<1,1>>>(d_double, d_double);

	double1* d_double1 = NULL;
	Kernel<LOAD_CS, STORE_WB><<<1,1>>>(d_double1, d_double1);

	double2* d_double2 = NULL;
	Kernel<LOAD_CS, STORE_WB><<<1,1>>>(d_double2, d_double2);

	double4* d_double4 = NULL;
	Kernel<LOAD_CG, STORE_CG><<<1,1>>>(d_double4, d_double4);

	Kernel2<<<1,1>>>(d_double4, d_double4);
*/

	Kernel<LOAD_CG, STORE_CG><<<1,1>>>(d_struct, d_struct);

	return 0;

}
