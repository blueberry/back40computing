#include "hip/hip_runtime.h"
/******************************************************************************
 *
 * Copyright (c) 2010-2012, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2012, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 ******************************************************************************/

/******************************************************************************
 * Experimental reduce-value-by-row COO implementation of SPMV
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <iterator>
#include <vector>
#include <algorithm>
#include <stdio.h>
#include <test_util.h>

#include "../cub.cuh"

using namespace hipcub;
using namespace std;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool    g_verbose       = false;
int     g_iterations    = 1;


//---------------------------------------------------------------------
// Graph building types and utilities
//---------------------------------------------------------------------

/**
 * COO graph type.  A COO graph is just a vector of edge tuples.
 */
template<typename VertexId, typename Value>
struct CooGraph
{
    /**
     * COO edge tuple.  (A COO graph is just a vector of these.)
     */
    struct CooTuple
    {
        VertexId            row;
        VertexId            col;
        Value               val;

        CooTuple() {}
        CooTuple(VertexId row, VertexId col) : row(row), col(col) {}
        CooTuple(VertexId row, VertexId col, Value val) : row(row), col(col), val(val) {}
    };

    /**
     * Comparator for sorting COO sparse format edges
     */
    static bool CooTupleCompare (const CooTuple &elem1, const CooTuple &elem2)
    {
        if (elem1.row < elem2.row)
        {
            return true;
        }
        else if ((elem1.row == elem2.row) && (elem1.col < elem2.col))
        {
            return true;
        }

        return false;
    }

    int                 row_dim;        // Num rows
    int                 col_dim;        // Num cols
    vector<CooTuple>    coo_tuples;     // Non-zero entries


    /**
     * CooGraph ostream operator
     */
    friend std::ostream& operator<<(std::ostream& os, const CooGraph& coo_graph)
    {
        os << "Sparse COO (" << coo_graph.row_dim << " rows, " << coo_graph.col_dim << " cols, " << coo_graph.coo_tuples.size() << " nonzeros):\n";
        os << "Ordinal, Row, Col, Val\n";
        for (int i = 0; i < coo_graph.coo_tuples.size(); i++)
        {
            os << i << ',' << coo_graph.coo_tuples[i].row << ',' << coo_graph.coo_tuples[i].col << ',' << coo_graph.coo_tuples[i].val << "\n";
        }
        return os;
    }

    /**
     * Update graph dims based upon COO tuples
     */
    void UpdateDims()
    {
        row_dim = -1;
        col_dim = -1;

        for (int i = 0; i < coo_tuples.size(); i++)
        {
            row_dim = CUB_MAX(row_dim, coo_tuples[i].row);
            col_dim = CUB_MAX(col_dim, coo_tuples[i].col);
        }

        row_dim++;
        col_dim++;
    }


    /**
     * Builds a wheel COO sparse graph having spokes spokes.
     */
    void InitWheel(VertexId spokes)
    {
        VertexId edges  = spokes + (spokes - 1);

        coo_tuples.clear();
        coo_tuples.reserve(edges);

        // Add spoke edges
        for (VertexId i = 0; i < spokes; i++)
        {
            coo_tuples.push_back(CooTuple(0, i + 1));
        }

        // Add rim
        for (VertexId i = 0; i < spokes; i++)
        {
            VertexId dest = (i + 1) % spokes;
            coo_tuples.push_back(CooTuple(i + 1, dest + 1));
        }

        // Sort by rows, then columns, update dims
        std::stable_sort(coo_tuples.begin(), coo_tuples.end(), CooTupleCompare);
        UpdateDims();
    }


    /**
     * Builds a square 3D grid COO sparse graph.  Interior nodes have degree 7 (including
     * a self-loop).  Values are unintialized, coo_tuples are sorted.
     */
    void InitGrid3d(VertexId width)
    {
        VertexId interior_nodes        = (width - 2) * (width - 2) * (width - 2);
        VertexId face_nodes            = (width - 2) * (width - 2) * 6;
        VertexId edge_nodes            = (width - 2) * 12;
        VertexId corner_nodes          = 8;
        VertexId nodes                 = width * width * width;
        VertexId edges                 = (interior_nodes * 6) + (face_nodes * 5) + (edge_nodes * 4) + (corner_nodes * 3) + nodes;

        coo_tuples.clear();
        coo_tuples.reserve(edges);

        for (VertexId i = 0; i < width; i++) {
            for (VertexId j = 0; j < width; j++) {
                for (VertexId k = 0; k < width; k++) {

                    VertexId me = (i * width * width) + (j * width) + k;

                    VertexId neighbor = (i * width * width) + (j * width) + (k - 1);
                    if (k - 1 >= 0) {
                        coo_tuples.push_back(CooTuple(me, neighbor));
                    }

                    neighbor = (i * width * width) + (j * width) + (k + 1);
                    if (k + 1 < width) {
                        coo_tuples.push_back(CooTuple(me, neighbor));
                    }

                    neighbor = (i * width * width) + ((j - 1) * width) + k;
                    if (j - 1 >= 0) {
                        coo_tuples.push_back(CooTuple(me, neighbor));
                    }

                    neighbor = (i * width * width) + ((j + 1) * width) + k;
                    if (j + 1 < width) {
                        coo_tuples.push_back(CooTuple(me, neighbor));
                    }

                    neighbor = ((i - 1) * width * width) + (j * width) + k;
                    if (i - 1 >= 0) {
                        coo_tuples.push_back(CooTuple(me, neighbor));
                    }

                    neighbor = ((i + 1) * width * width) + (j * width) + k;
                    if (i + 1 < width) {
                        coo_tuples.push_back(CooTuple(me, neighbor));
                    }

                    neighbor = me;
                    coo_tuples.push_back(CooTuple(me, neighbor));
                }
            }
        }

        // Sort by rows, then columns, update dims
        std::stable_sort(coo_tuples.begin(), coo_tuples.end(), CooTupleCompare);
        UpdateDims();

    }
};




//---------------------------------------------------------------------
// GPU types and device functions
//---------------------------------------------------------------------


/// Pairing of dot product partial sums and corresponding row-id
template <typename VertexId, typename Value>
struct PartialSum
{
    Value       partial;        /// PartialSum sum
    VertexId    row;            /// Row-id

    /// Tags indicating this structure provides overloaded ThreadLoad and ThreadStore operations
    typedef void ThreadLoadTag;
    typedef void ThreadStoreTag;

    /// ThreadLoad (simply defer to loading individual items)
    template <PtxLoadModifier MODIFIER>
    __device__ __forceinline__ void ThreadLoad(PartialSum *ptr)
    {
        partial = hipcub::ThreadLoad<MODIFIER>(&(ptr->partial));
        row = hipcub::ThreadLoad<MODIFIER>(&(ptr->row));
    }

     /// ThreadStore (simply defer to storing individual items)
    template <PtxStoreModifier MODIFIER>
    __device__ __forceinline__ void ThreadStore(PartialSum *ptr) const
    {
        // Always write partial first
        hipcub::ThreadStore<MODIFIER>(&(ptr->partial), partial);
        hipcub::ThreadStore<MODIFIER>(&(ptr->row), row);
    }

};


/// Scan progress
template <typename VertexId, typename Value>
struct ScanProgress
{
    /// The offset of the first waiting CTA
    int active_offset;

    /// The inter-CTA aggregate so far
    PartialSum<VertexId, Value> aggregate;
};


/// Templated Texture reference type for multiplicand vector
template <typename Value>
struct TexVector
{
    // Texture reference type
    typedef texture<Value, hipTextureType1D, hipReadModeElementType> TexRef;

    static TexRef ref;

    /**
     * Bind textures
     */
    static void BindTexture(void *d_in, int elements)
    {
        hipChannelFormatDesc tex_desc = hipCreateChannelDesc<Value>();
        if (d_in)
        {
            size_t offset;
            size_t bytes = sizeof(Value) * elements;
            CubDebugExit(hipBindTexture(&offset, ref, d_in, tex_desc, bytes));
        }
    }

    /**
     * Unbind textures
     */
    static void UnbindTexture()
    {
        CubDebugExit(hipUnbindTexture(ref));
    }
};

// Texture reference definitions
template <typename Value>
typename TexVector<Value>::TexRef TexVector<Value>::ref = 0;



/**
 * CTA abstraction for processing sparse SPMV tiles
 */
template <
    int             CTA_THREADS,
    int             ITEMS_PER_THREAD,
    typename        VertexId,
    typename        Value>
struct SpmvCta
{
    //---------------------------------------------------------------------
    // Types and constants
    //---------------------------------------------------------------------

    // Constants
    enum
    {
        TILE_ITEMS = CTA_THREADS * ITEMS_PER_THREAD,
    };

    // Head flag type
    typedef int                                                     HeadFlag;

    // Dot product partial sum type
    typedef PartialSum<VertexId, Value>                             PartialSum;

    // Parameterized CUB types for use in the current problem context
    typedef CtaScan<PartialSum, CTA_THREADS>                        CtaScan;
    typedef CtaExchange<PartialSum, CTA_THREADS, ITEMS_PER_THREAD>  CtaExchange;
    typedef CtaDiscontinuity<HeadFlag, CTA_THREADS>                 CtaDiscontinuity;

    // Shared memory type for this CTA
    struct SmemStorage
    {
        union
        {
            typename CtaScan::SmemStorage           scan;               // Smem needed for reduce-value-by-row scan
            typename CtaExchange::SmemStorage       exchange;           // Smem needed for striped->blocked transpose
            typename CtaDiscontinuity::SmemStorage  discontinuity;      // Smem needed for head-flagging
        };

        PartialSum prev_aggregate;
        PartialSum aggregate;
    };


    /// Reduce-by-row scan operator
    struct ScanOp
    {
        __device__ __forceinline__ PartialSum operator()(
            const PartialSum &first,
            const PartialSum &second)
        {
            PartialSum retval;

            retval.partial = (second.row != first.row) ?
                    second.partial :
                    first.partial + second.partial;

            retval.row = second.row;
            return retval;
        }
    };

    // Callback functor for waiting on the previous CTA to compute its partial sum (the prefix for this CTA)
    struct CtaPrefixOp
    {
        ScanProgress<VertexId, Value>       *d_scan_progress;
        int                                 cta_offset;
        PartialSum                          identity;
        PartialSum                          prev_aggregate;
        PartialSum                          aggregate;
        ScanOp                              scan_op;

        /// Constructor
        __device__ __forceinline__ CtaPrefixOp(
            ScanProgress<VertexId, Value>   *d_scan_progress,
            int                             cta_offset,
            PartialSum                      identity,
            ScanOp                          scan_op) :
                cta_offset(cta_offset),
                d_scan_progress(d_scan_progress),
                identity(identity),
                scan_op(scan_op)
        {}


        /**
         * CTA-wide prefix callback functor called by thread-0 in CtaScan::ExclusiveScan().
         * Returns the CTA-wide prefix to apply to all scan inputs.
         */
        __device__ __forceinline__ PartialSum operator()(
            const PartialSum &local_aggregate)              ///< The aggregate sum of the local prefix sum inputs
        {
            // Get aggregate from prior CTA
            if (cta_offset == 0)
            {
                // The first tile has no prior aggregate: use identity
                prev_aggregate = identity;
            }
            else
            {
                // Keep loading prior CTA's aggregate until valid
                while (ThreadLoad<PTX_LOAD_CG>(&d_scan_progress->active_offset) != cta_offset)
                {
                    __threadfence_block();
                }

                // It's our turn: load the inter-CTA aggregate up to this point
                prev_aggregate = ThreadLoad<PTX_LOAD_CG>(&d_scan_progress->aggregate);
            }

            // Write updated CTA-wide aggregate and signal to subsequent CTA that value is ready
            aggregate = scan_op(prev_aggregate, local_aggregate);

            ThreadStore<PTX_STORE_CG>(&d_scan_progress->aggregate, aggregate);
            __threadfence_block();
            ThreadStore<PTX_STORE_CG>(&d_scan_progress->active_offset, cta_offset + TILE_ITEMS);

            // Return CTA's prefix
            return prev_aggregate;
        }
    };


    /// Functor for detecting row discontinuities.
    struct NewRowOp
    {
        /// Returns true if row_b is the start of a new row
        template <typename VertexId>
        __device__ __forceinline__ bool operator()(
            const VertexId& row_a,
            const VertexId& row_b)
        {
            return (row_a != row_b);
        }
    };

    //---------------------------------------------------------------------
    // Operations
    //---------------------------------------------------------------------

    /**
     * Processes a COO input tile of edges, outputting dot products for each row
     */
    __device__ __forceinline__
    static void ProcessTile(
        SmemStorage                     &s_storage,
        ScanProgress<VertexId, Value>   *d_scan_progress,
        VertexId*                       d_rows,
        VertexId*                       d_columns,
        Value*                          d_values,
        Value*                          d_vector,
        Value*                          d_result,
        int                             num_edges,
        int                             cta_offset,
        int                             guarded_items = 0)
    {
        VertexId    columns[ITEMS_PER_THREAD];
        VertexId    rows[ITEMS_PER_THREAD];
        Value       values[ITEMS_PER_THREAD];
        PartialSum  partial_sums[ITEMS_PER_THREAD];
        HeadFlag    head_flags[ITEMS_PER_THREAD];

        // Load a CTA-striped tile of A (sparse row-ids, column-ids, and values)
        if (guarded_items)
        {
            // This is a partial-tile (e.g., the last tile of input).  Extend the coordinates of the last
            // vertex for out-of-bound items, but zero-valued
            VertexId last_row = d_rows[num_edges - 1];
            VertexId last_column = d_columns[num_edges - 1];

            CtaLoadDirectStriped(rows, d_rows, cta_offset, guarded_items, last_row);
            CtaLoadDirectStriped(columns, d_columns, cta_offset, guarded_items, last_column);
            CtaLoadDirectStriped(values, d_values, cta_offset, guarded_items, Value(0.0));
        }
        else
        {
            // Unguarded loads
            CtaLoadDirectStriped(rows, d_rows, cta_offset);
            CtaLoadDirectStriped(columns, d_columns, cta_offset);
            CtaLoadDirectStriped(values, d_values, cta_offset);
        }

        // Fence to prevent hoisting any dependent code below into the loads above
        // Mooch
        __threadfence_block();

        // Load the referenced values from x and compute the dot product partials sums
        #pragma unroll
        for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ITEM++)
        {
            Value vec_item = tex1Dfetch(TexVector<Value>::ref, columns[ITEM]);
            partial_sums[ITEM].partial = values[ITEM] * vec_item;
            partial_sums[ITEM].row = rows[ITEM];
        }

        // Transpose from CTA-striped to CTA-blocked arrangement
        CtaExchange::StripedToBlocked(s_storage.exchange, partial_sums);

        // Barrier for smem reuse
        __syncthreads();

        // Save a copy of the original row IDs.  We will use them to compute the row head flags
        // later.  (After the scan, the row fields in partial_sums will
        // be shifted by one element because the scan's exclusive nature.)
        #pragma unroll
        for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ITEM++)
        {
            rows[ITEM] = partial_sums[ITEM].row;
        }

        // Compute exclusive scan of partial_sums
        VertexId        first_row = d_rows[cta_offset];                             // The vertex id of the first row encountered by this CTA
        ScanOp          scan_op;                                                    // Reduce-by-row scan operator
        PartialSum      local_aggregate;                                            // CTA-wide aggregate in thread0 (unused)
        PartialSum      identity = {0.0, first_row};                                // Zero-valued identity (with row-id of first item)
        CtaPrefixOp     prefix_op(d_scan_progress, cta_offset, identity, scan_op);  // Callback functor for waiting on the previous CTA to compute its partial sum

/*

        CtaScan::ExclusiveScan(
            s_storage.scan,
            partial_sums,
            partial_sums,           // (Out)
            identity,
            scan_op,
            local_aggregate,        // (Out)
            prefix_op);             // (In-out)
*/

        CtaScan::ExclusiveScan(
            s_storage.scan,
            partial_sums,
            partial_sums,           // (Out)
            identity,
            scan_op,
            local_aggregate);       // (Out)

        // Get aggregate from prior CTA
        if (threadIdx.x == 0)
        {
            if (cta_offset == 0)
            {
                // The first tile has no prior aggregate: use identity
                s_storage.prev_aggregate = identity;
            }
            else
            {
                // Keep loading prior CTA's aggregate until valid
                while (ThreadLoad<PTX_LOAD_CG>(&d_scan_progress->active_offset) != cta_offset)
                {
                    __threadfence_block();
                }

                // It's our turn: load the inter-CTA aggregate up to this point
                s_storage.prev_aggregate = ThreadLoad<PTX_LOAD_CG>(&d_scan_progress->aggregate);
            }

            // Write updated CTA-wide aggregate and signal to subsequent CTA that value is ready
            s_storage.aggregate = scan_op(s_storage.prev_aggregate, local_aggregate);

            ThreadStore<PTX_STORE_CG>(&d_scan_progress->aggregate, s_storage.aggregate);
            __threadfence_block();
            ThreadStore<PTX_STORE_CG>(&d_scan_progress->active_offset, cta_offset + TILE_ITEMS);
        }

        // Barrier for smem reuse and coherence
        __syncthreads();

        #pragma unroll
        for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ITEM++)
        {
            partial_sums[ITEM] = scan_op(s_storage.prev_aggregate, partial_sums[ITEM]);
        }

        if (threadIdx.x == 0)
        {
            partial_sums[0] = s_storage.prev_aggregate;
        }


        // Flag row heads using saved row ids
        CtaDiscontinuity::Flag(
            s_storage.discontinuity,
            rows,                           // Original row ids
            prefix_op.prev_aggregate.row,   // Last row id from previous CTA
            NewRowOp(),                     // Functor for detecting start of new rows
            head_flags);                    // (Out) Head flags

        // Scatter the dot products if they are the head of a valid row
        #pragma unroll
        for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ITEM++)
        {
            if (head_flags[ITEM])
            {
                d_result[partial_sums[ITEM].row] = partial_sums[ITEM].partial;
            }
        }
/*
        // The last tile scatters its aggregate value as the last output
        if (
            (cta_offset + TILE_ITEMS >= num_edges) &&           // Last tile
            (threadIdx.x == 0) &&                               // First thread
            (prefix_op.aggregate.row >= 0))                     // Valid row ID
        {
            d_result[prefix_op.aggregate.row] = prefix_op.aggregate.partial;
        }
*/

        // The last tile scatters its aggregate value as the last output
        if (
            (cta_offset + TILE_ITEMS >= num_edges) &&           // Last tile
            (threadIdx.x == 0) &&                               // First thread
            (s_storage.aggregate.row >= 0))                     // Valid row ID
        {
            d_result[s_storage.aggregate.row] = s_storage.aggregate.partial;
        }

    }

};


/**
 * COO SpMV kernel
 */
template <
    int             CTA_THREADS,
    int             ITEMS_PER_THREAD,
    typename        VertexId,
    typename        Value>
__global__ void CooKernel(
    GridQueue<int>                  scan_queue,
    ScanProgress<VertexId, Value>*  d_scan_progress,
    VertexId*                       d_rows,
    VertexId*                       d_columns,
    Value*                          d_values,
    Value*                          d_vector,
    Value*                          d_result,
    int                             num_edges)
{
    const int TILE_SIZE = CTA_THREADS * ITEMS_PER_THREAD;

    // CTA type
    typedef SpmvCta<CTA_THREADS, ITEMS_PER_THREAD, VertexId, Value> SpmvCta;

    // Shared memory
    __shared__ typename SpmvCta::SmemStorage s_storage;
    __shared__ int cta_offset;

    // Process tiles of sparse matrix
    while (true)
    {
        // Thread0 steals a tile of work
        if (threadIdx.x == 0)
        {
            cta_offset = scan_queue.Drain(TILE_SIZE);
        }

        __syncthreads();

        if (cta_offset >= num_edges)
        {
            // Done
            break;
        }
        else if (cta_offset + TILE_SIZE < num_edges)
        {
            // Full tile
            SpmvCta::ProcessTile(
                s_storage,
                d_scan_progress,
                d_rows,
                d_columns,
                d_values,
                d_vector,
                d_result,
                num_edges,
                cta_offset);
        }
        else if (cta_offset < num_edges)
        {
            // Partial tile
            int guarded_items = num_edges - cta_offset;
            SpmvCta::ProcessTile(
                s_storage,
                d_scan_progress,
                d_rows,
                d_columns,
                d_values,
                d_vector,
                d_result,
                num_edges,
                cta_offset,
                guarded_items);
        }

        __syncthreads();
    }
}



/**
 * COO Initialization kernel.  Initializes queue counters and output vector to zero.
 */
template <typename VertexId, typename Value>
__global__ void InitKernel(
    GridQueue<int>                  scan_queue,         ///< Queue counters
    ScanProgress<VertexId, Value>*  d_scan_progress,    ///< Scan progress state
    Value*                          d_result,           ///< Output vector
    int                             vector_length)      ///< Output vector length
{
    // Reset queue counters to known state
    if ((blockIdx.x == 0) && (threadIdx.x == 0))
    {
        scan_queue.PrepareDrain();
        d_scan_progress->active_offset = 0;
    }

    // Initialize output vector elements to 0.0
    const int STRIDE = gridDim.x * blockDim.x;
    for (int idx = (blockIdx.x * blockDim.x) + threadIdx.x; idx < vector_length; idx += STRIDE)
    {
        d_result[idx] = 0;
    }
}




//---------------------------------------------------------------------
// Host subroutines
//---------------------------------------------------------------------


/**
 * Simple test of device
 */
template <
    int                         CTA_THREADS,
    int                         ITEMS_PER_THREAD,
    typename                    VertexId,
    typename                    Value>
void TestDevice(
    CooGraph<VertexId, Value>&  coo_graph,
    Value*                      h_vector,
    Value*                      h_reference)
{
    if (g_iterations <= 0) return;

    // SOA device storage
    VertexId*                       d_rows;             // SOA graph row coordinates
    VertexId*                       d_columns;          // SOA graph col coordinates
    Value*                          d_values;           // SOA graph values
    Value*                          d_vector;           // Vector multiplicand
    Value*                          d_result;           // Output row
    ScanProgress<VertexId, Value>*  d_scan_progress;    // Temporary storage for communicating dot product partials between CTAs

    // Create SOA version of coo_graph on host
    int                             num_edges       = coo_graph.coo_tuples.size();
    VertexId*                       h_rows          = new VertexId[num_edges];
    VertexId*                       h_columns       = new VertexId[num_edges];
    Value*                          h_values        = new Value[num_edges];
    for (int i = 0; i < num_edges; i++)
    {
        h_rows[i]       = coo_graph.coo_tuples[i].row;
        h_columns[i]    = coo_graph.coo_tuples[i].col;
        h_values[i]     = coo_graph.coo_tuples[i].val;
    }

    // Allocate COO device arrays
    CubDebugExit(DeviceAllocate((void**)&d_rows,           sizeof(VertexId) * num_edges));
    CubDebugExit(DeviceAllocate((void**)&d_columns,        sizeof(VertexId) * num_edges));
    CubDebugExit(DeviceAllocate((void**)&d_values,         sizeof(Value) * num_edges));
    CubDebugExit(DeviceAllocate((void**)&d_vector,         sizeof(Value) * coo_graph.col_dim));
    CubDebugExit(DeviceAllocate((void**)&d_result,         sizeof(Value) * coo_graph.row_dim));
    CubDebugExit(DeviceAllocate((void**)&d_scan_progress,  sizeof(ScanProgress<VertexId, Value>)));

    // Copy host arrays to device
    CubDebugExit(hipMemcpy(d_rows,     h_rows,     sizeof(VertexId) * num_edges, hipMemcpyHostToDevice));
    CubDebugExit(hipMemcpy(d_columns,  h_columns,  sizeof(VertexId) * num_edges, hipMemcpyHostToDevice));
    CubDebugExit(hipMemcpy(d_values,   h_values,   sizeof(Value) * num_edges, hipMemcpyHostToDevice));
    CubDebugExit(hipMemcpy(d_vector,   h_vector,   sizeof(Value) * coo_graph.col_dim, hipMemcpyHostToDevice));

    // Bind textures
    TexVector<Value>::BindTexture(d_vector, coo_graph.col_dim);

    // Queue for managing CTA work assignment
    GridQueue<int> scan_queue;
    scan_queue.Allocate();

    // Get kernel properties
    CudaProps cuda_props;
    KernelProps init_kernel_props;
    KernelProps coo_kernel_props;
    CubDebugExit(cuda_props.Init());
    CubDebugExit(init_kernel_props.Init(InitKernel<VertexId, Value>, CTA_THREADS, cuda_props));
    CubDebugExit(coo_kernel_props.Init(CooKernel<CTA_THREADS, ITEMS_PER_THREAD, VertexId, Value>, CTA_THREADS, cuda_props));

    // Determine launch configuration from kernel properties
    int init_grid_size = init_kernel_props.OversubscribedGridSize(CTA_THREADS, coo_graph.row_dim);
    int coo_grid_size = coo_kernel_props.ResidentGridSize();

    // Print debug info
    printf("InitKernel<<<%d, %d>>>(...), Max SM occupancy: %d\n",
        init_grid_size, CTA_THREADS, init_kernel_props.max_cta_occupancy);
    printf("CooKernel<%d, %d><<<%d, %d>>>(...), Max SM occupancy: %d\n",
        CTA_THREADS, ITEMS_PER_THREAD, coo_grid_size, CTA_THREADS, coo_kernel_props.max_cta_occupancy);
    fflush(stdout);

    // Run kernel
    GpuTimer gpu_timer;
    float elapsed_millis = 0.0;
    for (int i = 0; i < g_iterations; i++)
    {
        gpu_timer.Start();

        // Initialize output and temporaries
        InitKernel<<<init_grid_size, CTA_THREADS>>>(
            scan_queue,
            d_scan_progress,
            d_result,
            coo_graph.row_dim);

        // Run the COO kernel
        CooKernel<CTA_THREADS, ITEMS_PER_THREAD><<<coo_grid_size, CTA_THREADS>>>(
            scan_queue,
            d_scan_progress,
            d_rows,
            d_columns,
            d_values,
            d_vector,
            d_result,
            num_edges);

        gpu_timer.Stop();
        elapsed_millis += gpu_timer.ElapsedMillis();

        // Force any kernel stdio to screen
        CubDebugExit(hipDeviceSynchronize());
    }

    // Display timing
    float avg_elapsed = elapsed_millis / g_iterations;
    int total_bytes = ((sizeof(VertexId) + sizeof(VertexId) + sizeof(Value)) * num_edges) + (sizeof(Value) * 2 * coo_graph.row_dim);
    printf("%d iterations, average elapsed (%.3f ms), utilized bandwidth (%.3f GB/s), GFLOPS(%.3f)\n",
        g_iterations,
        avg_elapsed,
        total_bytes / avg_elapsed / 1000.0 / 1000.0,
        num_edges * 2 / avg_elapsed / 1000.0 / 1000.0);

    // Check results
    AssertEquals(0, CompareDeviceResults(h_reference, d_result, coo_graph.row_dim, g_verbose, g_verbose));

    // Cleanup
    TexVector<Value>::UnbindTexture();
    CubDebugExit(DeviceFree(d_scan_progress));
    CubDebugExit(DeviceFree(d_rows));
    CubDebugExit(DeviceFree(d_columns));
    CubDebugExit(DeviceFree(d_values));
    CubDebugExit(DeviceFree(d_vector));
    CubDebugExit(DeviceFree(d_result));
    delete[] h_rows;
    delete[] h_columns;
    delete[] h_values;
}


/**
 * Compute reference answer on CPU
 */
template <typename VertexId, typename Value>
void ComputeReference(
    CooGraph<VertexId, Value>&  coo_graph,
    Value*                      h_vector,
    Value*                      h_reference)
{
    for (VertexId i = 0; i < coo_graph.row_dim; i++)
    {
        h_reference[i] = 0.0;
    }

    for (VertexId i = 0; i < coo_graph.coo_tuples.size(); i++)
    {
        h_reference[coo_graph.coo_tuples[i].row] +=
            coo_graph.coo_tuples[i].val *
            h_vector[coo_graph.coo_tuples[i].col];
    }
}


/**
 * Assign arbitrary values to graph vertices
 */
template <typename CooGraph>
void AssignGraphValues(CooGraph &coo_graph)
{
    for (int i = 0; i < coo_graph.coo_tuples.size(); i++)
    {
        coo_graph.coo_tuples[i].val = i % 21;
    }
}


/**
 * Assign arbitrary values to vector items
 */
template <typename Value>
void AssignVectorValues(Value *vector, int col_dim)
{
    for (int i = 0; i < col_dim; i++)
    {
        vector[i] = 1.0;
    }
}


/**
 * Main
 */
int main(int argc, char** argv)
{
    // Graph of int32s as vertex ids, floats as values
    typedef int     VertexId;
    typedef float   Value;

    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("i", g_iterations);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s\n [--device=<device-id>] [--v] [--iterations=<test iterations>] [--grid-size=<grid-size>]\n"
            "\t--type=wheel --spokes=<spokes>\n"
            "\t--type=grid2d --width=<width>\n"
            "\t--type=grid3d --width=<width>\n"
            "\t--type=metis --file=<file>\n"
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    // Get graph type
    string type;
    args.GetCmdLineArgument("type", type);

    // Generate graph structure
    CooGraph<VertexId, Value> coo_graph;
    if (type == string("grid3d"))
    {
        VertexId width;
        args.GetCmdLineArgument("width", width);
        printf("Generating grid3d width(%d)... ", width); fflush(stdout);
        coo_graph.InitGrid3d(width);
        printf("Done.  %d non-zeros, %d rows, %d columns\n",
            coo_graph.coo_tuples.size(), coo_graph.row_dim, coo_graph.col_dim); fflush(stdout);
    }
    else if (type == string("wheel"))
    {
        VertexId spokes;
        args.GetCmdLineArgument("spokes", spokes);
        printf("Generating wheel spokes(%d)... ", spokes); fflush(stdout);
        coo_graph.InitWheel(spokes);
        printf("Done.  %d non-zeros, %d rows, %d columns\n",
            coo_graph.coo_tuples.size(), coo_graph.row_dim, coo_graph.col_dim); fflush(stdout);
    }
    else
    {
        printf("Unsupported graph type\n");
        exit(1);
    }
    AssignGraphValues(coo_graph);

    if (g_verbose)
    {
        cout << coo_graph << "\n";
    }

    // Create vector
    Value *h_vector = new Value[coo_graph.col_dim];
    AssignVectorValues(h_vector, coo_graph.col_dim);
    if (g_verbose)
    {
        printf("Vector[%d]: ", coo_graph.col_dim);
        DisplayResults(h_vector, coo_graph.col_dim);
        printf("\n\n");
    }

    // Compute reference answer
    Value *h_reference = new Value[coo_graph.row_dim];
    ComputeReference(coo_graph, h_vector, h_reference);
    if (g_verbose)
    {
        printf("Results[%d]: ", coo_graph.row_dim);
        DisplayResults(h_reference, coo_graph.row_dim);
        printf("\n\n");
    }

    // Run GPU version
    TestDevice<128, 5>(coo_graph, h_vector, h_reference);

    // Cleanup
    delete[] h_vector;
    delete[] h_reference;

    return 0;
}



