#include "hip/hip_runtime.h"
/******************************************************************************
 *
 * Copyright (c) 2010-2012, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2012, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of WarpScan utilities
 ******************************************************************************/

#define HIPCUB_STDERR

#include <stdio.h>
#include <test_util.h>
#include "../cub.cuh"

using namespace hipcub;

//---------------------------------------------------------------------
// Globals , constants and typedefs
//---------------------------------------------------------------------

/**
 * Verbose output
 */
bool g_verbose = false;


/**
 * Type of primitive to test
 */
enum TestMode
{
	BASIC,
	AGGREGATE,
	PREFIX_AGGREGATE,
};


/**
 * Uint2 reduction operator
 */
struct Uint2Sum
{
	// Scan op
	__host__ __device__ __forceinline__ uint2 operator()(uint2 a, uint2 b)
	{
		a.x += b.x;
		a.y += b.y;
		return a;
	}

	// Identity
	__host__ __device__ __forceinline__ uint2 operator()()
	{
		uint2 retval;
		retval.x = retval.y = 0;
		return retval;
	}
};


//---------------------------------------------------------------------
// Test kernels
//---------------------------------------------------------------------

/**
 * Exclusive WarpScan test kernel.
 */
template <
	TestMode	TEST_MODE,
	typename 	T,
	typename 	ScanOp,
	typename 	IdentityT>
__global__ void WarpScanKernel(
	T 			*d_in,
	T 			*d_out,
	ScanOp 		scan_op,
	IdentityT 	identity,
	T			prefix)
{
	// Cooperative warp-scan utility type (1 warp)
	typedef WarpScan<T, 1> WarpScan;

	// Shared memory
	__shared__ typename WarpScan::SmemStorage smem_storage;

	// Per-thread tile data
	T data = d_in[threadIdx.x];

	T aggregate;
	if (TEST_MODE == BASIC)
	{
		// Test basic warp scan
		WarpScan::ExclusiveScan(smem_storage, data, data, scan_op, identity);
	}
	else if (TEST_MODE == AGGREGATE)
	{
		// Test with warp-prefix and cumulative aggregate
		WarpScan::ExclusiveScan(smem_storage, data, data, scan_op, identity, aggregate);
	}
	else if (TEST_MODE == PREFIX_AGGREGATE)
	{
		// Test with warp-prefix and cumulative aggregate
		WarpScan::ExclusiveScan(smem_storage, data, data, scan_op, identity, aggregate, prefix);
	}

	// Store data
	d_out[threadIdx.x] = data;

	// Store aggregate
	if (threadIdx.x == 0)
	{
		d_out[blockDim.x] = aggregate;
	}
}


/**
 * Inclusive WarpScan test kernel.
 */
template <
	TestMode	TEST_MODE,
	typename 	T,
	typename 	ScanOp>
__global__ void WarpScanKernel(
	T 			*d_in,
	T 			*d_out,
	ScanOp 		scan_op,
	NullType,
	T			prefix)
{
	// Cooperative warp-scan utility type (1 warp)
	typedef WarpScan<T, 1> WarpScan;

	// Shared memory
	__shared__ typename WarpScan::SmemStorage smem_storage;

	// Per-thread tile data
	T data = d_in[threadIdx.x];

	T aggregate;
	if (TEST_MODE == BASIC)
	{
		// Test basic warp scan
		WarpScan::InclusiveScan(smem_storage, data, data, scan_op);
	}
	else if (TEST_MODE == AGGREGATE)
	{
		// Test with warp-prefix and cumulative aggregate
		WarpScan::InclusiveScan(smem_storage, data, data, scan_op, aggregate);
	}
	else if (TEST_MODE == PREFIX_AGGREGATE)
	{
		// Test with warp-prefix and cumulative aggregate
		WarpScan::InclusiveScan(smem_storage, data, data, scan_op, aggregate, prefix);
	}

	// Store data
	d_out[threadIdx.x] = data;

	// Store aggregate
	if (threadIdx.x == 0)
	{
		d_out[blockDim.x] = aggregate;
	}
}


//---------------------------------------------------------------------
// Host utility subroutines
//---------------------------------------------------------------------

/**
 * Initialize value at a given index
 */
template <typename T>
void InitValue(T &value, int index)
{
//	RandomBits(value);
//	value = 1;
	value = index;
}

/**
 * Initialize value at a given index.  Specialized for uint2.
 */
void InitValue(uint2 &value, int index)
{
//	RandomBits(value.x);
//	value.x = 1;
	value.x = index;

	value.y = value.x;
}


/**
 * Initialize exclusive-scan problem (and solution)
 */
template <
	typename 	T,
	typename 	ScanOp,
	typename 	IdentityT>
T Initialize(
	T 			*h_in,
	T 			*h_reference,
	int 		num_elements,
	ScanOp 		scan_op,
	IdentityT 	identity,
	T			*prefix)
{
	T inclusive = (prefix != NULL) ? *prefix : identity;

	for (int i = 0; i < num_elements; ++i)
	{
		InitValue(h_in[i], i);
		h_reference[i] = inclusive;
		inclusive = scan_op(inclusive, h_in[i]);
	}

	return inclusive;
}


/**
 * Initialize inclusive-scan problem (and solution)
 */
template <
	typename 	T,
	typename 	ScanOp>
T Initialize(
	T 			*h_in,
	T 			*h_reference,
	int 		num_elements,
	ScanOp 		scan_op,
	NullType,
	T			*prefix)
{
	T inclusive;
	for (int i = 0; i < num_elements; ++i)
	{
		InitValue(h_in[i], i);
		if (i == 0)
		{
			inclusive = (prefix != NULL) ?
				scan_op(*prefix, h_in[0]) :
				h_in[0];
		}
		else
		{
			inclusive = scan_op(inclusive, h_in[i]);
		}
		h_reference[i] = inclusive;
	}

	return inclusive;
}


/**
 * Test warp scan
 */
template <
	TestMode 	TEST_MODE,
	typename 	ScanOp,
	typename 	IdentityT,		// NullType for inclusive-scan
	typename 	T>
void Test(
	int 		warp_size,
	ScanOp 		scan_op,
	IdentityT 	identity,
	T			prefix)
{
	// Allocate host arrays
	T *h_in = new T[warp_size];
	T *h_reference = new T[warp_size];

	// Initialize problem
	T *p_prefix = (TEST_MODE == PREFIX_AGGREGATE) ? &prefix : NULL;
	T aggregate = Initialize(h_in, h_reference, warp_size, scan_op, identity, p_prefix);

	// Initialize device arrays
	T *d_in = NULL;
	T *d_out = NULL;
	DebugExit(hipMalloc((void**)&d_in, sizeof(T) * warp_size));
	DebugExit(hipMalloc((void**)&d_out, sizeof(T) * (warp_size + 1)));
	DebugExit(hipMemcpy(d_in, h_in, sizeof(T) * warp_size, hipMemcpyHostToDevice));

	// Run kernel
	printf("%s warpscan warp_size(%d) sizeof(T)(%d):\n",
		(Equals<IdentityT, NullType>::VALUE) ? "Inclusive" : "Exclusive",
		warp_size,
		(int) sizeof(T));
	fflush(stdout);

	// Run aggregate/prefix kernel
	WarpScanKernel<TEST_MODE><<<1, warp_size>>>(
		d_in,
		d_out,
		scan_op,
		identity,
		prefix);

	DebugExit(hipDeviceSynchronize());

	// Copy out and display results
	AssertEquals(0, CompareDeviceResults(h_reference, d_out, warp_size, g_verbose, g_verbose));
	printf("\n");

	// Copy out and display aggregate
	if ((TEST_MODE == AGGREGATE) || (TEST_MODE == PREFIX_AGGREGATE))
	{
		AssertEquals(0, CompareDeviceResults(&aggregate, d_out + warp_size, 1, g_verbose, g_verbose));
		printf("\n");
	}

	// Cleanup
	if (h_in) delete h_in;
	if (h_reference) delete h_in;
	if (d_in) DebugExit(hipFree(d_in));
	if (d_out) DebugExit(hipFree(d_out));
}


/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    DeviceInit(args);
    g_verbose = args.CheckCmdLineFlag("v");

    const int WARP_SIZE = 32;

    // int sum
    {
    	typedef int T;
    	Sum<T> scan_op;
    	T identity = 0;
    	T prefix = 99;

    	// Exclusive
    	Test<BASIC>(			WARP_SIZE, scan_op, identity, prefix);
    	Test<AGGREGATE>(		WARP_SIZE, scan_op, identity, prefix);
    	Test<PREFIX_AGGREGATE>(	WARP_SIZE, scan_op, identity, prefix);

    	// Inclusive
    	Test<BASIC>(			WARP_SIZE, scan_op, NullType(), prefix);
    	Test<AGGREGATE>(		WARP_SIZE, scan_op, NullType(), prefix);
    	Test<PREFIX_AGGREGATE>(	WARP_SIZE, scan_op, NullType(), prefix);
    }

    // uint max
    {
    	typedef unsigned int T;
    	Max<T> scan_op;
    	T identity = 0;
    	T prefix = 99;

    	// Exclusive
    	Test<BASIC>(			WARP_SIZE, scan_op, identity, prefix);
    	Test<AGGREGATE>(		WARP_SIZE, scan_op, identity, prefix);
    	Test<PREFIX_AGGREGATE>(	WARP_SIZE, scan_op, identity, prefix);

    	// Inclusive
    	Test<BASIC>(			WARP_SIZE, scan_op, NullType(), prefix);
    	Test<AGGREGATE>(		WARP_SIZE, scan_op, NullType(), prefix);
    	Test<PREFIX_AGGREGATE>(	WARP_SIZE, scan_op, NullType(), prefix);

    }

    // uint2 sum
    {
    	typedef uint2 T;
    	Uint2Sum scan_op;
    	T identity = scan_op();
    	T prefix = {14, 21};

    	// Exclusive
    	Test<BASIC>(			WARP_SIZE, scan_op, identity, prefix);
    	Test<AGGREGATE>(		WARP_SIZE, scan_op, identity, prefix);
    	Test<PREFIX_AGGREGATE>(	WARP_SIZE, scan_op, identity, prefix);

    	// Inclusive
    	Test<BASIC>(			WARP_SIZE, scan_op, NullType(), prefix);
    	Test<AGGREGATE>(		WARP_SIZE, scan_op, NullType(), prefix);
    	Test<PREFIX_AGGREGATE>(	WARP_SIZE, scan_op, NullType(), prefix);
    }

    return 0;
}



