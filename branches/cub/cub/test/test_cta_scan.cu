#include "hip/hip_runtime.h"
/******************************************************************************
 *
 * Copyright (c) 2010-2012, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2012, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of CtaScan utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>
#include <iostream>
#include <test_util.h>
#include "../cub.cuh"

using namespace hipcub;

//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

/**
 * Verbose output
 */
bool g_verbose = false;


/**
 * Primitive variant to test
 */
enum TestMode
{
	BASIC,
	AGGREGATE,
	PREFIX_AGGREGATE,
};

//---------------------------------------------------------------------
// Complex data type Foo
//---------------------------------------------------------------------

/**
 * Foo complex data type
 */
struct Foo
{
	long long 	x;
	int 		y;
	short 		z;
	char 		w;

	// Factory
	static __host__ __device__ __forceinline__ Foo MakeFoo(long long x, int y, short z, char w)
	{
		Foo retval = {x, y, z, w};
		return retval;
	}

	// Summation operator
	__host__ __device__ __forceinline__ Foo operator+(const Foo &b) const
	{
		return MakeFoo(x + b.x, y + b.y, z + b.z, w + b.w);
	}

	// Inequality operator
	__host__ __device__ __forceinline__ bool operator !=(const Foo &b)
	{
		return (x != b.x) && (y != b.y) && (z != b.z) && (w != b.w);
	}
};

/**
 * Foo ostream operator
 */
std::ostream& operator<<(std::ostream& os, const Foo& val)
{
	os << '(' << val.x << ',' << val.y << ',' << val.z << ',' << CoutCast(val.w) << ')';
	return os;
}

/**
 * Foo test initialization
 */
void InitValue(int gen_mode, Foo &value, int index = 0)
{
	InitValue(gen_mode, value.x, index);
	InitValue(gen_mode, value.y, index);
	InitValue(gen_mode, value.z, index);
	InitValue(gen_mode, value.w, index);
}


//---------------------------------------------------------------------
// Complex data type Bar (with optimizations for fence-free warp-synchrony)
//---------------------------------------------------------------------

/**
 * Bar complex data type
 */
struct Bar
{
	typedef void ThreadLoadTag;
	typedef void ThreadStoreTag;

	long long 	x;
	int 		y;

	// Factory
	static __host__ __device__ __forceinline__ Bar MakeBar(long long x, int y)
	{
		Bar retval = {x, y};
		return retval;
	}

	// Summation operator
	__host__ __device__ __forceinline__ Bar operator+(const Bar &b) const
	{
		return MakeBar(x + b.x, y + b.y);
	}

	// Inequality operator
	__host__ __device__ __forceinline__ bool operator !=(const Bar &b)
	{
		return (x != b.x) && (y != b.y);
	}

	// ThreadLoad
	template <LoadModifier MODIFIER>
	__device__ __forceinline__
	void ThreadLoad(Bar *ptr)
	{
		x = hipcub::ThreadLoad<MODIFIER>(&(ptr->x));
		y = hipcub::ThreadLoad<MODIFIER>(&(ptr->y));
	}

	 // ThreadStore
	template <StoreModifier MODIFIER>
	__device__ __forceinline__ void ThreadStore(Bar *ptr) const
	{
		hipcub::ThreadStore<MODIFIER>(&(ptr->x), x);
		hipcub::ThreadStore<MODIFIER>(&(ptr->y), y);
	}
};

/**
 * Bar ostream operator
 */
std::ostream& operator<<(std::ostream& os, const Bar& val)
{
	os << '(' << val.x << ',' << val.y << ')';
	return os;
}

/**
 * Bar test initialization
 */
void InitValue(int gen_mode, Bar &value, int index = 0)
{
	InitValue(gen_mode, value.x, index);
	InitValue(gen_mode, value.y, index);
}





//---------------------------------------------------------------------
// Test kernels
//---------------------------------------------------------------------

/**
 * Exclusive CtaScan test kernel.
 */
template <
	int 		CTA_THREADS,
	int			STRIPS_PER_TILE,
	int			ITEMS_PER_STRIP,
	TestMode	TEST_MODE,
	typename 	T,
	typename 	ScanOp,
	typename 	IdentityT>
__global__ void CtaScanKernel(
	T 			*d_in,
	T 			*d_out,
	ScanOp 		scan_op,
	IdentityT 	identity,
	T			prefix,
	clock_t		*d_elapsed)
{
	// Cooperative warp-scan utility type (1 warp)
	typedef CtaScan<T, CTA_THREADS, STRIPS_PER_TILE> CtaScan;

	// Shared memory
	__shared__ typename CtaScan::SmemStorage smem_storage;

	// Per-thread tile data
	T data[STRIPS_PER_TILE][ITEMS_PER_STRIP];

	// Load items
	CtaLoad<CTA_THREADS>::LoadUnguarded(data, d_in, 0);

	// Record elapsed clocks
	clock_t start = clock();


//	T aggregate;
	if (TEST_MODE == BASIC)
	{
		// Test basic warp scan
		CtaScan::ExclusiveScan(smem_storage, data, data, scan_op, identity);
	}
/*	else if (TEST_MODE == AGGREGATE)
	{
		// Test with cumulative aggregate
		CtaScan::ExclusiveScan(smem_storage, data, data, scan_op, identity, aggregate);
	}
	else if (TEST_MODE == PREFIX_AGGREGATE)
	{
		// Test with warp-prefix and cumulative aggregate
		CtaScan::ExclusiveScan(smem_storage, data, data, scan_op, identity, aggregate, prefix);
	}
*/
	// Record elapsed clocks
	*d_elapsed = clock() - start;

	// Store data
	CtaStore<CTA_THREADS>::StoreUnguarded(data, d_out, 0);

/*
	// Store aggregate
	if (threadIdx.x == 0)
	{
		d_out[blockDim.x] = aggregate;
	}
*/
}


/**
 * Inclusive CtaScan test kernel.
 */
template <
	int 		CTA_THREADS,
	int			STRIPS_PER_TILE,
	int			ITEMS_PER_STRIP,
	TestMode	TEST_MODE,
	typename 	T,
	typename 	ScanOp>
__global__ void CtaScanKernel(
	T 			*d_in,
	T 			*d_out,
	ScanOp 		scan_op,
	NullType,
	T			prefix,
	clock_t		*d_elapsed)
{
	// Cooperative warp-scan utility type (1 warp)
	typedef CtaScan<T, 1, CTA_THREADS> CtaScan;

	// Shared memory
	__shared__ typename CtaScan::SmemStorage smem_storage;

	// Per-thread tile data
	T data = d_in[threadIdx.x];

	// Record elapsed clocks
	clock_t start = clock();

	T aggregate;
/*
	if (TEST_MODE == BASIC)
	{
		// Test basic warp scan
		CtaScan::InclusiveScan(smem_storage, data, data, scan_op);
	}
	else if (TEST_MODE == AGGREGATE)
	{
		// Test with cumulative aggregate
		CtaScan::InclusiveScan(smem_storage, data, data, scan_op, aggregate);
	}
	else if (TEST_MODE == PREFIX_AGGREGATE)
	{
		// Test with warp-prefix and cumulative aggregate
		CtaScan::InclusiveScan(smem_storage, data, data, scan_op, aggregate, prefix);
	}
*/
	// Record elapsed clocks
	*d_elapsed = clock() - start;

	// Store data
	d_out[threadIdx.x] = data;

	// Store aggregate
	if (threadIdx.x == 0)
	{
		d_out[blockDim.x] = aggregate;
	}
}


//---------------------------------------------------------------------
// Host utility subroutines
//---------------------------------------------------------------------

/**
 * Initialize exclusive-scan problem (and solution)
 */
template <
	typename 	T,
	typename 	ScanOp,
	typename 	IdentityT>
T Initialize(
	int		 	gen_mode,
	T 			*h_in,
	T 			*h_reference,
	int 		num_elements,
	ScanOp 		scan_op,
	IdentityT 	identity,
	T			*prefix)
{
	T inclusive = (prefix != NULL) ? *prefix : identity;

	for (int i = 0; i < num_elements; ++i)
	{
		InitValue(gen_mode, h_in[i], i);
		h_reference[i] = inclusive;
		inclusive = scan_op(inclusive, h_in[i]);
	}

	return inclusive;
}


/**
 * Initialize inclusive-scan problem (and solution)
 */
template <
	typename 	T,
	typename 	ScanOp>
T Initialize(
	int		 	gen_mode,
	T 			*h_in,
	T 			*h_reference,
	int 		num_elements,
	ScanOp 		scan_op,
	NullType,
	T			*prefix)
{
	T inclusive;
	for (int i = 0; i < num_elements; ++i)
	{
		InitValue(gen_mode, h_in[i], i);
		if (i == 0)
		{
			inclusive = (prefix != NULL) ?
				scan_op(*prefix, h_in[0]) :
				h_in[0];
		}
		else
		{
			inclusive = scan_op(inclusive, h_in[i]);
		}
		h_reference[i] = inclusive;
	}

	return inclusive;
}


/**
 * Test warp scan
 */
template <
	int 		CTA_THREADS,
	int			STRIPS_PER_TILE,
	int			ITEMS_PER_STRIP,
	TestMode 	TEST_MODE,
	typename 	ScanOp,
	typename 	IdentityT,		// NullType implies inclusive-scan, otherwise inclusive scan
	typename 	T>
void Test(
	int 		gen_mode,
	ScanOp 		scan_op,
	IdentityT 	identity,
	T			prefix,
	char		*type_string)
{
	const int TILE_SIZE = CTA_THREADS * STRIPS_PER_TILE * ITEMS_PER_STRIP;

	// Allocate host arrays
	T *h_in = new T[TILE_SIZE];
	T *h_reference = new T[TILE_SIZE];

	// Initialize problem
	T *p_prefix = (TEST_MODE == PREFIX_AGGREGATE) ? &prefix : NULL;
	T aggregate = Initialize(gen_mode, h_in, h_reference, TILE_SIZE, scan_op, identity, p_prefix);

	// Initialize device arrays
	T *d_in = NULL;
	T *d_out = NULL;
	clock_t *d_elapsed = NULL;
	CubDebugExit(hipMalloc((void**)&d_in, sizeof(T) * TILE_SIZE));
	CubDebugExit(hipMalloc((void**)&d_out, sizeof(T) * (TILE_SIZE + 1)));
	CubDebugExit(hipMalloc((void**)&d_elapsed, sizeof(clock_t)));
	CubDebugExit(hipMemcpy(d_in, h_in, sizeof(T) * TILE_SIZE, hipMemcpyHostToDevice));

	// Run kernel
	printf("Test-mode %d, gen-mode %d, %s warpscan, %d warp threads, %s (%d bytes) elements:\n",
		TEST_MODE,
		gen_mode,
		(Equals<IdentityT, NullType>::VALUE) ? "Inclusive" : "Exclusive",
		CTA_THREADS,
		type_string,
		(int) sizeof(T));
	fflush(stdout);

	// Run aggregate/prefix kernel
	CtaScanKernel<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP, TEST_MODE><<<1, CTA_THREADS>>>(
		d_in,
		d_out,
		scan_op,
		identity,
		prefix,
		d_elapsed);

	if (g_verbose)
	{
		printf("\tElapsed clocks: ");
		DisplayDeviceResults(d_elapsed, 1);
	}

	CubDebugExit(hipDeviceSynchronize());

	// Copy out and display results
	printf("\tScan results: ");
	AssertEquals(0, CompareDeviceResults(h_reference, d_out, TILE_SIZE, g_verbose, g_verbose));
	printf("\n");

	// Copy out and display aggregate
	if ((TEST_MODE == AGGREGATE) || (TEST_MODE == PREFIX_AGGREGATE))
	{
		printf("\tScan aggregate: ");
		AssertEquals(0, CompareDeviceResults(&aggregate, d_out + TILE_SIZE, 1, g_verbose, g_verbose));
		printf("\n");
	}

	// Cleanup
	if (h_in) delete h_in;
	if (h_reference) delete h_in;
	if (d_in) CubDebugExit(hipFree(d_in));
	if (d_out) CubDebugExit(hipFree(d_out));
}


/**
 * Run battery of tests for different primitive variants
 */
template <
	int 		CTA_THREADS,
	int			STRIPS_PER_TILE,
	int			ITEMS_PER_STRIP,
	typename 	ScanOp,
	typename 	T>
void Test(
	int 		gen_mode,
	ScanOp 		scan_op,
	T 			identity,
	T			prefix,
	char *		type_string)
{
	// Exclusive
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP, BASIC>(gen_mode, scan_op, identity, prefix, type_string);
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP, AGGREGATE>(gen_mode, scan_op, identity, prefix, type_string);
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP, PREFIX_AGGREGATE>(gen_mode, scan_op, identity, prefix, type_string);

	// Inclusive
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP, BASIC>(gen_mode, scan_op, NullType(), prefix, type_string);
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP, AGGREGATE>(gen_mode, scan_op, NullType(), prefix, type_string);
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP, PREFIX_AGGREGATE>(gen_mode, scan_op, NullType(), prefix, type_string);
}


/**
 * Run battery of tests for different data types and scan ops
 */
template <int CTA_THREADS, int STRIPS_PER_TILE, int ITEMS_PER_STRIP>
void Test(int gen_mode)
{
	// primitive
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP>(gen_mode, Sum<unsigned char>(), (unsigned char) 0, (unsigned char) 99, CUB_TYPE_STRING(unsigned char));
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP>(gen_mode, Sum<unsigned short>(), (unsigned short) 0, (unsigned short) 99, CUB_TYPE_STRING(unsigned short));
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP>(gen_mode, Sum<unsigned int>(), (unsigned int) 0, (unsigned int) 99, CUB_TYPE_STRING(unsigned int));
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP>(gen_mode, Sum<unsigned long long>(), (unsigned long long) 0, (unsigned long long) 99, CUB_TYPE_STRING(unsigned long long));

	// primitive (alternative scan op)
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP>(gen_mode, Max<unsigned char>(), (unsigned char) 0, (unsigned char) 99, CUB_TYPE_STRING(unsigned char));
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP>(gen_mode, Max<unsigned short>(), (unsigned short) 0, (unsigned short) 99, CUB_TYPE_STRING(unsigned short));
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP>(gen_mode, Max<unsigned int>(), (unsigned int) 0, (unsigned int) 99, CUB_TYPE_STRING(unsigned int));
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP>(gen_mode, Max<unsigned long long>(), (unsigned long long) 0, (unsigned long long) 99, CUB_TYPE_STRING(unsigned long long));

	// vec-2
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP>(gen_mode, Sum<uchar2>(), make_uchar2(0, 0), make_uchar2(17, 21), CUB_TYPE_STRING(uchar2));
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP>(gen_mode, Sum<ushort2>(), make_ushort2(0, 0), make_ushort2(17, 21), CUB_TYPE_STRING(ushort2));
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP>(gen_mode, Sum<uint2>(), make_uint2(0, 0), make_uint2(17, 21), CUB_TYPE_STRING(uint2));
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP>(gen_mode, Sum<ulonglong2>(), make_ulonglong2(0, 0), make_ulonglong2(17, 21), CUB_TYPE_STRING(ulonglong2));

	// vec-4
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP>(gen_mode, Sum<uchar4>(), make_uchar4(0, 0, 0, 0), make_uchar4(17, 21, 32, 85), CUB_TYPE_STRING(uchar4));
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP>(gen_mode, Sum<ushort4>(), make_ushort4(0, 0, 0, 0), make_ushort4(17, 21, 32, 85), CUB_TYPE_STRING(ushort4));
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP>(gen_mode, Sum<uint4>(), make_uint4(0, 0, 0, 0), make_uint4(17, 21, 32, 85), CUB_TYPE_STRING(uint4));
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP>(gen_mode, Sum<ulonglong4>(), make_ulonglong4(0, 0, 0, 0), make_ulonglong4(17, 21, 32, 85), CUB_TYPE_STRING(ulonglong4));

	// complex
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP>(gen_mode, Sum<Foo>(), Foo::MakeFoo(0, 0, 0, 0), Foo::MakeFoo(17, 21, 32, 85), CUB_TYPE_STRING(Foo));
	Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP>(gen_mode, Sum<Bar>(), Bar::MakeBar(0, 0), Bar::MakeBar(17, 21), CUB_TYPE_STRING(Bar));
}


/**
 * Run battery of tests for different problem generation options
 */
template <
	int CTA_THREADS,
	int STRIPS_PER_TILE,
	int ITEMS_PER_STRIP>
void Test()
{
	for (int gen_mode = UNIFORM; gen_mode < GEN_MODE_END; gen_mode++)
	{
		Test<CTA_THREADS, STRIPS_PER_TILE, ITEMS_PER_STRIP>(gen_mode);
	}
}


/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    bool quick = args.CheckCmdLineFlag("quick");

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
    	printf("%s "
    		"[--device=<device-id>] "
    		"[--v] "
    		"[--quick]"
    		"\n", argv[0]);
    	exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());


//    if (quick)
    {
        // Quick exclusive test
        Test<128, 1, 4, BASIC>(UNIFORM, Sum<int>(), int(0), int(10), CUB_TYPE_STRING(int));
    }
/*
    else
    {
        // Test logical warp sizes
        Test<32>();
        Test<16>();
        Test<9>();
        Test<7>();
    }
*/

    // Flush any stdout from the kernel;
    hipDeviceSynchronize();

    return 0;
}



