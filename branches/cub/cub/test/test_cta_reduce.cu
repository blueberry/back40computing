#include "hip/hip_runtime.h"
/******************************************************************************
 *
 * Copyright (c) 2010-2012, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2012, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of CtaReduce utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>
#include "../cub.cuh"
#include <test_util.h>

using namespace hipcub;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool g_verbose = false;


//---------------------------------------------------------------------
// Test kernels
//---------------------------------------------------------------------

/**
 * Test full-tile reduction kernel (where num_elements is an even
 * multiple of CTA_THREADS)
 */
template <
	int 		CTA_THREADS,
	int 		ITEMS_PER_THREAD,
	typename 	T,
	typename 	ReductionOp>
__launch_bounds__ (CTA_THREADS, 1)
__global__ void FullTileReduceKernel(
	T 				*d_in,
	T 				*d_out,
	ReductionOp 	reduction_op,
	int				tiles)
{
	const int TILE_SIZE = CTA_THREADS * ITEMS_PER_THREAD;

	// Cooperative CTA reduction utility type (returns aggregate in thread 0)
	typedef CtaReduce<T, CTA_THREADS> CtaReduce;

	// Shared memory
	__shared__ typename CtaReduce::SmemStorage smem_storage;

	// Per-thread tile data
	T data[ITEMS_PER_THREAD];

	// Load first tile of data
	int cta_offset = 0;
	CtaLoadDirect(d_in + cta_offset, data);
	cta_offset += TILE_SIZE;

	// Cooperative reduce first tile
	T cta_aggregate = CtaReduce::Reduce(smem_storage, data, reduction_op);

	// Loop over input tiles
	while (cta_offset < TILE_SIZE * tiles)
	{
		// TestBarrier between CTA reductions
		__syncthreads();

		// Load tile of data
		CtaLoadDirect(d_in + cta_offset, data);
		cta_offset += TILE_SIZE;

		// Cooperatively reduce the tile's aggregate
		T tile_aggregate = CtaReduce::Reduce(smem_storage, data, reduction_op);

		// Reduce CTA aggregate
		cta_aggregate = reduction_op(cta_aggregate, tile_aggregate);
	}

	// Store data
	if (threadIdx.x == 0)
	{
		d_out[0] = cta_aggregate;
	}
}



/**
 * Test partial-tile reduction kernel (where num_elements < CTA_THREADS)
 */
template <
	int 		CTA_THREADS,
	typename 	T,
	typename 	ReductionOp>
__launch_bounds__ (CTA_THREADS, 1)
__global__ void PartialTileReduceKernel(
	T 				*d_in,
	T 				*d_out,
	int 			num_elements,
	ReductionOp 	reduction_op)
{
	// Cooperative CTA reduction utility type (returns aggregate only in thread-0)
	typedef CtaReduce<T, CTA_THREADS> CtaReduce;

	// Shared memory
	__shared__ typename CtaReduce::SmemStorage smem_storage;

	// Per-thread tile data
	T partial;

	// Load partial tile data
	if (threadIdx.x < num_elements)
	{
		partial = d_in[threadIdx.x];
	}

	// Cooperatively reduce the tile's aggregate
	T tile_aggregate = CtaReduce::Reduce(
		smem_storage,
		partial,
		reduction_op,
		num_elements);

	// Store data
	if (threadIdx.x == 0)
	{
		d_out[0] = tile_aggregate;
	}
}


//---------------------------------------------------------------------
// Host utility subroutines
//---------------------------------------------------------------------

/**
 * Initialize problem (and solution)
 */
template <typename T, typename ReductionOp>
void Initialize(
	int		 		gen_mode,
	T 				*h_in,
	T 				h_reference[1],
	ReductionOp 	reduction_op,
	int 			num_elements)
{
	for (int i = 0; i < num_elements; ++i)
	{
		InitValue(gen_mode, h_in[i], i);
		if (i == 0)
			h_reference[0] = h_in[0];
		else
			h_reference[0] = reduction_op(h_reference[0], h_in[i]);
	}
}


//---------------------------------------------------------------------
// Full tile test generation
//---------------------------------------------------------------------


/**
 * Test full-tile reduction
 */
template <
	int 		CTA_THREADS,
	int			ITEMS_PER_THREAD,
	typename 	T,
	typename 	ReductionOp>
void TestFullTile(
	int 			gen_mode,
	int 			tiles,
	ReductionOp 	reduction_op,
	char			*type_string)
{
	const int TILE_SIZE = CTA_THREADS * ITEMS_PER_THREAD;

	int num_elements = TILE_SIZE * tiles;

	// Allocate host arrays
	T *h_in = new T[num_elements];
	T h_reference[1];

	// Initialize problem
	Initialize(gen_mode, h_in, h_reference, reduction_op, num_elements);

	// Initialize device arrays
	T *d_in = NULL;
	T *d_out = NULL;
	CubDebugExit(hipMalloc((void**)&d_in, sizeof(T) * num_elements));
	CubDebugExit(hipMalloc((void**)&d_out, sizeof(T) * 1));
	CubDebugExit(hipMemcpy(d_in, h_in, sizeof(T) * num_elements, hipMemcpyHostToDevice));

	// Test multi-tile (unguarded)
	printf("TestFullTile, gen-mode %d, num_elements(%d), CTA_THREADS(%d), ITEMS_PER_THREAD(%d), %s (%d bytes) elements:\n",
		gen_mode,
		num_elements,
		CTA_THREADS,
		ITEMS_PER_THREAD,
		type_string,
		(int) sizeof(T));
	fflush(stdout);

	FullTileReduceKernel<CTA_THREADS, ITEMS_PER_THREAD><<<1, CTA_THREADS>>>(
		d_in,
		d_out,
		reduction_op,
		tiles);

	CubDebugExit(hipDeviceSynchronize());

	// Copy out and display results
	printf("\tReduction results: ");
	AssertEquals(0, CompareDeviceResults(h_reference, d_out, 1, g_verbose, g_verbose));
	printf("\n");

	// Cleanup
	if (h_in) delete[] h_in;
	if (d_in) CubDebugExit(hipFree(d_in));
	if (d_out) CubDebugExit(hipFree(d_out));
}

/**
 * Run battery of tests for different thread items
 */
template <
	int 		CTA_THREADS,
	typename 	T,
	typename 	ReductionOp>
void TestFullTile(
	int 			gen_mode,
	int 			tiles,
	ReductionOp 	reduction_op,
	char			*type_string)
{
	TestFullTile<CTA_THREADS, 1, T>(gen_mode, tiles, reduction_op, type_string);
	TestFullTile<CTA_THREADS, 4, T>(gen_mode, tiles, reduction_op, type_string);
}


/**
 * Run battery of full-tile tests for different cta sizes
 */
template <
	typename 	T,
	typename 	ReductionOp>
void TestFullTile(
	int 			gen_mode,
	int 			tiles,
	ReductionOp 	reduction_op,
	char			*type_string)
{
	TestFullTile<7, T>(gen_mode, tiles, reduction_op, type_string);
	TestFullTile<32, T>(gen_mode, tiles, reduction_op, type_string);
	TestFullTile<63, T>(gen_mode, tiles, reduction_op, type_string);
	TestFullTile<65, T>(gen_mode, tiles, reduction_op, type_string);
	TestFullTile<128, T>(gen_mode, tiles, reduction_op, type_string);
}


/**
 * Run battery of full-tile tests for different numbers of tiles
 */
template <
	typename 	T,
	typename 	ReductionOp>
void TestFullTile(
	int 			gen_mode,
	ReductionOp 	reduction_op,
	char			*type_string)
{
	for (int tiles = 1; tiles < 3; tiles++)
	{
		TestFullTile<T>(gen_mode, tiles, reduction_op, type_string);
	}
}


//---------------------------------------------------------------------
// Partial-tile test generation
//---------------------------------------------------------------------

/**
 * Test partial-tile reduction
 */
template <
	int 		CTA_THREADS,
	typename 	T,
	typename 	ReductionOp>
void TestPartialTile(
	int 			gen_mode,
	int 			num_elements,
	ReductionOp 	reduction_op,
	char			*type_string)
{
	const int TILE_SIZE = CTA_THREADS;

	// Allocate host arrays
	T *h_in = new T[num_elements];
	T h_reference[1];

	// Initialize problem
	Initialize(gen_mode, h_in, h_reference, reduction_op, num_elements);

	// Initialize device arrays
	T *d_in = NULL;
	T *d_out = NULL;
	CubDebugExit(hipMalloc((void**)&d_in, sizeof(T) * TILE_SIZE));
	CubDebugExit(hipMalloc((void**)&d_out, sizeof(T) * 1));
	CubDebugExit(hipMemcpy(d_in, h_in, sizeof(T) * num_elements, hipMemcpyHostToDevice));

	printf("TestPartialTile, gen-mode %d, num_elements(%d), CTA_THREADS(%d), %s (%d bytes) elements:\n",
		gen_mode,
		num_elements,
		CTA_THREADS,
		type_string,
		(int) sizeof(T));
	fflush(stdout);

	PartialTileReduceKernel<CTA_THREADS><<<1, CTA_THREADS>>>(
		d_in,
		d_out,
		num_elements,
		reduction_op);

	CubDebugExit(hipDeviceSynchronize());

	// Copy out and display results
	printf("\tReduction results: ");
	AssertEquals(0, CompareDeviceResults(h_reference, d_out, 1, g_verbose, g_verbose));
	printf("\n");

	// Cleanup
	if (h_in) delete[] h_in;
	if (d_in) CubDebugExit(hipFree(d_in));
	if (d_out) CubDebugExit(hipFree(d_out));
}


/**
 *  Run battery of partial-tile tests for different numbers of effective threads
 */
template <
	int 		CTA_THREADS,
	typename 	T,
	typename 	ReductionOp>
void TestPartialTile(
	int 			gen_mode,
	ReductionOp 	reduction_op,
	char			*type_string)
{
	for (
		int num_elements = 1;
		num_elements < CTA_THREADS;
		num_elements += CUB_MAX(1, CTA_THREADS / 5))
	{
		TestPartialTile<CTA_THREADS, T>(gen_mode, num_elements, reduction_op, type_string);
	}
}


/**
 * Run battery of full-tile tests for different cta sizes
 */
template <
	typename 	T,
	typename 	ReductionOp>
void TestPartialTile(
	int 			gen_mode,
	ReductionOp 	reduction_op,
	char			*type_string)
{
	TestPartialTile<7, T>(gen_mode, reduction_op, type_string);
	TestPartialTile<32, T>(gen_mode, reduction_op, type_string);
	TestPartialTile<63, T>(gen_mode, reduction_op, type_string);
	TestPartialTile<65, T>(gen_mode, reduction_op, type_string);
	TestPartialTile<128, T>(gen_mode, reduction_op, type_string);
}


//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Run battery of full-tile tests for different gen modes
 */
template <typename T, typename ReductionOp>
void Test(ReductionOp reduction_op, char *type_string)
{
	TestFullTile<T>(UNIFORM, reduction_op, type_string);
	TestPartialTile<T>(UNIFORM, reduction_op, type_string);

	TestFullTile<T>(SEQ_INC, reduction_op, type_string);
	TestPartialTile<T>(SEQ_INC, reduction_op, type_string);

	TestFullTile<T>(RANDOM, reduction_op, type_string);
	TestPartialTile<T>(RANDOM, reduction_op, type_string);
}


/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    bool quick = args.CheckCmdLineFlag("quick");

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
    	printf("%s "
    		"[--device=<device-id>] "
    		"[--v] "
    		"[--quick]"
    		"\n", argv[0]);
    	exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    if (quick)
    {
        // Quick test
    	typedef int T;
    	TestFullTile<128, 4, T>(UNIFORM, 1, Sum<T>(), CUB_TYPE_STRING(T));
    }
    else
    {
		// primitives
		Test<char>(Sum<char>(), CUB_TYPE_STRING(char));
		Test<short>(Sum<short>(), CUB_TYPE_STRING(short));
		Test<int>(Sum<int>(), CUB_TYPE_STRING(int));
		Test<long long>(Sum<long long>(), CUB_TYPE_STRING(long long));

		// vector types
		Test<char2>(Sum<char2>(), CUB_TYPE_STRING(char2));
		Test<short2>(Sum<short2>(), CUB_TYPE_STRING(short2));
		Test<int2>(Sum<int2>(), CUB_TYPE_STRING(int2));
		Test<longlong2>(Sum<longlong2>(), CUB_TYPE_STRING(longlong2));

		Test<char4>(Sum<char4>(), CUB_TYPE_STRING(char4));
		Test<short4>(Sum<short4>(), CUB_TYPE_STRING(short4));
		Test<int4>(Sum<int4>(), CUB_TYPE_STRING(int4));
		Test<longlong4>(Sum<longlong4>(), CUB_TYPE_STRING(longlong4));

		// Complex types
		Test<TestFoo>(Sum<TestFoo>(), CUB_TYPE_STRING(TestFoo));
		Test<TestBar>(Sum<TestBar>(), CUB_TYPE_STRING(TestBar));
    }

    return 0;
}



