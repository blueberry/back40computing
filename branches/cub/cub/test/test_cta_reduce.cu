#include "hip/hip_runtime.h"
/******************************************************************************
 *
 * Copyright (c) 2010-2012, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2012, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of CtaReduce utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>
#include "../cub.cuh"
#include <test_util.h>

using namespace hipcub;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool g_verbose = false;


//---------------------------------------------------------------------
// Complex data type Foo
//---------------------------------------------------------------------

/**
 * Foo complex data type
 */
struct Foo
{
	long long 	x;
	int 		y;
	short 		z;
	char 		w;

	// Factory
	static __host__ __device__ __forceinline__ Foo MakeFoo(long long x, int y, short z, char w)
	{
		Foo retval = {x, y, z, w};
		return retval;
	}

	// Summation operator
	__host__ __device__ __forceinline__ Foo operator+(const Foo &b) const
	{
		return MakeFoo(x + b.x, y + b.y, z + b.z, w + b.w);
	}

	// Inequality operator
	__host__ __device__ __forceinline__ bool operator !=(const Foo &b)
	{
		return (x != b.x) && (y != b.y) && (z != b.z) && (w != b.w);
	}
};

/**
 * Foo ostream operator
 */
std::ostream& operator<<(std::ostream& os, const Foo& val)
{
	os << '(' << val.x << ',' << val.y << ',' << val.z << ',' << CoutCast(val.w) << ')';
	return os;
}

/**
 * Foo test initialization
 */
void InitValue(int gen_mode, Foo &value, int index = 0)
{
	InitValue(gen_mode, value.x, index);
	InitValue(gen_mode, value.y, index);
	InitValue(gen_mode, value.z, index);
	InitValue(gen_mode, value.w, index);
}


//---------------------------------------------------------------------
// Complex data type Bar (with optimizations for fence-free warp-synchrony)
//---------------------------------------------------------------------

/**
 * Bar complex data type
 */
struct Bar
{
	typedef void ThreadLoadTag;
	typedef void ThreadStoreTag;

	long long 	x;
	int 		y;

	// Factory
	static __host__ __device__ __forceinline__ Bar MakeBar(long long x, int y)
	{
		Bar retval = {x, y};
		return retval;
	}

	// Summation operator
	__host__ __device__ __forceinline__ Bar operator+(const Bar &b) const
	{
		return MakeBar(x + b.x, y + b.y);
	}

	// Inequality operator
	__host__ __device__ __forceinline__ bool operator !=(const Bar &b)
	{
		return (x != b.x) && (y != b.y);
	}

	// ThreadLoad
	template <LoadModifier MODIFIER>
	__device__ __forceinline__
	void ThreadLoad(Bar *ptr)
	{
		x = hipcub::ThreadLoad<MODIFIER>(&(ptr->x));
		y = hipcub::ThreadLoad<MODIFIER>(&(ptr->y));
	}

	 // ThreadStore
	template <StoreModifier MODIFIER>
	__device__ __forceinline__ void ThreadStore(Bar *ptr) const
	{
		hipcub::ThreadStore<MODIFIER>(&(ptr->x), x);
		hipcub::ThreadStore<MODIFIER>(&(ptr->y), y);
	}
};

/**
 * Bar ostream operator
 */
std::ostream& operator<<(std::ostream& os, const Bar& val)
{
	os << '(' << val.x << ',' << val.y << ')';
	return os;
}

/**
 * Bar test initialization
 */
void InitValue(int gen_mode, Bar &value, int index = 0)
{
	InitValue(gen_mode, value.x, index);
	InitValue(gen_mode, value.y, index);
}


//---------------------------------------------------------------------
// Test kernels
//---------------------------------------------------------------------

/**
 * Test full-tile reduction kernel (where num_elements is an even
 * multiple of CTA_THREADS)
 */
template <
	int 		CTA_THREADS,
	int 		STRIPS,
	int 		ELEMENTS,
	typename 	T,
	typename 	ReductionOp>
__launch_bounds__ (CTA_THREADS, 1)
__global__ void FullTileReduceKernel(
	T 				*d_in,
	T 				*d_out,
	ReductionOp 	reduction_op,
	int				tiles)
{
	const int TILE_SIZE = CTA_THREADS * STRIPS * ELEMENTS;

	// Cooperative CTA reduction utility type (returns aggregate in thread 0)
	typedef CtaReduce<T, CTA_THREADS, STRIPS> CtaReduce;

	// Shared memory
	__shared__ typename CtaReduce::SmemStorage smem_storage;

	// Per-thread tile data
	T data[STRIPS][ELEMENTS];

	// Load first tile of data
	int cta_offset = 0;
	CtaLoad<CTA_THREADS>::LoadUnguarded(data, d_in, cta_offset);
	cta_offset += TILE_SIZE;

	// Cooperative reduce first tile
	T cta_aggregate = CtaReduce::Reduce(smem_storage, data, reduction_op);

	// Loop over input tiles
	while (cta_offset < TILE_SIZE * tiles)
	{
		// Barrier between CTA reductions
		__syncthreads();

		// Load tile of data
		CtaLoad<CTA_THREADS>::LoadUnguarded(data, d_in, cta_offset);
		cta_offset += TILE_SIZE;

		// Cooperatively reduce the tile's aggregate
		T tile_aggregate = CtaReduce::Reduce(smem_storage, data, reduction_op);

		// Reduce CTA aggregate
		cta_aggregate = reduction_op(cta_aggregate, tile_aggregate);
	}

	// Store data
	if (threadIdx.x == 0)
	{
		d_out[0] = cta_aggregate;
	}
}



/**
 * Test partial-tile reduction kernel (where num_elements < CTA_THREADS)
 */
template <
	int 		CTA_THREADS,
	typename 	T,
	typename 	ReductionOp>
__launch_bounds__ (CTA_THREADS, 1)
__global__ void PartialTileReduceKernel(
	T 				*d_in,
	T 				*d_out,
	int 			num_elements,
	ReductionOp 	reduction_op)
{
	// Cooperative CTA reduction utility type (returns aggregate only in thread-0)
	typedef CtaReduce<T, CTA_THREADS> CtaReduce;

	// Shared memory
	__shared__ typename CtaReduce::SmemStorage smem_storage;

	// Per-thread tile data
	T partial;

	// Load partial tile data
	if (threadIdx.x < num_elements)
	{
		partial = d_in[threadIdx.x];
	}

	// Cooperatively reduce the tile's aggregate
	T tile_aggregate = CtaReduce::Reduce(
		smem_storage,
		partial,
		num_elements,
		reduction_op);

	// Store data
	if (threadIdx.x == 0)
	{
		d_out[0] = tile_aggregate;
	}
}


//---------------------------------------------------------------------
// Host utility subroutines
//---------------------------------------------------------------------

/**
 * Initialize problem (and solution)
 */
template <typename T, typename ReductionOp>
void Initialize(
	int		 		gen_mode,
	T 				*h_in,
	T 				h_reference[1],
	ReductionOp 	reduction_op,
	int 			num_elements)
{
	for (int i = 0; i < num_elements; ++i)
	{
		InitValue(gen_mode, h_in[i], i);
		if (i == 0)
			h_reference[0] = h_in[0];
		else
			h_reference[0] = reduction_op(h_reference[0], h_in[i]);
	}
}


//---------------------------------------------------------------------
// Full tile test generation
//---------------------------------------------------------------------


/**
 * Test full-tile reduction
 */
template <
	int 		CTA_THREADS,
	int 		STRIPS,
	int			ELEMENTS,
	typename 	T,
	typename 	ReductionOp>
void TestFullTile(
	int 			gen_mode,
	int 			tiles,
	ReductionOp 	reduction_op,
	char			*type_string)
{
	const int TILE_SIZE = CTA_THREADS * STRIPS * ELEMENTS;

	int num_elements = TILE_SIZE * tiles;

	// Allocate host arrays
	T *h_in = new T[num_elements];
	T h_reference[1];

	// Initialize problem
	Initialize(gen_mode, h_in, h_reference, reduction_op, num_elements);

	// Initialize device arrays
	T *d_in = NULL;
	T *d_out = NULL;
	CubDebugExit(hipMalloc((void**)&d_in, sizeof(T) * num_elements));
	CubDebugExit(hipMalloc((void**)&d_out, sizeof(T) * 1));
	CubDebugExit(hipMemcpy(d_in, h_in, sizeof(T) * num_elements, hipMemcpyHostToDevice));

	// Test multi-tile (unguarded)
	printf("TestFullTile, gen-mode %d, num_elements(%d), CTA_THREADS(%d), STRIPS(%d), ELEMENTS(%d), %s (%d bytes) elements:\n",
		gen_mode,
		num_elements,
		CTA_THREADS,
		STRIPS,
		ELEMENTS,
		type_string,
		(int) sizeof(T));
	fflush(stdout);

	FullTileReduceKernel<CTA_THREADS, STRIPS, ELEMENTS><<<1, CTA_THREADS>>>(
		d_in,
		d_out,
		reduction_op,
		tiles);

	CubDebugExit(hipDeviceSynchronize());

	// Copy out and display results
	printf("\tReduction results: ");
	AssertEquals(0, CompareDeviceResults(h_reference, d_out, 1, g_verbose, g_verbose));
	printf("\n");

	// Cleanup
	if (h_in) free(h_in);
	if (d_in) CubDebugExit(hipFree(d_in));
	if (d_out) CubDebugExit(hipFree(d_out));
}

/**
 * Run battery of tests for different thread strip elements
 */
template <
	int 		CTA_THREADS,
	int 		STRIPS,
	typename 	T,
	typename 	ReductionOp>
void TestFullTile(
	int 			gen_mode,
	int 			tiles,
	ReductionOp 	reduction_op,
	char			*type_string)
{
	TestFullTile<CTA_THREADS, STRIPS, 1, T>(gen_mode, tiles, reduction_op, type_string);
//	TestFullTile<CTA_THREADS, STRIPS, 4, T>(gen_mode, tiles, reduction_op, type_string);
}


/**
 * Run battery of tests for different strips
 */
template <
	int 		CTA_THREADS,
	typename 	T,
	typename 	ReductionOp>
void TestFullTile(
	int 			gen_mode,
	int 			tiles,
	ReductionOp 	reduction_op,
	char			*type_string)
{
	TestFullTile<CTA_THREADS, 1, T>(gen_mode, tiles, reduction_op, type_string);
	TestFullTile<CTA_THREADS, 4, T>(gen_mode, tiles, reduction_op, type_string);
}


/**
 * Run battery of full-tile tests for different cta sizes
 */
template <
	typename 	T,
	typename 	ReductionOp>
void TestFullTile(
	int 			gen_mode,
	int 			tiles,
	ReductionOp 	reduction_op,
	char			*type_string)
{
	TestFullTile<7, T>(gen_mode, tiles, reduction_op, type_string);
	TestFullTile<32, T>(gen_mode, tiles, reduction_op, type_string);
	TestFullTile<63, T>(gen_mode, tiles, reduction_op, type_string);
	TestFullTile<65, T>(gen_mode, tiles, reduction_op, type_string);
	TestFullTile<128, T>(gen_mode, tiles, reduction_op, type_string);
}


/**
 * Run battery of full-tile tests for different numbers of tiles
 */
template <
	typename 	T,
	typename 	ReductionOp>
void TestFullTile(
	int 			gen_mode,
	ReductionOp 	reduction_op,
	char			*type_string)
{
	for (int tiles = 1; tiles < 3; tiles++)
	{
		TestFullTile<T>(gen_mode, tiles, reduction_op, type_string);
	}
}


//---------------------------------------------------------------------
// Partial-tile test generation
//---------------------------------------------------------------------

/**
 * Test partial-tile reduction
 */
template <
	int 		CTA_THREADS,
	typename 	T,
	typename 	ReductionOp>
void TestPartialTile(
	int 			gen_mode,
	int 			num_elements,
	ReductionOp 	reduction_op,
	char			*type_string)
{
	const int TILE_SIZE = CTA_THREADS;

	// Allocate host arrays
	T *h_in = new T[num_elements];
	T h_reference[1];

	// Initialize problem
	Initialize(gen_mode, h_in, h_reference, reduction_op, num_elements);

	// Initialize device arrays
	T *d_in = NULL;
	T *d_out = NULL;
	CubDebugExit(hipMalloc((void**)&d_in, sizeof(T) * TILE_SIZE));
	CubDebugExit(hipMalloc((void**)&d_out, sizeof(T) * 1));
	CubDebugExit(hipMemcpy(d_in, h_in, sizeof(T) * TILE_SIZE, hipMemcpyHostToDevice));

	printf("TestPartialTile, gen-mode %d, num_elements(%d), CTA_THREADS(%d), %s (%d bytes) elements:\n",
		gen_mode,
		num_elements,
		CTA_THREADS,
		type_string,
		(int) sizeof(T));
	fflush(stdout);

	PartialTileReduceKernel<CTA_THREADS><<<1, CTA_THREADS>>>(
		d_in,
		d_out,
		num_elements,
		reduction_op);

	CubDebugExit(hipDeviceSynchronize());

	// Copy out and display results
	printf("\tReduction results: ");
	AssertEquals(0, CompareDeviceResults(h_reference, d_out, 1, g_verbose, g_verbose));
	printf("\n");

	// Cleanup
	if (h_in) free(h_in);
	if (d_in) CubDebugExit(hipFree(d_in));
	if (d_out) CubDebugExit(hipFree(d_out));
}


/**
 *  Run battery of partial-tile tests for different numbers of effective threads
 */
template <
	int 		CTA_THREADS,
	typename 	T,
	typename 	ReductionOp>
void TestPartialTile(
	int 			gen_mode,
	ReductionOp 	reduction_op,
	char			*type_string)
{
	for (
		int num_elements = 1;
		num_elements < CTA_THREADS;
		num_elements += CUB_MAX(1, CTA_THREADS / 5))
	{
		TestPartialTile<CTA_THREADS, T>(gen_mode, num_elements, reduction_op, type_string);
	}
}


/**
 * Run battery of full-tile tests for different cta sizes
 */
template <
	typename 	T,
	typename 	ReductionOp>
void TestPartialTile(
	int 			gen_mode,
	ReductionOp 	reduction_op,
	char			*type_string)
{
	TestPartialTile<7, T>(gen_mode, reduction_op, type_string);
	TestPartialTile<32, T>(gen_mode, reduction_op, type_string);
	TestPartialTile<63, T>(gen_mode, reduction_op, type_string);
	TestPartialTile<65, T>(gen_mode, reduction_op, type_string);
	TestPartialTile<128, T>(gen_mode, reduction_op, type_string);
}


//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Run battery of full-tile tests for different gen modes
 */
template <typename T, typename ReductionOp>
void Test(ReductionOp reduction_op, char *type_string)
{
	for (int gen_mode = UNIFORM; gen_mode < GEN_MODE_END; gen_mode++)
	{
		TestFullTile<T>(gen_mode, reduction_op, type_string);
		TestPartialTile<T>(gen_mode, reduction_op, type_string);
	}
}


/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    bool quick = args.CheckCmdLineFlag("quick");

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
    	printf("%s "
    		"[--device=<device-id>] "
    		"[--v] "
    		"[--quick]"
    		"\n", argv[0]);
    	exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    if (quick)
    {
        // Quick test
    	typedef int T;
    	TestFullTile<128, 1, 4, T>(UNIFORM, 1, Sum<T>(), CUB_TYPE_STRING(T));
    }
    else
    {
		// primitives
		Test<char>(Sum<char>(), CUB_TYPE_STRING(char));
		Test<short>(Sum<short>(), CUB_TYPE_STRING(short));
		Test<int>(Sum<int>(), CUB_TYPE_STRING(int));
		Test<long long>(Sum<long long>(), CUB_TYPE_STRING(long long));

		// vector types
		Test<char2>(Sum<char2>(), CUB_TYPE_STRING(char2));
		Test<short2>(Sum<short2>(), CUB_TYPE_STRING(short2));
		Test<int2>(Sum<int2>(), CUB_TYPE_STRING(int2));
		Test<longlong2>(Sum<longlong2>(), CUB_TYPE_STRING(longlong2));

		Test<char4>(Sum<char4>(), CUB_TYPE_STRING(char4));
		Test<short4>(Sum<short4>(), CUB_TYPE_STRING(short4));
		Test<int4>(Sum<int4>(), CUB_TYPE_STRING(int4));
		Test<longlong4>(Sum<longlong4>(), CUB_TYPE_STRING(longlong4));

		// Complex types
		Test<Foo>(Sum<Foo>(), CUB_TYPE_STRING(Foo));
		Test<Bar>(Sum<Bar>(), CUB_TYPE_STRING(Bar));
    }

    return 0;
}



