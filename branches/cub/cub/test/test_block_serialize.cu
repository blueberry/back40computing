#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2013, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 *
 * Threadblock-serialization benchmark.  Each threadblock waits to proceed until the previous
 * completes.
 *
 *****************************************************************************/

#include <stdio.h>
#include <map>
#include <string>
#include <vector>
#include <sstream>
#include <iostream>

/******************************************************************************
 * Debug
 ******************************************************************************/

hipError_t Debug(hipError_t error, const char *funcname, const char *filename, int line)
{
    if (error) {
        fprintf(stderr, "[%s:%d %s] (CUDA error %d: %s)\n", filename, line, funcname, error, hipGetErrorString(error));
        fflush(stderr);
    }
    return error;
}

/**
 * Debug macro
 */
#define HipcubDebug(f) Debug(f, #f, __FILE__, __LINE__)


/**
 * Debug macro with exit
 */
#define CubDebugExit(f) if (Debug(f, #f, __FILE__, __LINE__)) exit(1)


/******************************************************************************
 * Command-line parsing functionality
 ******************************************************************************/

/**
 * Utility for parsing command line arguments
 */
class CommandLineArgs
{
protected:

    std::map<std::string, std::string> pairs;

public:

    /**
     * Constructor
     */
    CommandLineArgs(int argc, char **argv)
    {
        using namespace std;

        for (int i = 1; i < argc; i++)
        {
            string arg = argv[i];

            if ((arg[0] != '-') || (arg[1] != '-')) {
                continue;
            }

            string::size_type pos;
            string key, val;
            if ((pos = arg.find( '=')) == string::npos) {
                key = string(arg, 2, arg.length() - 2);
                val = "";
            } else {
                key = string(arg, 2, pos - 2);
                val = string(arg, pos + 1, arg.length() - 1);
            }
            pairs[key] = val;
        }
    }


    /**
     * Checks whether a flag "--<flag>" is present in the commandline
     */
    bool CheckCmdLineFlag(const char* arg_name)
    {
        using namespace std;
        map<string, string>::iterator itr;
        if ((itr = pairs.find(arg_name)) != pairs.end()) {
            return true;
        }
        return false;
    }


    /**
     * Returns the value specified for a given commandline parameter --<flag>=<value>
     */
    template <typename T>
    void GetCmdLineArgument(const char *arg_name, T &val)
    {
        using namespace std;
        map<string, string>::iterator itr;
        if ((itr = pairs.find(arg_name)) != pairs.end()) {
            istringstream str_stream(itr->second);
            str_stream >> val;
        }
    }


    /**
     * Returns the values specified for a given commandline parameter --<flag>=<value>,<value>*
     */
    template <typename T>
    void GetCmdLineArguments(const char *arg_name, std::vector<T> &vals)
    {
        using namespace std;

        // Recover multi-value string
        map<string, string>::iterator itr;
        if ((itr = pairs.find(arg_name)) != pairs.end()) {

            // Clear any default values
            vals.clear();

            string val_string = itr->second;
            istringstream str_stream(val_string);
            string::size_type old_pos = 0;
            string::size_type new_pos = 0;

            // Iterate comma-separated values
            T val;
            while ((new_pos = val_string.find(',', old_pos)) != string::npos) {

                if (new_pos != old_pos) {
                    str_stream.width(new_pos - old_pos);
                    str_stream >> val;
                    vals.push_back(val);
                }

                // skip over comma
                str_stream.ignore(1);
                old_pos = new_pos + 1;
            }

            // Read last value
            str_stream >> val;
            vals.push_back(val);
        }
    }


    /**
     * The number of pairs parsed
     */
    int ParsedArgc()
    {
        return pairs.size();
    }

    /**
     * Initialize device
     */
    hipError_t DeviceInit(int dev = -1)
    {
        hipError_t error = hipSuccess;

        do {
            int deviceCount;
            error = HipcubDebug(hipGetDeviceCount(&deviceCount));
            if (error) break;

            if (deviceCount == 0) {
                fprintf(stderr, "No devices supporting CUDA.\n");
                exit(1);
            }
            if (dev < 0)
            {
                GetCmdLineArgument("device", dev);
            }
            if ((dev > deviceCount - 1) || (dev < 0))
            {
                dev = 0;
            }

            hipDeviceProp_t deviceProp;
            error = HipcubDebug(hipGetDeviceProperties(&deviceProp, dev));
            if (error) break;

            if (deviceProp.major < 1) {
                fprintf(stderr, "Device does not support CUDA.\n");
                exit(1);
            }
            if (!CheckCmdLineFlag("quiet")) {
                printf("Using device %d: %s\n", dev, deviceProp.name);
                fflush(stdout);
            }

            error = HipcubDebug(hipSetDevice(dev));
            if (error) break;

        } while (0);

        return error;
    }
};


/******************************************************************************
 * Timing
 ******************************************************************************/

/**
 * Flag-based kernel performance timer
 */
struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float ElapsedMillis()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};


/******************************************************************************
 * GPU device routines
 ******************************************************************************/

/**
 * Load global
 */
__device__ __forceinline__ int LoadCg(int *ptr)
{
    int val;

#if defined(_WIN64) || defined(__LP64__)
    asm volatile ("ld.global.cg.s32 %0, [%1];" : "=r"(val) : "l"(ptr));
#else
    asm volatile ("ld.global.cg.s32 %0, [%1];" : "=r"(val) : "r"(ptr));
#endif

    return val;
}

/**
 * Block-serialization kernel.  Each threadblock waits to proceed until the previous completes.
 */
__global__ void Kernel(
    int *d_progress,    ///< Counter indicating which threadblock is allowed to complete next
    int *d_block_id)    ///< Counter for obtaining a "resident" block ID
{
    __shared__ int sblock_id;

    // Get a unique block ID that guarantees the previous block
    // has already activated.
    if (threadIdx.x == 0) sblock_id = atomicAdd(d_block_id, 1);

    __syncthreads();

    int block_id = sblock_id;
    if (threadIdx.x == 0)
    {
        if (block_id != 0)
        {
            // Wait for previous block to complete
            while (true)
            {
                if (LoadCg(d_progress) == block_id) break;
                if (LoadCg(d_progress) == block_id) break;
            }
        }

        // Signal the next threadblock
        *d_progress = block_id + 1;
    }
}


/**
 * Block-serialization kernel.  Each threadblock waits to proceed until the previous completes.
 * Prints clocks counts
 */
__global__ void Kernel2(
    int *d_progress,    ///< Counter indicating which threadblock is allowed to complete next
    int *d_block_id,    ///< Counter for obtaining a "resident" block ID
    int *d_clocks,
    int *d_wait_cycles)
{
    if (threadIdx.x == 0)
    {
        int iterations = 0;
        int a = clock();
        if (blockIdx.x > 0)
        {
            // Wait for previous block to complete
            while (true)
            {
                if (LoadCg(d_progress) == blockIdx.x) break;
                iterations++;
                if (LoadCg(d_progress) == blockIdx.x) break;
                iterations++;
            }
        }
        d_progress[0] = blockIdx.x + 1;
        int b = clock();

        d_clocks[blockIdx.x] = b - a;
        d_wait_cycles[blockIdx.x] = iterations;
    }
}


/**
 * Main
 */
int main(int argc, char** argv)
{
    int iterations      = 100;
    int num_blocks        = 1024 * 63;
    int block_size        = 32;
    int occupancy       = -1;

    CommandLineArgs args(argc, argv);
    CubDebugExit(args.DeviceInit());
    args.GetCmdLineArgument("i", iterations);
    args.GetCmdLineArgument("num-blocks", num_blocks);
    args.GetCmdLineArgument("block-size", block_size);
    args.GetCmdLineArgument("occupancy", occupancy);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>] "
            "[--i=<iterations>] "
            "[--block-size=<block-size>] "
            "[--num-blocks=<num-blocks>] "
            "[--occupancy=<occupancy>] "
            "\n", argv[0]);
        exit(0);
    }

    int dynamic_smem = (occupancy <= 0) ?
        0 :
        (1024 * 48 - 128) / occupancy;

    printf("%d iterations of Kernel<<<%d, %d, %d>>>(...)\n", iterations, num_blocks, block_size, dynamic_smem);
    fflush(stdout);

    // Device storage
    int *d_progress, *d_block_id, *d_clocks, *d_wait_cycles;

    // Allocate device words
    CubDebugExit(hipMalloc((void**)&d_progress, sizeof(int)));
    CubDebugExit(hipMalloc((void**)&d_block_id, sizeof(int)));

    /**
     * Experiment 1: threadblock-serialization throughput.
     */

    printf("Experiment 1: threadblock-serialization throughput.\n");
    fflush(stdout);


    GpuTimer gpu_timer;
    float elapsed_millis = 0.0;
    for (int i = 0; i < iterations; i++)
    {
        // Zero-out the counters
        CubDebugExit(hipMemset(d_progress, 0, sizeof(int)));
        CubDebugExit(hipMemset(d_block_id, 0, sizeof(int)));

        gpu_timer.Start();

        Kernel<<<num_blocks, block_size, dynamic_smem>>>(d_progress, d_block_id);

        gpu_timer.Stop();
        elapsed_millis += gpu_timer.ElapsedMillis();
    }
    float avg_elapsed = elapsed_millis / iterations;

    printf("%d iterations, average elapsed (%.4f ms), %.4f M threadblocks/s\n",
        iterations,
        avg_elapsed,
        float(num_blocks) / avg_elapsed / 1000.0);
    fflush(stdout);



    /**
     * Experiment 2: Launch 1 "sequentialized" threadblock per SM, record the number of clocks
     * elapsed until each is able to reture.
     */

    printf("\n\nBenchmark 2: clocks per retired threadblock\n");
    fflush(stdout);

    num_blocks = 7;
    block_size = 32;
    CubDebugExit(hipMalloc((void**)&d_clocks, sizeof(int) * num_blocks));
    CubDebugExit(hipMalloc((void**)&d_wait_cycles, sizeof(int) * num_blocks));
    CubDebugExit(hipMemset(d_clocks, 0, sizeof(int) * num_blocks));
    CubDebugExit(hipMemset(d_wait_cycles, 0, sizeof(int) * num_blocks));

    CubDebugExit(hipMemset(d_progress, 0, sizeof(int)));
    CubDebugExit(hipMemset(d_block_id, 0, sizeof(int)));
    Kernel2<<<num_blocks, block_size, 1024 * 40>>>(d_progress, d_block_id, d_clocks, d_wait_cycles);

    int *h_clocks = new int[num_blocks];
    int *h_wait_cycles = new int[num_blocks];

    HipcubDebug(hipMemcpy(h_clocks, d_clocks, sizeof(int) * num_blocks, hipMemcpyDeviceToHost));
    HipcubDebug(hipMemcpy(h_wait_cycles, d_wait_cycles, sizeof(int) * num_blocks, hipMemcpyDeviceToHost));

    for (int i = 0; i < num_blocks; i++)
    {
        printf("Block %d clocks(%d) wait_cycles(%d), avg clocks per predecessor(%.2f)\n",
            i, h_clocks[i], h_wait_cycles[i], (i == 0) ? 0 : float(h_clocks[i]) / i);
    }

    // Force any kernel stdio to screen
    CubDebugExit(hipDeviceSynchronize());

    // Cleanup
    CubDebugExit(hipFree(d_progress));
    CubDebugExit(hipFree(d_block_id));
    CubDebugExit(hipFree(d_clocks));
    CubDebugExit(hipFree(d_wait_cycles));

    delete[] h_clocks;
    delete[] h_wait_cycles;

    return 0;
}
