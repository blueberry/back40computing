#include "hip/hip_runtime.h"
/******************************************************************************
 *
 * Copyright (c) 2010-2012, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2012, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of CtaReduce utilities
 ******************************************************************************/

#define HIPCUB_STDERR

#include <stdio.h>
#include <hipcub/hipcub.hpp>
#include <test_util.h>

using namespace hipcub;

bool g_verbose = false;


//---------------------------------------------------------------------
// Kernels
//---------------------------------------------------------------------

/**
 * Test unguarded load/store kernel.
 */
template <
	int CTA_THREADS,
	int CTA_STRIDES,
	typename T,
	typename ReductionOp>
__global__ void UnguardedKernel(
	T *d_in,
	T *d_out,
	ReductionOp reduction_op,
	int iterations)
{
	typedef CtaLoad<CTA_THREADS> CtaLoad;
	typedef CtaReduce<CTA_THREADS, T, CTA_STRIDES> CtaReduce;

	__shared__ typename CtaReduce::SmemStorage smem_storage;

	// Data
	T data[CTA_STRIDES][1];

	// Load data
	int cta_offset = 0;
	CtaLoad::LoadUnguarded(data, d_in, cta_offset);
	cta_offset += CTA_THREADS;

	// Cooperative reduce
	T partial = CtaReduce::Reduce(smem_storage, data, reduction_op);

	while (cta_offset < CTA_THREADS * iterations)
	{
		// Load data
		T next = d_in[cta_offset + threadIdx.x];
		cta_offset += CTA_THREADS;

		// Cooperative reduce
		next = CtaReduce::Reduce(smem_storage, data, reduction_op);
		partial = reduction_op(partial, next);
	}

	// Store data
	if (threadIdx.x == 0)
	{
		d_out[0] = partial;
	}
}


/**
 * Test guarded load/store kernel.
 */
template <int CTA_THREADS, typename T, typename ReductionOp>
__global__ void GuardedKernel(
	T *d_in,
	T *d_out,
	int num_elements,
	ReductionOp reduction_op)
{
	typedef CtaLoad<CTA_THREADS> CtaLoad;
	typedef CtaReduce<CTA_THREADS, T> CtaReduce;

	__shared__ typename CtaReduce::SmemStorage smem_storage;

	T partial;

	// Load data
	if (threadIdx.x < num_elements) {
		partial = d_in[threadIdx.x];
	}

	// Cooperative reduce
	partial = CtaReduce::Reduce(smem_storage, partial, num_elements, reduction_op);

	// Store data
	if (threadIdx.x == 0)
	{
		d_out[0] = partial;
	}
}


//---------------------------------------------------------------------
// Test routines
//---------------------------------------------------------------------

/**
 * Uint2 reduction operator
 */
struct Uint2Sum
{
	__host__ __device__ __forceinline__ uint2 operator()(uint2 a, uint2 b)
	{
		a.x += b.x;
		a.y += b.y;
		return a;
	}
};


/**
 * Initialize problem (and solution)
 */
template <typename T, typename ReductionOp>
void Initialize(
	T *h_in,
	T h_result[1],
	ReductionOp reduction_op,
	int num_elements)
{
	for (int i = 0; i < num_elements; ++i)
	{
		RandomBits(h_in[i]);
//		h_in[i] = 1;
//		h_in[i] = i;
		if (i == 0)
			h_result[0] = h_in[0];
		else
			h_result[0] = reduction_op(h_result[0], h_in[i]);
	}
}


/**
 * Test reduction
 */
template <int CTA_THREADS, int CTA_STRIDES, typename T, typename ReductionOp>
void Test(int num_elements, ReductionOp reduction_op)
{
	const int TILE_SIZE = CTA_THREADS * CTA_STRIDES;

	// Allocate host arrays
	T h_in[TILE_SIZE];
	T h_result[1];

	// Initialize problem
	Initialize(h_in, h_result, reduction_op, num_elements);

	// Initialize device arrays
	T *d_in = NULL;
	T *d_out = NULL;
	DebugExit(hipMalloc((void**)&d_in, sizeof(T) * TILE_SIZE));
	DebugExit(hipMalloc((void**)&d_out, sizeof(T) * 1));
	DebugExit(hipMemcpy(d_in, h_in, sizeof(T) * TILE_SIZE, hipMemcpyHostToDevice));

	// Run kernel
	if (num_elements == TILE_SIZE)
	{
		// Test unguarded
		printf("Unguarded test CTA_THREADS(%d) CTA_STRIDES(%d) sizeof(T)(%d):\n\t ",
			CTA_THREADS, CTA_STRIDES, (int) sizeof(T));
		fflush(stdout);

		UnguardedKernel<CTA_THREADS, CTA_STRIDES><<<1, CTA_THREADS>>>(
			d_in, d_out, reduction_op, 1);
	}
	else
	{
		// Test guarded
		printf("Guarded test CTA_THREADS(%d) num_elements(%d) sizeof(T)(%d):\n\t ",
			CTA_THREADS, num_elements, (int) sizeof(T));
		fflush(stdout);

		GuardedKernel<CTA_THREADS><<<1, CTA_THREADS>>>(
			d_in, d_out, num_elements, reduction_op);
	}

	DebugExit(hipDeviceSynchronize());

	// Copy out and display results
	AssertEquals(0, CompareDeviceResults(h_result, d_out, 1, g_verbose, g_verbose));
	printf("\n");

	// Cleanup
	if (d_in) DebugExit(hipFree(d_in));
	if (d_out) DebugExit(hipFree(d_out));
}


/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    DeviceInit(args);
    g_verbose = args.CheckCmdLineFlag("v");

    Test<32, 	1, int>(32, 	Sum<int>());
    Test<8, 	1, int>(8, 		Sum<int>());
    Test<23, 	1, int>(23, 	Sum<int>());
    Test<512, 	1, int>(512, 	Sum<int>());
    Test<121,	1, int>(121, 	Sum<int>());
    Test<133, 	1, int>(133, 	Sum<int>());
    Test<96, 	1, int>(96, 	Sum<int>());
    Test<32, 	1, int>(12, 	Sum<int>());
    Test<512, 	1, int>(509,	Sum<int>());
    Test<32,	1, uint2>(32, 	Uint2Sum());
    Test<512,	1, uint2>(512, 	Uint2Sum());
    Test<512, 	1, uint2>(509,	Uint2Sum());
    Test<128, 	2, int>(256, 	Sum<int>());
    Test<32, 	2, int>(64, 	Sum<int>());
    Test<16, 	2, int>(32, 	Sum<int>());
    Test<55, 	2, int>(110, 	Sum<int>());
    Test<23, 	2, int>(46, 	Sum<int>());

	return 0;
}



