#include "hip/hip_runtime.h"
/******************************************************************************
 * 
 * Copyright 2010-2012 Duane Merrill
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 ******************************************************************************/


/******************************************************************************
 * Tuning tool for establishing optimal scan granularity configuration types
 ******************************************************************************/

#include <stdio.h> 

#include <map>
#include <vector>

#include <b40c/util/arch_dispatch.cuh>
#include <b40c/util/cuda_properties.cuh>
#include <b40c/util/numeric_traits.cuh>
#include <b40c/util/parameter_generation.cuh>
#include <b40c/util/enactor_base.cuh>
#include <b40c/util/spine.cuh>

#include <b40c/scan/problem_type.cuh>
#include <b40c/scan/policy.cuh>

// Test utils
#include "b40c_test_util.h"

using namespace b40c;


/******************************************************************************
 * Defines, constants, globals, and utility types
 ******************************************************************************/

#ifndef TUNE_ARCH
	#define TUNE_ARCH (200)
#endif
#ifndef TUNE_SIZE
	#define TUNE_SIZE (4)
#endif

bool 	g_verbose;
int 	g_max_ctas = 0;
int 	g_iterations = 0;
bool 	g_verify;
int 	g_policy_id = 0;


struct KernelDetails
{
	int threads;
	int tile_elements;

	KernelDetails(
		int threads,
		int tile_elements) :
			threads(threads),
			tile_elements(tile_elements) {}
};

struct PassDetails
{
	int uniform_smem_allocation;
	int uniform_grid_size;
	int over_subscribed;

	// Factory initializer
	PassDetails (
		int uniform_smem_allocation,
		int uniform_grid_size,
		int over_subscribed) :
			uniform_smem_allocation(uniform_smem_allocation),
			uniform_grid_size(uniform_grid_size),
			over_subscribed(over_subscribed) {}

	// CSV string format
	std::string TypeString()
	{
		char buffer[1024];
		sprintf(buffer, "%s, %s, %s",
			uniform_smem_allocation 		? "true" : "false",
			uniform_grid_size 				? "true" : "false",
			over_subscribed 				? "true" : "false");
		return buffer;
	}
};




/******************************************************************************
 * Test wrappers for binary, associative operations
 ******************************************************************************/

template <typename T>
struct Sum
{
	// Binary reduction
	__host__ __device__ __forceinline__ T operator()(const T &a, const T &b)
	{
		return a + b;
	}

	// Identity
	__host__ __device__ __forceinline__ T operator()()
	{
		return 0;
	}

	static const bool IS_COMMUTATIVE = true;
};

template <typename T>
struct Max
{
	// Binary reduction
	__host__ __device__ __forceinline__ T Op(const T &a, const T &b)
	{
		return (a > b) ? a : b;
	}

	// Identity
	__host__ __device__ __forceinline__ T operator()()
	{
		return 0;
	}

	static const bool IS_COMMUTATIVE = true;
};



/******************************************************************************
 * Utility routines
 ******************************************************************************/

/**
 * Displays the commandline usage for this tool
 */
void Usage()
{
	printf("\ntune_scan [--device=<device index>] [--v] [--i=<num-iterations>] "
			"[--max-ctas=<max-thread-blocks>] [--n=<num-words>] [--verify]\n");
	printf("\n");
	printf("\t--v\tDisplays verbose configuration to the console.\n");
	printf("\n");
	printf("\t--verify\tChecks the result.\n");
	printf("\n");
	printf("\t--i\tPerforms the operation <num-iterations> times\n");
	printf("\t\t\ton the device. Default = 1\n");
	printf("\n");
	printf("\t--n\tThe number of 32-bit words to comprise the sample problem\n");
	printf("\n");
	printf("\t--max-ctas\tThe number of CTAs to launch\n");
	printf("\n");
}


/******************************************************************************
 * Upsweep Tuning Parameter Enumerations and Ranges
 ******************************************************************************/

struct UpsweepTuningRanges
{
	/**
	 * Tuning params
	 */
	enum Param
	{
		BEGIN,
			LOG_THREADS,
			LOG_LOAD_VEC_SIZE,
			LOG_LOADS_PER_TILE,
			LOG_SCHEDULE_GRANULARITY,
		END,
	};


	/**
	 * Ranges for the tuning params
	 */
	template <typename ParamList, int PARAM> struct Ranges;

	// LOG_THREADS
	template <typename ParamList>
	struct Ranges<ParamList, LOG_THREADS> {
		enum {
			MIN = 5,	// 32
			MAX = 10	// 1024
		};
	};

	// LOG_LOAD_VEC_SIZE
	template <typename ParamList>
	struct Ranges<ParamList, LOG_LOAD_VEC_SIZE> {
		enum {
			MIN = 0,
			MAX = 2
		};
	};

	// LOG_LOADS_PER_TILE
	template <typename ParamList>
	struct Ranges<ParamList, LOG_LOADS_PER_TILE> {
		enum {
			MIN = 0,
			MAX = 2
		};
	};

	// LOG_SCHEDULE_GRANULARITY
	template <typename ParamList>
	struct Ranges<ParamList, LOG_SCHEDULE_GRANULARITY> {
		enum {
			MIN = util::Access<ParamList, LOG_THREADS>::VALUE +
				util::Access<ParamList, LOG_LOAD_VEC_SIZE>::VALUE +
				util::Access<ParamList, LOG_LOADS_PER_TILE>::VALUE,
			MAX = Ranges<ParamList, LOG_THREADS>::MAX +
				Ranges<ParamList, LOG_LOAD_VEC_SIZE>::MAX +
				Ranges<ParamList, LOG_LOADS_PER_TILE>::MAX
		};
	};


	/**
	 * Policy
	 */
	template <
		typename ProblemType,
		typename ParamList,
		typename BaseKernelPolicy = scan::KernelPolicy <
			ProblemType,
			TUNE_ARCH,
			true,														// CHECK_ALIGNMENT
			0,															// MIN_CTA_OCCUPANCY,
			util::Access<ParamList, LOG_THREADS>::VALUE, 				// LOG_THREADS,
			util::Access<ParamList, LOG_LOAD_VEC_SIZE>::VALUE,			// LOG_LOAD_VEC_SIZE,
			util::Access<ParamList, LOG_LOADS_PER_TILE>::VALUE,			// LOG_LOADS_PER_TILE,
			CUB_LOG_WARP_THREADS(TUNE_ARCH),							// LOG_RAKING_THREADS,
			util::io::ld::NONE,											// LOAD_MODIFIER,
			util::io::st::NONE,											// STORE_MODIFIER,
			util::Access<ParamList, LOG_SCHEDULE_GRANULARITY>::VALUE> >	// LOG_SCHEDULE_GRANULARITY
	struct KernelPolicy : BaseKernelPolicy
	{
		typedef typename ProblemType::T T;
		typedef typename ProblemType::SizeT SizeT;
		typedef typename ProblemType::ReductionOp ReductionOp;
		typedef typename ProblemType::IdentityOp IdentityOp;

		typedef void (*KernelPtr)(T*, T*, ReductionOp, IdentityOp, util::CtaWorkDistribution<SizeT>);

		// Check if this configuration is worth compiling
		enum {
			REG_MULTIPLIER = (sizeof(T) + 4 - 1) / 4,
			REGS_ESTIMATE = (REG_MULTIPLIER * KernelPolicy::TILE_ELEMENTS_PER_THREAD) + 2,
			EST_REGS_OCCUPANCY = CUB_SM_REGISTERS(TUNE_ARCH) / (REGS_ESTIMATE * KernelPolicy::THREADS),

			VALID_COMPILE =
				((BaseKernelPolicy::VALID > 0) &&
				((TUNE_ARCH >= 200) || (BaseKernelPolicy::LOAD_MODIFIER == util::io::ld::NONE)) &&
				((TUNE_ARCH >= 200) || (BaseKernelPolicy::STORE_MODIFIER == util::io::st::NONE)) &&
				(BaseKernelPolicy::LOG_THREADS <= CUB_LOG_CTA_THREADS(TUNE_ARCH)) &&
				(EST_REGS_OCCUPANCY > 0)),
		};

		static std::string TypeString()
		{
			char buffer[32];
			sprintf(buffer, "%d, %d, %d",
				KernelPolicy::LOG_THREADS,
				KernelPolicy::LOG_LOAD_VEC_SIZE,
				KernelPolicy::LOG_LOADS_PER_TILE);
			return buffer;
		}

		template <int VALID, int DUMMY = 0>
		struct GenKernel
		{
			static KernelPtr Kernel() {
				return scan::upsweep::Kernel<KernelPolicy>;
			}
		};

		template <int DUMMY>
		struct GenKernel<0, DUMMY>
		{
			static KernelPtr Kernel() {
				return NULL;
			}
		};

		static KernelPtr Kernel() {
			return GenKernel<VALID_COMPILE>::Kernel();
		}
	};

};


/******************************************************************************
 * Spine Tuning Parameter Enumerations and Ranges
 ******************************************************************************/

struct SpineTuningRanges
{
	/**
	 * Tuning params
	 */
	enum Param
	{
		BEGIN,
			LOG_THREADS,
			LOG_LOAD_VEC_SIZE,
			LOG_LOADS_PER_TILE,
		END,
	};


	/**
	 * Ranges for the tuning params
	 */
	template <typename ParamList, int PARAM> struct Ranges;

	// LOG_THREADS
	template <typename ParamList>
	struct Ranges<ParamList, LOG_THREADS> {
		enum {
			MIN = 5,	// 32
			MAX = 10	// 1024
		};
	};

	// LOG_LOAD_VEC_SIZE
	template <typename ParamList>
	struct Ranges<ParamList, LOG_LOAD_VEC_SIZE> {
		enum {
			MIN = 0,
			MAX = 2
		};
	};

	// LOG_LOADS_PER_TILE
	template <typename ParamList>
	struct Ranges<ParamList, LOG_LOADS_PER_TILE> {
		enum {
			MIN = 0,
			MAX = 2
		};
	};


	/**
	 * Policy
	 */
	template <
		typename ProblemType,
		typename ParamList,
		typename BaseKernelPolicy =	scan::KernelPolicy <
			ProblemType,
			TUNE_ARCH,
			false,														// CHECK_ALIGNMENT
			1,															// MIN_CTA_OCCUPANCY,
			util::Access<ParamList, LOG_THREADS>::VALUE, 				// LOG_THREADS,
			util::Access<ParamList, LOG_LOAD_VEC_SIZE>::VALUE,			// LOG_LOAD_VEC_SIZE,
			util::Access<ParamList, LOG_LOADS_PER_TILE>::VALUE,			// LOG_LOADS_PER_TILE,
			CUB_LOG_WARP_THREADS(TUNE_ARCH),							// LOG_RAKING_THREADS,
			util::io::ld::NONE,											// LOAD_MODIFIER,
			util::io::st::NONE,											// STORE_MODIFIER,
			(util::Access<ParamList, LOG_THREADS>::VALUE +
				util::Access<ParamList, LOG_LOAD_VEC_SIZE>::VALUE +
				util::Access<ParamList, LOG_LOADS_PER_TILE>::VALUE) > >	// LOG_SCHEDULE_GRANULARITY
	struct KernelPolicy : BaseKernelPolicy
	{
		typedef typename ProblemType::T T;
		typedef typename ProblemType::SizeT SizeT;
		typedef typename ProblemType::ReductionOp ReductionOp;
		typedef typename ProblemType::IdentityOp IdentityOp;

		typedef void (*KernelPtr)(T*, T*, SizeT, ReductionOp, IdentityOp);

		// Check if this configuration is worth compiling
		enum {
			REG_MULTIPLIER = (sizeof(T) + 4 - 1) / 4,
			REGS_ESTIMATE = (REG_MULTIPLIER * KernelPolicy::TILE_ELEMENTS_PER_THREAD) + 2,
			EST_REGS_OCCUPANCY = CUB_SM_REGISTERS(TUNE_ARCH) / (REGS_ESTIMATE * KernelPolicy::THREADS),

			// ptxas dies on this special case
			INVALID_SPECIAL =
				(TUNE_ARCH < 200) &&
				(sizeof(T) > 4) &&
				(BaseKernelPolicy::LOG_TILE_ELEMENTS > 9),

			VALID_COMPILE =
				((BaseKernelPolicy::VALID > 0) &&
				((TUNE_ARCH >= 200) || (BaseKernelPolicy::LOAD_MODIFIER == util::io::ld::NONE)) &&
				((TUNE_ARCH >= 200) || (BaseKernelPolicy::STORE_MODIFIER == util::io::st::NONE)) &&
				(BaseKernelPolicy::LOG_THREADS <= CUB_LOG_CTA_THREADS(TUNE_ARCH)) &&
				(EST_REGS_OCCUPANCY > 0) &&
				(INVALID_SPECIAL == 0)),
		};

		static std::string TypeString()
		{
			char buffer[4096];
			sprintf(buffer, "%d, %d, %d",
				KernelPolicy::LOG_THREADS,
				KernelPolicy::LOG_LOAD_VEC_SIZE,
				KernelPolicy::LOG_LOADS_PER_TILE);
			return buffer;
		}

		template <int VALID, int DUMMY = 0>
		struct GenKernel
		{
			static KernelPtr Kernel() {
				return scan::spine::Kernel<KernelPolicy>;
			}
		};

		template <int DUMMY>
		struct GenKernel<0, DUMMY>
		{
			static KernelPtr Kernel() {
				return NULL;
			}
		};

		static KernelPtr Kernel() {
			return GenKernel<VALID_COMPILE>::Kernel();
		}
	};

};


/******************************************************************************
 * Downsweep Tuning Parameter Enumerations and Ranges
 ******************************************************************************/

struct DownsweepTuningRanges
{
	/**
	 * Tuning params
	 */
	enum Param
	{
		BEGIN,
			LOG_THREADS,
			LOG_LOAD_VEC_SIZE,
			LOG_LOADS_PER_TILE,
			LOG_SCHEDULE_GRANULARITY,
		END,
	};


	/**
	 * Ranges for the tuning params
	 */
	template <typename ParamList, int PARAM> struct Ranges;

	// LOG_THREADS
	template <typename ParamList>
	struct Ranges<ParamList, LOG_THREADS> {
		enum {
			MIN = 5,	// 32
			MAX = 10	// 1024
		};
	};

	// LOG_LOAD_VEC_SIZE
	template <typename ParamList>
	struct Ranges<ParamList, LOG_LOAD_VEC_SIZE> {
		enum {
			MIN = 0,
			MAX = 2
		};
	};

	// LOG_LOADS_PER_TILE
	template <typename ParamList>
	struct Ranges<ParamList, LOG_LOADS_PER_TILE> {
		enum {
			MIN = 0,
			MAX = 2
		};
	};

	// LOG_SCHEDULE_GRANULARITY
	template <typename ParamList>
	struct Ranges<ParamList, LOG_SCHEDULE_GRANULARITY> {
		enum {
			MIN = util::Access<ParamList, LOG_THREADS>::VALUE +
				util::Access<ParamList, LOG_LOAD_VEC_SIZE>::VALUE +
				util::Access<ParamList, LOG_LOADS_PER_TILE>::VALUE,

			MAX = Ranges<ParamList, LOG_THREADS>::MAX +
				Ranges<ParamList, LOG_LOAD_VEC_SIZE>::MAX +
				Ranges<ParamList, LOG_LOADS_PER_TILE>::MAX
		};
	};


	/**
	 * Policy
	 */
	template <
		typename ProblemType,
		typename ParamList,
		typename BaseKernelPolicy = scan::KernelPolicy <
			ProblemType,
			TUNE_ARCH,
			true,														// CHECK_ALIGNMENT
			0,															// MIN_CTA_OCCUPANCY,
			util::Access<ParamList, LOG_THREADS>::VALUE, 				// LOG_THREADS,
			util::Access<ParamList, LOG_LOAD_VEC_SIZE>::VALUE,			// LOG_LOAD_VEC_SIZE,
			util::Access<ParamList, LOG_LOADS_PER_TILE>::VALUE,			// LOG_LOADS_PER_TILE,
			CUB_LOG_WARP_THREADS(TUNE_ARCH),							// LOG_RAKING_THREADS,
			util::io::ld::NONE,											// LOAD_MODIFIER,
			util::io::st::NONE,											// STORE_MODIFIER,
			util::Access<ParamList, LOG_SCHEDULE_GRANULARITY>::VALUE> >	// LOG_SCHEDULE_GRANULARITY
	struct KernelPolicy : BaseKernelPolicy
	{
		typedef typename ProblemType::T T;
		typedef typename ProblemType::SizeT SizeT;
		typedef typename ProblemType::ReductionOp ReductionOp;
		typedef typename ProblemType::IdentityOp IdentityOp;

		typedef void (*KernelPtr)(T*, T*, T*, ReductionOp, IdentityOp, util::CtaWorkDistribution<SizeT>);

		// Check if this configuration is worth compiling
		enum {
			REG_MULTIPLIER = (sizeof(T) + 4 - 1) / 4,
			REGS_ESTIMATE = (REG_MULTIPLIER * KernelPolicy::TILE_ELEMENTS_PER_THREAD) + 2,
			EST_REGS_OCCUPANCY = CUB_SM_REGISTERS(TUNE_ARCH) / (REGS_ESTIMATE * KernelPolicy::THREADS),

			// ptxas dies on this special case
			INVALID_SPECIAL =
				(TUNE_ARCH < 200) &&
				(sizeof(T) > 4) &&
				(BaseKernelPolicy::LOG_TILE_ELEMENTS > 9),

			VALID_COMPILE =
				((BaseKernelPolicy::VALID > 0) &&
				((TUNE_ARCH >= 200) || (BaseKernelPolicy::LOAD_MODIFIER == util::io::ld::NONE)) &&
				((TUNE_ARCH >= 200) || (BaseKernelPolicy::STORE_MODIFIER == util::io::st::NONE)) &&
				(BaseKernelPolicy::LOG_THREADS <= CUB_LOG_CTA_THREADS(TUNE_ARCH)) &&
				(EST_REGS_OCCUPANCY > 0) &&
				(INVALID_SPECIAL == 0)),
		};

		static std::string TypeString()
		{
			char buffer[32];
			sprintf(buffer, "%d, %d, %d",
				KernelPolicy::LOG_THREADS,
				KernelPolicy::LOG_LOAD_VEC_SIZE,
				KernelPolicy::LOG_LOADS_PER_TILE);
			return buffer;
		}

		template <int VALID, int DUMMY = 0>
		struct GenKernel
		{
			static KernelPtr Kernel() {
				return scan::downsweep::Kernel<KernelPolicy>;
			}
		};

		template <int DUMMY>
		struct GenKernel<0, DUMMY>
		{
			static KernelPtr Kernel() {
				return NULL;
			}
		};

		static KernelPtr Kernel() {
			return GenKernel<VALID_COMPILE>::Kernel();
		}
	};
};


/******************************************************************************
 * Pass Tuning Parameter Enumerations and Ranges
 ******************************************************************************/

struct PassTuningRanges
{
	enum Param
	{
		BEGIN,
			OVERSUBSCRIBED_GRID_SIZE,
		END,

		// Parameters below here are currently not part of the tuning sweep
		LOAD_MODIFIER,
		STORE_MODIFIER,
		UNIFORM_SMEM_ALLOCATION,
		UNIFORM_GRID_SIZE
	};


	/**
	 * Ranges for the tuning params
	 */
	template <typename ParamList, int PARAM> struct Ranges;

	// LOAD_MODIFIER
	template <typename ParamList>
	struct Ranges<ParamList, LOAD_MODIFIER> {
		enum {
			MIN = util::io::ld::NONE,
			MAX = util::io::ld::LIMIT - 1,
		};
	};

	// STORE_MODIFIER
	template <typename ParamList>
	struct Ranges<ParamList, STORE_MODIFIER> {
		enum {
			MIN = util::io::st::NONE,
			MAX = util::io::st::LIMIT - 1,
		};
	};

	// UNIFORM_SMEM_ALLOCATION
	template <typename ParamList>
	struct Ranges<ParamList, UNIFORM_SMEM_ALLOCATION> {
		enum {
			MIN = 0,
			MAX = 1
		};
	};

	// UNIFORM_GRID_SIZE
	template <typename ParamList>
	struct Ranges<ParamList, UNIFORM_GRID_SIZE> {
		enum {
			MIN = 0,
			MAX = 1
		};
	};

	// OVERSUBSCRIBED_GRID_SIZE
	template <typename ParamList>
	struct Ranges<ParamList, OVERSUBSCRIBED_GRID_SIZE> {
		enum {
			MIN = 0,
			MAX = 1
		};
	};


	// Return pass details configuration from param tuple
	template <typename ParamList>
	static PassDetails Details()
	{
		return PassDetails(
			0,																// UNIFORM_SMEM_ALLOCATION
			0,																// UNIFORM_GRID_SIZE
			util::Access<ParamList, OVERSUBSCRIBED_GRID_SIZE>::VALUE);
	}
};


/******************************************************************************
 * Generators
 ******************************************************************************/



/**
 * Kernel-policy generator (callback)
 */
template <
	typename ProblemType,
	typename TuningRanges,
	typename ConfigMap>
struct KernelGen
{
	typedef typename ConfigMap::mapped_type 	GrainMap;				// map (scheduling-granularity -> LaunchDetails)
	typedef typename GrainMap::value_type 		GrainLaunchDetails;		// tuple (scheduling-granularity, LaunchDetails)

	typedef typename ConfigMap::value_type 		ConfigMapPair;			// tuple (string, GrainMap)
	typedef typename GrainMap::mapped_type 		LaunchDetails;			// tuple (KernelDetails, kernel function ptr)

	// Constructed map of kernel tuning configurations
	ConfigMap *config_map;

	// Constructor
	KernelGen(ConfigMap *config_map) : config_map(config_map) {}

	// Interface
	void Generate()
	{
		util::ParamListSweep<
			TuningRanges::BEGIN + 1,
			TuningRanges::END,
			TuningRanges::template Ranges>::template Invoke<util::EmptyTuple>(*this);
	}

	// Callback
	template <typename ParamList>
	void Invoke()
	{
		typedef typename TuningRanges::template KernelPolicy<
			ProblemType,
			ParamList> KernelPolicy;

		// Type string for this config family
		std::string typestring = KernelPolicy::TypeString();

		// Create pairing between kernel-details and kernel-pointer
		LaunchDetails launch_details(
			KernelDetails(
				KernelPolicy::THREADS,
				KernelPolicy::TILE_ELEMENTS),
			KernelPolicy::Kernel());

		// Create pairing between granularity and launch-details
		GrainLaunchDetails grain_launch_details(
			KernelPolicy::LOG_SCHEDULE_GRANULARITY,
			launch_details);

		// Check to see if we've started a grain list
		if (config_map->find(typestring) == config_map->end()) {

			// Not found.  Insert grain pair into new grain map, insert grain map into config map
			GrainMap grain_map;
			grain_map.insert(grain_launch_details);

			config_map->insert(ConfigMapPair(typestring, grain_map));

		} else {

			// Add this scheduling granularity to the config list
			config_map->find(typestring)->second.insert(grain_launch_details);
		}
	}
};


/**
 * Pass policy-generator (callback)
 */
template <
	typename TuningRanges,
	typename ConfigList>
struct PassGen
{
	// Constructed map of kernel tuning configurations
	ConfigList *config_list;

	// Constructor
	PassGen(ConfigList *config_list) : config_list(config_list) {}

	// Interface
	void Generate()
	{
		util::ParamListSweep<
			TuningRanges::BEGIN + 1,
			TuningRanges::END,
			TuningRanges::template Ranges>::template Invoke<util::EmptyTuple>(*this);
	}

	// Callback
	template <typename ParamList>
	void Invoke()
	{
		PassDetails pass_details = TuningRanges::template Details<ParamList>();

		config_list->push_back(pass_details);
	}
};



template <typename ProblemType>
struct Enactor : public util::EnactorBase
{
	typedef typename ProblemType::T T;
	typedef typename ProblemType::SizeT SizeT;
	typedef typename ProblemType::ReductionOp ReductionOp;
	typedef typename ProblemType::IdentityOp IdentityOp;

	// Spine problem type
	typedef scan::ProblemType<
		typename ProblemType::T,
		typename ProblemType::SizeT,
		typename ProblemType::ReductionOp,
		typename ProblemType::IdentityOp,
		true,								// EXCLUSIVE
		ProblemType::COMMUTATIVE>
			SpineProblemType;

	// Kernel pointer types
	typedef void (*UpsweepKernelPtr)(T*, T*, ReductionOp, IdentityOp, util::CtaWorkDistribution<SizeT>);
	typedef void (*SpineKernelPtr)(T*, T*, SizeT, ReductionOp, IdentityOp);
	typedef void (*DownsweepKernelPtr)(T*, T*, T*, ReductionOp, IdentityOp, util::CtaWorkDistribution<SizeT>);

	typedef std::pair<KernelDetails, UpsweepKernelPtr> 		UpsweepLaunchDetails;
	typedef std::pair<KernelDetails, SpineKernelPtr> 		SpineLaunchDetails;
	typedef std::pair<KernelDetails, DownsweepKernelPtr> 	DownsweepLaunchDetails;

	// Config grain-map types (LOG_GRANULARITY -> kernel pointer)
	typedef std::map<int, UpsweepLaunchDetails> 		UpsweepGrainMap;
	typedef std::map<int, SpineLaunchDetails> 			SpineGrainMap;
	typedef std::map<int, DownsweepLaunchDetails>		DownsweepGrainMap;

	// Config map types (tune-string -> grain map)
	typedef std::map<std::string, UpsweepGrainMap>		UpsweepMap;
	typedef std::map<std::string, SpineGrainMap> 		SpineMap;
	typedef std::map<std::string, DownsweepGrainMap>	DownsweepMap;

	// Pass config list
	typedef std::vector<PassDetails> 					PassConfigList;

	// Configuration maps
	UpsweepMap 		upsweep_configs;
	SpineMap 		spine_configs;
	DownsweepMap 	downsweep_configs;
	PassConfigList	pass_configs;

	// Temporary device storage needed for reducing partials produced
	// by separate CTAs
	util::Spine spine;

	T *d_dest;
	T *d_src;
	T *h_data;
	T *h_reference;
	SizeT num_elements;
	ReductionOp reduction_op;
	IdentityOp identity_op;

	/**
	 * Constructor
	 */
	Enactor(
		ReductionOp reduction_op,
		IdentityOp identity_op) :
			d_dest(NULL),
			d_src(NULL),
			h_data(NULL),
			h_reference(NULL),
			reduction_op(reduction_op),
			identity_op(identity_op)
	{
		// Pre-allocate our spine
		if (spine.Setup<long long>(SmCount() * 8 * 8)) exit(1);

		// Generates all kernel config maps
		KernelGen<ProblemType, UpsweepTuningRanges, UpsweepMap> 		upsweep_gen(&upsweep_configs);
		KernelGen<SpineProblemType, SpineTuningRanges, SpineMap> 		spine_gen(&spine_configs);
		KernelGen<ProblemType, DownsweepTuningRanges, DownsweepMap> 	downsweep_gen(&downsweep_configs);
		PassGen<PassTuningRanges, PassConfigList>						pass_gen(&pass_configs);

		upsweep_gen.Generate();
		spine_gen.Generate();
		downsweep_gen.Generate();
		pass_gen.Generate();
	}


	/**
	 *
	 */
	hipError_t RunSample(
		int log_schedule_granularity,
		UpsweepLaunchDetails upsweep_details,
		SpineLaunchDetails spine_details,
		DownsweepLaunchDetails downsweep_details)
	{
		const bool OVERSUBSCRIBED_GRID_SIZE = true;
		const bool UNIFORM_SMEM_ALLOCATION = false;
		const bool UNIFORM_GRID_SIZE = false;

		hipError_t retval = hipSuccess;
		do {

			// Max CTA occupancy for the actual target device
			int max_cta_occupancy;
			if (retval = MaxCtaOccupancy(
				max_cta_occupancy,
				upsweep_details.second,
				upsweep_details.first.threads,
				downsweep_details.second,
				downsweep_details.first.threads)) break;

			// Compute sweep grid size
			int sweep_grid_size = GridSize(
				OVERSUBSCRIBED_GRID_SIZE,
				1 << log_schedule_granularity,
				max_cta_occupancy,
				num_elements,
				g_max_ctas);

			// Use single-CTA kernel instead of multi-pass if problem is small enough
			if (num_elements <= spine_details.first.tile_elements * 3) {
				sweep_grid_size = 1;
			}

			// Compute spine elements: one element per CTA, rounded
			// up to nearest spine tile size
			int spine_elements = ((sweep_grid_size + spine_details.first.tile_elements - 1) / spine_details.first.tile_elements) * spine_details.first.tile_elements;

			// Obtain a CTA work distribution
			util::CtaWorkDistribution<SizeT> work;
			work.Init(num_elements, sweep_grid_size, log_schedule_granularity);

			if (ENACTOR_DEBUG) {
				printf("Work: ");
				work.Print();
			}

			if (work.grid_size == 1) {

				if (ENACTOR_DEBUG) {
					printf("Sweep<<<%d,%d,%d>>>\n", 1, spine_details.first.threads, 0);
				}

				// Single-CTA, single-grid operation
				spine_details.second<<<1, spine_details.first.threads, 0>>>(
					d_src,
					d_dest,
					work.num_elements,
					reduction_op,
					identity_op);

				if (ENACTOR_DEBUG && (retval = util::B40CPerror(hipDeviceSynchronize(), "Enactor SingleKernel failed ", __FILE__, __LINE__, ENACTOR_DEBUG))) break;

			} else {

				// Make sure our spine is big enough
				if (retval = spine.Setup<T>(spine_elements)) break;

				int dynamic_smem[3] = 	{0, 0, 0};
				int grid_size[3] = 		{work.grid_size, 1, work.grid_size};

				// Tuning option: make sure all kernels have the same overall smem allocation
				if (UNIFORM_SMEM_ALLOCATION) if (retval = PadUniformSmem(
					dynamic_smem,
					upsweep_details.second,
					spine_details.second,
					downsweep_details.second)) break;

				// Tuning option: make sure that all kernels launch the same number of CTAs)
				if (UNIFORM_GRID_SIZE) grid_size[1] = grid_size[0];

				if (ENACTOR_DEBUG) {
					printf("Upsweep<<<%d,%d,%d>>> Spine<<<%d,%d,%d>>> Downsweep<<<%d,%d,%d>>>\n",
						grid_size[0], upsweep_details.first.threads, dynamic_smem[0],
						grid_size[1], spine_details.first.threads, dynamic_smem[1],
						grid_size[2], downsweep_details.first.threads, dynamic_smem[2]);
				}

				// Upsweep into spine
				upsweep_details.second<<<grid_size[0], upsweep_details.first.threads, dynamic_smem[0]>>>(
					d_src,
					(T*) spine(),
					reduction_op,
					identity_op,
					work);

				if (ENACTOR_DEBUG && (retval = util::B40CPerror(hipDeviceSynchronize(), "Enactor UpsweepKernel failed ", __FILE__, __LINE__, ENACTOR_DEBUG))) break;

				// Spine scan
				spine_details.second<<<grid_size[1], spine_details.first.threads, dynamic_smem[1]>>>(
					(T*) spine(),
					(T*) spine(),
					spine_elements,
					reduction_op,
					identity_op);

				if (ENACTOR_DEBUG && (retval = util::B40CPerror(hipDeviceSynchronize(), "Enactor SpineKernel failed ", __FILE__, __LINE__, ENACTOR_DEBUG))) break;

				// Downsweep from spine
				downsweep_details.second<<<grid_size[2], downsweep_details.first.threads, dynamic_smem[2]>>>(
					d_src,
					d_dest,
					(T*) spine(),
					reduction_op,
					identity_op,
					work);

				if (ENACTOR_DEBUG && (retval = util::B40CPerror(hipDeviceSynchronize(), "Enactor DownsweepKernel failed ", __FILE__, __LINE__, ENACTOR_DEBUG))) break;
			}

		} while (0);

		return retval;
	}


	/**
	 *
	 */
	void TimeSample(
		int log_schedule_granularity,
		UpsweepLaunchDetails upsweep_details,
		SpineLaunchDetails spine_details,
		DownsweepLaunchDetails downsweep_details)
	{
		// Check if valid for dispatch
		if (!upsweep_details.second || !spine_details.second || !downsweep_details.second) {
			return;
		}

		// Invoke kernels (warmup)
		ENACTOR_DEBUG = g_verbose;
		if (RunSample(
			log_schedule_granularity,
			upsweep_details,
			spine_details,
			downsweep_details))
		{
			exit(1);
		}
		ENACTOR_DEBUG = false;

		// Perform the timed number of iterations
		GpuTimer timer;
		double elapsed = 0;
		for (int i = 0; i < g_iterations; i++) {

			// Start cuda timing record
			timer.Start();

			// Invoke kernels
			if (RunSample(
				log_schedule_granularity,
				upsweep_details,
				spine_details,
				downsweep_details))
			{
				exit(1);
			}

			// End cuda timing record
			timer.Stop();
			elapsed += timer.ElapsedMillis();

			// Flushes any stdio from the GPU
			if (util::B40CPerror(hipDeviceSynchronize(), "TimedCopy hipDeviceSynchronize failed: ", __FILE__, __LINE__)) {
				exit(1);
			}
		}

		// Display timing information
		double avg_runtime = elapsed / g_iterations;
		double throughput =  0.0;
		if (avg_runtime > 0.0) throughput = ((double) num_elements) / avg_runtime / 1000.0 / 1000.0;
		printf(", %f, %f, %f, ",
			avg_runtime, throughput, throughput * sizeof(T) * 3);
		fflush(stdout);

		if (g_verify) {
			// Copy out data
			if (util::B40CPerror(hipMemcpy(
				h_data,
				d_dest,
				sizeof(T) * num_elements,
				hipMemcpyDeviceToHost),
					"TimedScan hipMemcpy d_dest failed: ", __FILE__, __LINE__)) exit(1);

			// Verify solution
			CompareResults(
				h_data,
				h_reference,
				num_elements,
				true);
		}
	}


	/**
	 * Iterates over configuration space
	 */
	void IterateConfigSpace()
	{
		int config_id = 0;

		// Iterate upsweep configs
		for (typename UpsweepMap::iterator upsweep_config_itr = upsweep_configs.begin();
			upsweep_config_itr != upsweep_configs.end();
			upsweep_config_itr++)
		{
			std::string upsweep_string = upsweep_config_itr->first;

			// Iterate downsweep configs
			for (typename DownsweepMap::iterator downsweep_config_itr = downsweep_configs.begin();
				downsweep_config_itr != downsweep_configs.end();
				downsweep_config_itr++)
			{
				std::string downsweep_string = downsweep_config_itr->first;

				typename UpsweepGrainMap::iterator upsweep_grain_itr = upsweep_config_itr->second.begin();
				typename DownsweepGrainMap::iterator downsweep_grain_itr = downsweep_config_itr->second.begin();

				while (true) {

					if ((upsweep_grain_itr == upsweep_config_itr->second.end()) ||
						(downsweep_grain_itr == downsweep_config_itr->second.end()))
					{
						// Could not match grain

						printf("Could not match upsweep(%s) with downsweep(%s)\n",
							upsweep_string.c_str(),
							downsweep_string.c_str());

						exit(1);

					}
					else if (upsweep_grain_itr->first == downsweep_grain_itr->first)
					{
						// Matched grain
						std::string downsweep_string = downsweep_config_itr->first;

						// Iterate spine configs
						for (typename SpineMap::iterator spine_config_itr = spine_configs.begin();
							spine_config_itr != spine_configs.end();
							spine_config_itr++)
						{
							std::string spine_string = spine_config_itr->first;

							// Iterate pass configs
							for (typename PassConfigList::iterator pass_config_itr = pass_configs.begin();
								pass_config_itr != pass_configs.end();
								pass_config_itr++)
							{
								std::string pass_string = pass_config_itr->TypeString();

								printf("%d, %s, %d, %s, %s, %s",
									config_id,
									pass_string.c_str(),
									upsweep_grain_itr->first,		// schedule grain
									upsweep_string.c_str(),
									spine_string.c_str(),
									downsweep_string.c_str());
								config_id++;

								TimeSample(
									upsweep_grain_itr->first,
									upsweep_grain_itr->second,
									spine_config_itr->second.begin()->second,
									downsweep_grain_itr->second);

								printf("\n");
								fflush(stdout);
							}
						}

						break;

					} else if (upsweep_grain_itr->first < downsweep_grain_itr->first) {
						upsweep_grain_itr++;
					} else {
						downsweep_grain_itr++;
					}
				}
			}
		}
	}


	/**
	 * Creates an example problem and then dispatches the iterations
	 * to the GPU for the given number of iterations, displaying runtime information.
	 */
	void Test(SizeT num_elements)
	{
		this->num_elements = num_elements;

		if (util::B40CPerror(hipMalloc((void**) &d_src, sizeof(T) * num_elements),
			"TimedScan hipMalloc d_src failed: ", __FILE__, __LINE__)) exit(1);

		if (util::B40CPerror(hipMalloc((void**) &d_dest, sizeof(T) * num_elements),
			"TimedScan hipMalloc d_dest failed: ", __FILE__, __LINE__)) exit(1);

		if ((h_data = (T*) malloc(sizeof(T) * num_elements)) == NULL) {
			fprintf(stderr, "Host malloc of problem data failed\n");
			exit(1);
		}
		if ((h_reference = (T*) malloc(sizeof(T) * num_elements)) == NULL) {
			fprintf(stderr, "Host malloc of problem data failed\n");
			exit(1);
		}

		h_reference[0] = identity_op();

		for (SizeT i = 0; i < num_elements; ++i) {
	//		util::RandomBits<T>(h_data[i], 0);
			h_data[i] = i;

			h_reference[i] = (i == 0) ?
				identity_op() :
				reduction_op(h_reference[i - 1], h_data[i - 1]);
		}

		// Move a fresh copy of the problem into device storage
		if (util::B40CPerror(hipMemcpy(d_src, h_data, sizeof(T) * num_elements, hipMemcpyHostToDevice),
			"TimedScan hipMemcpy d_src failed: ", __FILE__, __LINE__)) exit(1);

		// Iterate configuration space
		IterateConfigSpace();

		// Free allocated memory
		if (d_src) hipFree(d_src);
		if (d_dest) hipFree(d_dest);

		// Free our allocated host memory
		if (h_data) free(h_data);
		if (h_reference) free(h_reference);
	}

};



/******************************************************************************
 * Test
 ******************************************************************************/



/**
 * Creates an example problem and then dispatches the iterations
 * to the GPU for the given number of iterations, displaying runtime information.
 */
template<
	typename T,
	typename SizeT,
	typename ReductionOp,
	typename IdentityOp>
void Test(
	SizeT num_elements,
	ReductionOp reduction_op,
	IdentityOp identity_op)
{
	// Establish the problem types
	typedef scan::ProblemType<
		T,
		SizeT,
		ReductionOp,
		IdentityOp,
		true,								// EXCLUSIVE,
		true>								// COMMUTATIVE
			ProblemType;

	// Create enactor
	Enactor<ProblemType> enactor(
		reduction_op,
		identity_op);

	// Run test
	enactor.Test(num_elements);
}


/******************************************************************************
 * Main
 ******************************************************************************/

int main(int argc, char** argv)
{

	CommandLineArgs args(argc, argv);
	DeviceInit(args);

	// Seed random number generator
	srand(0);				// presently deterministic

	// Use 32-bit integer for array indexing
	typedef int SizeT;
	SizeT num_elements = 1024;

	// Parse command line arguments
    if (args.CheckCmdLineFlag("help")) {
		Usage();
		return 0;
	}
    args.GetCmdLineArgument("i", g_iterations);
    args.GetCmdLineArgument("n", num_elements);
    args.GetCmdLineArgument("max-ctas", g_max_ctas);
    g_verify = args.CheckCmdLineFlag("verify");
	g_verbose = args.CheckCmdLineFlag("v");

	util::CudaProperties cuda_props;

	printf("Test Scan: %d iterations, %lu elements", g_iterations, (unsigned long) num_elements);
	printf("\nCodeGen: \t[device_sm_version: %d, kernel_ptx_version: %d]\n\n",
		cuda_props.device_sm_version, cuda_props.kernel_ptx_version);

	printf(""
		"TuneID, "

		"UNIFORM_SMEM_ALLOCATION, "
		"UNIFORM_GRID_SIZE, "
		"OVERSUBSCRIBED_GRID_SIZE, "

		"SCHEDULING_GRANULARITY, "

		"UPSWEEP_LOG_THREADS, "
		"UPSWEEP_LOG_LOAD_VEC_SIZE, "
		"UPSWEEP_LOG_LOADS_PER_TILE, "

		"SPINE_LOG_THREADS, "
		"SPINE_LOG_LOAD_VEC_SIZE, "
		"SPINE_LOG_LOADS_PER_TILE, "

		"DOWNSWEEP_LOG_THREADS, "
		"DOWNSWEEP_LOG_LOAD_VEC_SIZE, "
		"DOWNSWEEP_LOG_LOADS_PER_TILE, "

		"elapsed time (ms), "
		"throughput (10^9 items/s), "
		"bandwidth (10^9 B/s)");
	if (g_verify) printf(", Correctness");
	printf("\n");


	// Execute test(s)
#if (TUNE_SIZE == 0) || (TUNE_SIZE == 1)
	{
		typedef unsigned char T;
		Sum<T> binary_op;
		Test<T>(num_elements * 4, binary_op, binary_op);
	}
#endif
#if (TUNE_SIZE == 0) || (TUNE_SIZE == 2)
	{
		typedef unsigned short T;
		Sum<T> binary_op;
		Test<T>(num_elements * 2, binary_op, binary_op);
	}
#endif
#if (TUNE_SIZE == 0) || (TUNE_SIZE == 4)
	{
		typedef unsigned int T;
		Sum<T> binary_op;
		Test<T>(num_elements, binary_op, binary_op);
	}
#endif
#if (TUNE_SIZE == 0) || (TUNE_SIZE == 8)
	{
		typedef unsigned long long T;
		Sum<T> binary_op;
		Test<T>(num_elements / 2, binary_op, binary_op);
	}
#endif

	return 0;
}



