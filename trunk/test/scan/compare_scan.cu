#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2010-2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2013, NVIDIA CORPORATION.  All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/


/******************************************************************************
 * Simple test driver program for scan.
 ******************************************************************************/

#include <stdio.h> 

// Test utils
#include "b40c_test_util.h"
#include "test_scan.h"

#include <thrust/device_vector.h>
#include <thrust/scan.h>

using namespace b40c;


/******************************************************************************
 * Defines, constants, globals 
 ******************************************************************************/

bool 	g_verbose 						= false;
int 	g_max_ctas 						= 0;
int 	g_iterations  					= 1;
bool 	g_inclusive						= false;


/******************************************************************************
 * Utility Routines
 ******************************************************************************/

/**
 * Displays the commandline usage for this tool
 */
void Usage() 
{
	printf("\ntest_scan [--device=<device index>] [--v] [--i=<num-iterations>] "
			"[--max-ctas=<max-thread-blocks>] [--n=<num-elements>] [--inclusive]\n");
	printf("\n");
	printf("\t--v\tDisplays copied results to the console.\n");
	printf("\n");
	printf("\t--i\tPerforms the scan operation <num-iterations> times\n");
	printf("\t\t\ton the device. Re-copies original input each time. Default = 1\n");
	printf("\n");
	printf("\t--n\tThe number of elements to comprise the sample problem\n");
	printf("\t\t\tDefault = 512\n");
	printf("\n");
}





/**
 * Timed scan.  Uses the GPU to copy the specified vector of elements for the given
 * number of iterations, displaying runtime information.
 */
template <
	bool EXCLUSIVE,
	typename T,
	typename SizeT,
	typename ReductionOp,
	typename IdentityOp>
double TimedThrustScan(
	T *h_data,
	T *h_reference,
	SizeT num_elements,
	ReductionOp scan_op,
	IdentityOp identity_op)
{
	using namespace b40c;

	// Allocate device storage  
	T *d_src, *d_dest;
	if (util::B40CPerror(hipMalloc((void**) &d_src, sizeof(T) * num_elements),
		"TimedScan hipMalloc d_src failed: ", __FILE__, __LINE__)) exit(1);
	if (util::B40CPerror(hipMalloc((void**) &d_dest, sizeof(T) * num_elements),
		"TimedScan hipMalloc d_dest failed: ", __FILE__, __LINE__)) exit(1);

	// Move a fresh copy of the problem into device storage
	if (util::B40CPerror(hipMemcpy(d_src, h_data, sizeof(T) * num_elements, hipMemcpyHostToDevice),
		"TimedScan hipMemcpy d_src failed: ", __FILE__, __LINE__)) exit(1);
	
	// Marker kernel in profiling stream
	util::FlushKernel<void><<<1,1>>>();

	// Perform a single iteration to allocate any memory if needed, prime code caches, etc.
	thrust::device_ptr<T> dev_src(d_src);
	thrust::device_ptr<T> dev_dest(d_dest);
	if (EXCLUSIVE) {
		thrust::exclusive_scan(dev_src, dev_src + num_elements, dev_dest);
	} else {
		thrust::inclusive_scan(dev_src, dev_src + num_elements, dev_dest);
	}
	
	// Perform the timed number of iterations
	GpuTimer timer;

	double elapsed = 0;
	for (int i = 0; i < g_iterations; i++) {

		// Marker kernel in profiling stream
		util::FlushKernel<void><<<1,1>>>();

		// Start timing record
		timer.Start();

		if (EXCLUSIVE) {
			thrust::exclusive_scan(dev_src, dev_src + num_elements, dev_dest);
		} else {
			thrust::inclusive_scan(dev_src, dev_src + num_elements, dev_dest);
		}
		
		// End timing record
		timer.Stop();
		elapsed += (double) timer.ElapsedMillis();
	}

	// Display timing information
	double avg_runtime = elapsed / g_iterations;
	double throughput = ((double) num_elements) / avg_runtime / 1000.0 / 1000.0;
	printf("\nThrust Scan: %d iterations, %lu elements, ", g_iterations, (unsigned long) num_elements);
    printf("%f GPU ms, %f x10^9 elts/sec",
		avg_runtime, throughput);

    // Copy out data
	T *h_dest = (T*) malloc(num_elements * sizeof(T));
    if (util::B40CPerror(hipMemcpy(h_dest, d_dest, sizeof(T) * num_elements, hipMemcpyDeviceToHost),
		"TimedScan hipMemcpy d_dest failed: ", __FILE__, __LINE__)) exit(1);

    // Free allocated memory
    if (d_src) hipFree(d_src);
    if (d_dest) hipFree(d_dest);

	// Flushes any stdio from the GPU
	hipDeviceSynchronize();

	// Display copied data
	if (g_verbose) {
		printf("\n\nData:\n");
		for (int i = 0; i < num_elements; i++) {
			PrintValue<T>(h_dest[i]);
			printf(", ");
		}
		printf("\n\n");
	}

    // Verify solution
	CompareResults(h_dest, h_reference, num_elements, true);
	printf("\n");
	fflush(stdout);

	if (h_dest) free(h_dest);

	return throughput;
}



/**
 * Creates an example scan problem and then dispatches the problem
 * to the GPU for the given number of iterations, displaying runtime information.
 */
template<
	typename T,
	bool EXCLUSIVE,
	typename SizeT,
	typename ReductionOp,
	typename IdentityOp>
void TestScan(
	SizeT num_elements,
	ReductionOp scan_op,
	IdentityOp identity_op)
{
    // Allocate the scan problem on the host and fill the keys with random bytes

	T *h_data 			= (T*) malloc(num_elements * sizeof(T));
	T *h_reference 		= (T*) malloc(num_elements * sizeof(T));

	if ((h_data == NULL) || (h_reference == NULL)){
		fprintf(stderr, "Host malloc of problem data failed\n");
		exit(1);
	}

	for (size_t i = 0; i < num_elements; ++i) {
//		util::RandomBits<T>(h_data[i], 0);
		h_data[i] = i;
		if (EXCLUSIVE)
		{
			h_reference[i] = (i == 0) ?
				identity_op() :
				scan_op(h_reference[i - 1], h_data[i - 1]);
		} else {
			h_reference[i] = (i == 0) ?
				h_data[i] :
				scan_op(h_reference[i - 1], h_data[i]);
		}
	}

	//
    // Run the timing test(s)
	//

	double b40c = TimedScan<EXCLUSIVE, scan::UNKNOWN_SIZE>(
		h_data, h_reference, num_elements, scan_op, identity_op, g_max_ctas, g_verbose, g_iterations);

	double thrust = TimedThrustScan<EXCLUSIVE>(
		h_data, h_reference, num_elements, scan_op, identity_op);

	printf("B40C speedup: %.2f\n", b40c/thrust);
	

	// Free our allocated host memory 
	if (h_data) free(h_data);
    if (h_reference) free(h_reference);
}


/**
 * Creates an example scan problem and then dispatches the problem
 * to the GPU for the given number of iterations, displaying runtime information.
 */
template<
	typename T,
	typename SizeT,
	typename ReductionOp,
	typename IdentityOp>
void TestScanVariety(
	SizeT num_elements,
	ReductionOp scan_op,
	IdentityOp identity_op)
{
	if (g_inclusive) {
		TestScan<T, false>(num_elements, scan_op, identity_op);
	} else {
		TestScan<T, true>(num_elements, scan_op, identity_op);
	}
}


/******************************************************************************
 * Main
 ******************************************************************************/

int main(int argc, char** argv)
{

	CommandLineArgs args(argc, argv);
	DeviceInit(args);

	//srand(time(NULL));	
	srand(0);				// presently deterministic

    //
	// Check command line arguments
    //

	size_t num_elements = 1024;

    if (args.CheckCmdLineFlag("help")) {
		Usage();
		return 0;
	}

    g_inclusive = args.CheckCmdLineFlag("inclusive");
    args.GetCmdLineArgument("i", g_iterations);
    args.GetCmdLineArgument("n", num_elements);
    args.GetCmdLineArgument("max-ctas", g_max_ctas);
	g_verbose = args.CheckCmdLineFlag("v");

	// Execute test(s)
	{
		printf("\n-- UNSIGNED CHAR ----------------------------------------------\n");
		typedef unsigned char T;
		Sum<T> op;
		TestScanVariety<T>(num_elements * 4, op, op);
	}
	{
		printf("\n-- UNSIGNED SHORT ----------------------------------------------\n");
		typedef unsigned short T;
		Sum<T> op;
		TestScanVariety<T>(num_elements * 2, op, op);
	}
	{
		printf("\n-- UNSIGNED INT -----------------------------------------------\n");
		typedef unsigned int T;
		Sum<T> op;
		TestScanVariety<T>(num_elements, op, op);
	}
	{
		printf("\n-- UNSIGNED LONG LONG -----------------------------------------\n");
		typedef unsigned long long T;
		Sum<T> op;
		TestScanVariety<T>(num_elements / 2, op, op);
	}

	return 0;
}



