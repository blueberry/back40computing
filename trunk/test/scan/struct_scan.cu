#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2010-2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2013, NVIDIA CORPORATION.  All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Simple test driver program for scan.
 ******************************************************************************/

#include <stdio.h> 
#include <b40c/scan/enactor.cuh>

// Test utils
#include "b40c_test_util.h"


/******************************************************************************
 * Utility Routines
 ******************************************************************************/

/**
 * Struct for doing addition and max scan simultaneously
 */
struct Foo
{
	int a, b;

	__host__ __device__ __forceinline__ Foo() :
		a(0), b(0) {}

	__host__ __device__ __forceinline__ Foo(int a, int b) :
		a(a), b(b) {}

	__host__ __device__ __forceinline__ bool operator == (const Foo& other) const
	{
		return ((a == other.a) && (b == other.b));
	}

	__host__ __device__ __forceinline__ bool operator != (const Foo& other) const
	{
		return ((a != other.a) || (b != other.b));
	}

	void Print()
	{
		printf("[a: %d, b: %d]", a, b);
	}
};


/**
 * Foo binary scan operator
 */
struct MultiScan
{
	// Associative reduction operator
	__host__ __device__ __forceinline__ Foo operator()(const Foo &x, const Foo &y)
	{
		return Foo(
			x.a + y.a,
			(x.b > y.b) ? x.b : y.b);
	}

	// Identity operator
	__host__ __device__ __forceinline__ Foo operator()()
	{
		return Foo();
	}

	enum {
		NON_COMMUTATIVE = true,
	};
};


/******************************************************************************
 * Main
 ******************************************************************************/

int main(int argc, char** argv)
{
	b40c::CommandLineArgs args(argc, argv);

	// Usage/help
    if (args.CheckCmdLineFlag("help") || args.CheckCmdLineFlag("h")) {
    	printf("\nsimple_scan [--device=<device index>]\n");
    	return 0;
    }

    b40c::DeviceInit(args);
    int num_elements = 564;
    bool verbose = args.CheckCmdLineFlag("v");
    bool exclusive = args.CheckCmdLineFlag("exclusive");
    args.GetCmdLineArgument("n", num_elements);

	// Allocate and initialize host problem data and host reference solution
	Foo *h_src = new Foo[num_elements];
	Foo *h_reference = new Foo[num_elements];
	MultiScan max_op;

	for (size_t i = 0; i < num_elements; ++i) {
		h_src[i] = Foo(i, i);

		if (exclusive) {
			h_reference[i] = (i == 0) ?
				max_op() :									// identity
				max_op(h_reference[i - 1], h_src[i - 1]);
		} else {
			h_reference[i] = (i == 0) ?
				h_src[i] :
				max_op(h_reference[i - 1], h_src[i]);
		}
	}

	
	// Allocate and initialize device data
	Foo *d_src, *d_dest;
	hipMalloc((void**) &d_src, sizeof(Foo) * num_elements);
	hipMalloc((void**) &d_dest, sizeof(Foo) * num_elements);
	hipMemcpy(d_src, h_src, sizeof(Foo) * num_elements, hipMemcpyHostToDevice);


	// Create a scan enactor
	b40c::scan::Enactor scan_enactor;

	// Enact simple exclusive scan using internal tuning heuristics
	if (exclusive) {
		scan_enactor.Scan<true, MultiScan::NON_COMMUTATIVE>(
			d_dest, d_src, num_elements, max_op, max_op);
	} else {
		scan_enactor.Scan<false, MultiScan::NON_COMMUTATIVE>(
			d_dest, d_src, num_elements, max_op, max_op);
	}
	
	printf("Simple scan: "); b40c::CompareDeviceResults(h_reference, d_dest, num_elements, verbose, verbose); printf("\n");

	delete h_src;
	delete h_reference;

	return 0;
}

