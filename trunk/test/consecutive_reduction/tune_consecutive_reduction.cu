#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2010-2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2013, NVIDIA CORPORATION.  All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/


/******************************************************************************
 * Tuning tool for establishing optimal consecutive removal granularity configuration types
 ******************************************************************************/

#include <stdio.h> 

#include <map>

#include <b40c/util/arch_dispatch.cuh>
#include <b40c/util/cuda_properties.cuh>
#include <b40c/util/numeric_traits.cuh>
#include <b40c/util/parameter_generation.cuh>
#include <b40c/util/enactor_base.cuh>
#include <b40c/util/spine.cuh>
#include <b40c/util/multiple_buffering.cuh>

#include <b40c/consecutive_reduction/problem_type.cuh>
#include <b40c/consecutive_reduction/policy.cuh>


// Test utils
#include "b40c_test_util.h"

using namespace b40c;


/******************************************************************************
 * Defines, constants, globals, and utility types
 ******************************************************************************/

#ifndef TUNE_ARCH
	#define TUNE_ARCH (200)
#endif
#ifndef TUNE_SIZE
	#define TUNE_SIZE (4)
#endif

bool 	g_verbose;
int 	g_max_ctas = 0;
int 	g_iterations = 0;
bool 	g_verify;
int 	g_policy_id = 0;


struct KernelDetails
{
	int threads;
	int tile_elements;

	KernelDetails(
		int threads,
		int tile_elements) :
			threads(threads),
			tile_elements(tile_elements) {}
};


/******************************************************************************
 * Test wrappers for binary, associative operations
 ******************************************************************************/

template <typename T>
struct Sum
{
	__host__ __device__ __forceinline__ T operator()(const T &a, const T &b)
	{
		return a + b;
	}
};

template <typename T>
struct Max
{
	__host__ __device__ __forceinline__ T operator()(const T &a, const T &b)
	{
		return (a > b) ? a : b;
	}
};

template <typename T>
struct Equality
{
	// Equality test
	__host__ __device__ __forceinline__ bool operator()(const T &a, const T &b)
	{
		return a == b;
	}
};


/******************************************************************************
 * Utility routines
 ******************************************************************************/

/**
 * Displays the commandline usage for this tool
 */
void Usage()
{
	printf("\ntune_consecutive_reduction [--device=<device index>] [--v] [--i=<num-iterations>] "
			"[--max-ctas=<max-thread-blocks>] [--n=<num-words>] [--verify]\n");
	printf("\n");
	printf("\t--v\tDisplays verbose configuration to the console.\n");
	printf("\n");
	printf("\t--verify\tChecks the result.\n");
	printf("\n");
	printf("\t--i\tPerforms the operation <num-iterations> times\n");
	printf("\t\t\ton the device. Default = 1\n");
	printf("\n");
	printf("\t--n\tThe number of 32-bit words to comprise the sample problem\n");
	printf("\n");
	printf("\t--max-ctas\tThe number of CTAs to launch\n");
	printf("\n");
}


/******************************************************************************
 * Upsweep Tuning Parameter Enumerations and Ranges
 ******************************************************************************/

struct UpsweepTuning
{
	/**
	 * Tuning params
	 */
	enum Param
	{
		BEGIN,
			LOG_THREADS,
			LOG_LOAD_VEC_SIZE,
			LOG_LOADS_PER_TILE,
			LOG_SCHEDULE_GRANULARITY,
		END,
		CONSECUTIVE_SMEM_ASSIST,
	};

	/**
	 * Policy
	 */
	template <
		typename ProblemType,
		typename ParamList,
		typename BaseKernelPolicy =
			consecutive_reduction::upsweep::KernelPolicy <
				ProblemType,
				TUNE_ARCH,
				true,														// CHECK_ALIGNMENT
				0,															// MIN_CTA_OCCUPANCY,
				util::Access<ParamList, LOG_THREADS>::VALUE, 				// LOG_THREADS,
				util::Access<ParamList, LOG_LOAD_VEC_SIZE>::VALUE,			// LOG_LOAD_VEC_SIZE,
				util::Access<ParamList, LOG_LOADS_PER_TILE>::VALUE,			// LOG_LOADS_PER_TILE,
				B40C_LOG_WARP_THREADS(TUNE_ARCH),							// LOG_RAKING_THREADS,
				util::io::ld::NONE,											// READ_MODIFIER,
				util::io::st::NONE,											// WRITE_MODIFIER,
				util::Access<ParamList, LOG_SCHEDULE_GRANULARITY>::VALUE,	// LOG_SCHEDULE_GRANULARITY
				1> >														// CONSECUTIVE_SMEM_ASSIST

	struct KernelPolicy : BaseKernelPolicy
	{
		typedef typename ProblemType::KeyType 			KeyType;
		typedef typename ProblemType::ValueType			ValueType;
		typedef typename ProblemType::SizeT 			SizeT;
		typedef typename ProblemType::ReductionOp 		ReductionOp;
		typedef typename ProblemType::EqualityOp		EqualityOp;
		typedef typename ProblemType::SpineSizeT		SpineSizeT;

		typedef void (*KernelPtr)(KeyType*, ValueType*, ValueType*, SizeT*, ReductionOp, EqualityOp, util::CtaWorkDistribution<SizeT>);

		// Check if this configuration is worth compiling
		enum {
			REG_MULTIPLIER =
				((sizeof(KeyType) + 4 - 1) / 4) + 		// Keys
				((sizeof(ValueType) + 4 - 1) / 4) +		// Values
				((sizeof(SizeT) + 4 - 1) / 4),			// Ranks
			REGS_ESTIMATE = (REG_MULTIPLIER * KernelPolicy::TILE_ELEMENTS_PER_THREAD) + 2,
			EST_REGS_OCCUPANCY = B40C_SM_REGISTERS(TUNE_ARCH) / (REGS_ESTIMATE * KernelPolicy::THREADS),

			VALID_COMPILE =
				((BaseKernelPolicy::VALID > 0) &&
				(REGS_ESTIMATE < 63) &&
				((TUNE_ARCH >= 200) || (BaseKernelPolicy::READ_MODIFIER == util::io::ld::NONE)) &&
				((TUNE_ARCH >= 200) || (BaseKernelPolicy::WRITE_MODIFIER == util::io::st::NONE)) &&
				(BaseKernelPolicy::LOG_THREADS <= B40C_LOG_CTA_THREADS(TUNE_ARCH)) &&
				(EST_REGS_OCCUPANCY > 0)),
		};

		static std::string TypeString()
		{
			char buffer[32];
			sprintf(buffer, "%d, %d, %d",
				KernelPolicy::LOG_THREADS,
				KernelPolicy::LOG_LOAD_VEC_SIZE,
				KernelPolicy::LOG_LOADS_PER_TILE);
			return buffer;
		}

		template <int VALID, int DUMMY = 0>
		struct GenKernel
		{
			static KernelPtr Kernel() {
				return consecutive_reduction::upsweep::Kernel<KernelPolicy>;
			}
		};

		template <int DUMMY>
		struct GenKernel<0, DUMMY>
		{
			static KernelPtr Kernel() {
				return NULL;
			}
		};

		static KernelPtr Kernel() {
			return GenKernel<VALID_COMPILE>::Kernel();
		}
	};


	/**
	 * Ranges for the tuning params
	 */
	template <typename ParamList, int PARAM> struct Ranges;

	// LOG_THREADS
	template <typename ParamList>
	struct Ranges<ParamList, LOG_THREADS> {
		enum {
			MIN = 5,	// 32
			MAX = 10	// 1024
		};
	};

	// LOG_LOAD_VEC_SIZE
	template <typename ParamList>
	struct Ranges<ParamList, LOG_LOAD_VEC_SIZE> {
		enum {
			MIN = 0,
			MAX = 2
		};
	};

	// LOG_LOADS_PER_TILE
	template <typename ParamList>
	struct Ranges<ParamList, LOG_LOADS_PER_TILE> {
		enum {
			MIN = 0,
			MAX = 2
		};
	};

	// CONSECUTIVE_SMEM_ASSIST
	template <typename ParamList>
	struct Ranges<ParamList, CONSECUTIVE_SMEM_ASSIST> {
		enum {
			MIN = 1,
			MAX = 1
		};
	};

	// LOG_SCHEDULE_GRANULARITY
	template <typename ParamList>
	struct Ranges<ParamList, LOG_SCHEDULE_GRANULARITY> {
		enum {
			MIN = util::Access<ParamList, LOG_THREADS>::VALUE +
				util::Access<ParamList, LOG_LOAD_VEC_SIZE>::VALUE +
				util::Access<ParamList, LOG_LOADS_PER_TILE>::VALUE,
			MAX = Ranges<ParamList, LOG_THREADS>::MAX +
				Ranges<ParamList, LOG_LOAD_VEC_SIZE>::MAX +
				Ranges<ParamList, LOG_LOADS_PER_TILE>::MAX
		};
	};
};


/******************************************************************************
 * Spine Tuning Parameter Enumerations and Ranges
 ******************************************************************************/

struct SpineTuning
{
	/**
	 * Tuning params
	 */
	enum Param
	{
		BEGIN,
			LOG_THREADS,
			LOG_LOAD_VEC_SIZE,
			LOG_LOADS_PER_TILE,
			LOG_SCHEDULE_GRANULARITY,
		END,
		CONSECUTIVE_SMEM_ASSIST,
	};

	/**
	 * Policy
	 */
	template <
		typename ProblemType,
		typename ParamList,
		typename BaseKernelPolicy =
			consecutive_reduction::upsweep::KernelPolicy <
				ProblemType,
				TUNE_ARCH,
				false,														// CHECK_ALIGNMENT
				1,															// MIN_CTA_OCCUPANCY,
				util::Access<ParamList, LOG_THREADS>::VALUE, 				// LOG_THREADS,
				util::Access<ParamList, LOG_LOAD_VEC_SIZE>::VALUE,			// LOG_LOAD_VEC_SIZE,
				util::Access<ParamList, LOG_LOADS_PER_TILE>::VALUE,			// LOG_LOADS_PER_TILE,
				B40C_LOG_WARP_THREADS(TUNE_ARCH),							// LOG_RAKING_THREADS,
				util::io::ld::NONE,											// READ_MODIFIER,
				util::io::st::NONE,											// WRITE_MODIFIER,
				util::Access<ParamList, LOG_SCHEDULE_GRANULARITY>::VALUE,	// LOG_SCHEDULE_GRANULARITY
				1> >														// CONSECUTIVE_SMEM_ASSIST

	struct KernelPolicy : BaseKernelPolicy
	{
		typedef typename ProblemType::KeyType 			KeyType;
		typedef typename ProblemType::ValueType			ValueType;
		typedef typename ProblemType::SizeT 			SizeT;
		typedef typename ProblemType::ReductionOp 		ReductionOp;
		typedef typename ProblemType::EqualityOp		EqualityOp;
		typedef typename ProblemType::SpineSizeT		SpineSizeT;

		typedef void (*KernelPtr)(ValueType*, ValueType*, SizeT*, SizeT*, SpineSizeT, ReductionOp);

		// Check if this configuration is worth compiling
		enum {
			REG_MULTIPLIER =
				((sizeof(ValueType) + 4 - 1) / 4) +		// Values
				((sizeof(SizeT) + 4 - 1) / 4),			// Ranks
			REGS_ESTIMATE = (REG_MULTIPLIER * KernelPolicy::TILE_ELEMENTS_PER_THREAD) + 2,
			EST_REGS_OCCUPANCY = B40C_SM_REGISTERS(TUNE_ARCH) / (REGS_ESTIMATE * KernelPolicy::THREADS),
/*
			// ptxas dies on this special case
			INVALID_SPECIAL =
				(TUNE_ARCH < 200) &&
				(sizeof(T) > 4) &&
				(BaseKernelPolicy::LOG_TILE_ELEMENTS > 9),
*/
			VALID_COMPILE =
				((BaseKernelPolicy::VALID > 0) &&
//				(INVALID_SPECIAL == 0) &&
				(REGS_ESTIMATE < 63) &&
				((TUNE_ARCH >= 200) || (BaseKernelPolicy::READ_MODIFIER == util::io::ld::NONE)) &&
				((TUNE_ARCH >= 200) || (BaseKernelPolicy::WRITE_MODIFIER == util::io::st::NONE)) &&
				(BaseKernelPolicy::LOG_THREADS <= B40C_LOG_CTA_THREADS(TUNE_ARCH)) &&
				(EST_REGS_OCCUPANCY > 0))
		};

		static std::string TypeString()
		{
			char buffer[32];
			sprintf(buffer, "%d, %d, %d",
				KernelPolicy::LOG_THREADS,
				KernelPolicy::LOG_LOAD_VEC_SIZE,
				KernelPolicy::LOG_LOADS_PER_TILE);
			return buffer;
		}

		template <int VALID, int DUMMY = 0>
		struct GenKernel
		{
			static KernelPtr Kernel() {
				return consecutive_reduction::spine::Kernel<KernelPolicy>;
			}
		};

		template <int DUMMY>
		struct GenKernel<0, DUMMY>
		{
			static KernelPtr Kernel() {
				return NULL;
			}
		};

		static KernelPtr Kernel() {
			return GenKernel<VALID_COMPILE>::Kernel();
		}
	};


	/**
	 * Ranges for the tuning params
	 */
	template <typename ParamList, int PARAM> struct Ranges;

	// LOG_THREADS
	template <typename ParamList>
	struct Ranges<ParamList, LOG_THREADS> {
		enum {
			MIN = 5,	// 32
			MAX = 10	// 1024
		};
	};

	// LOG_LOAD_VEC_SIZE
	template <typename ParamList>
	struct Ranges<ParamList, LOG_LOAD_VEC_SIZE> {
		enum {
			MIN = 0,
			MAX = 2
		};
	};

	// LOG_LOADS_PER_TILE
	template <typename ParamList>
	struct Ranges<ParamList, LOG_LOADS_PER_TILE> {
		enum {
			MIN = 0,
			MAX = 2
		};
	};

	// CONSECUTIVE_SMEM_ASSIST
	template <typename ParamList>
	struct Ranges<ParamList, CONSECUTIVE_SMEM_ASSIST> {
		enum {
			MIN = 1,
			MAX = 1
		};
	};

	// LOG_SCHEDULE_GRANULARITY
	template <typename ParamList>
	struct Ranges<ParamList, LOG_SCHEDULE_GRANULARITY> {
		enum {
			MIN = util::Access<ParamList, LOG_THREADS>::VALUE +
				util::Access<ParamList, LOG_LOAD_VEC_SIZE>::VALUE +
				util::Access<ParamList, LOG_LOADS_PER_TILE>::VALUE,
			MAX = MIN
		};
	};

};


/******************************************************************************
 * Downsweep Tuning Parameter Enumerations and Ranges
 ******************************************************************************/

struct DownsweepTuning
{
	/**
	 * Tuning params
	 */
	enum Param
	{
		BEGIN,
			LOG_THREADS,
			LOG_LOAD_VEC_SIZE,
			LOG_LOADS_PER_TILE,
			LOG_SCHEDULE_GRANULARITY,
		END,
		CONSECUTIVE_SMEM_ASSIST,
	};

	/**
	 * Policy
	 */
	template <
		typename ProblemType,
		typename ParamList,
		typename BaseKernelPolicy =
			consecutive_reduction::downsweep::KernelPolicy <
				ProblemType,
				TUNE_ARCH,
				true,														// CHECK_ALIGNMENT
				0,															// MIN_CTA_OCCUPANCY,
				util::Access<ParamList, LOG_THREADS>::VALUE, 				// LOG_THREADS,
				util::Access<ParamList, LOG_LOAD_VEC_SIZE>::VALUE,			// LOG_LOAD_VEC_SIZE,
				util::Access<ParamList, LOG_LOADS_PER_TILE>::VALUE,			// LOG_LOADS_PER_TILE,
				B40C_LOG_WARP_THREADS(TUNE_ARCH),							// LOG_RAKING_THREADS,
				util::io::ld::NONE,											// READ_MODIFIER,
				util::io::st::NONE,											// WRITE_MODIFIER,
				util::Access<ParamList, LOG_SCHEDULE_GRANULARITY>::VALUE,	// LOG_SCHEDULE_GRANULARITY
				false,														// TWO_PHASE_SCATTER
				1> >														// CONSECUTIVE_SMEM_ASSIST

	struct KernelPolicy : BaseKernelPolicy
	{
		typedef typename ProblemType::KeyType 			KeyType;
		typedef typename ProblemType::ValueType			ValueType;
		typedef typename ProblemType::SizeT 			SizeT;
		typedef typename ProblemType::ReductionOp 		ReductionOp;
		typedef typename ProblemType::EqualityOp		EqualityOp;
		typedef typename ProblemType::SpineSizeT		SpineSizeT;

		typedef void (*KernelPtr)(KeyType*, KeyType*, ValueType*, ValueType*, ValueType*,  SizeT*, SizeT*, ReductionOp, EqualityOp, util::CtaWorkDistribution<SizeT>);

		// Check if this configuration is worth compiling
		enum {
			REG_MULTIPLIER =
				((sizeof(KeyType) + 4 - 1) / 4) + 		// keys
				((sizeof(ValueType) + 4 - 1) / 4) +		// values
				((sizeof(int) + 4 - 1) / 4) +			// head_flags
				((sizeof(SizeT) + 4 - 1) / 4),			// ranks
			REGS_ESTIMATE = (REG_MULTIPLIER * KernelPolicy::TILE_ELEMENTS_PER_THREAD) + 6,

			EST_REGS_OCCUPANCY = B40C_SM_REGISTERS(TUNE_ARCH) / (REGS_ESTIMATE * KernelPolicy::THREADS),

			VALID_COMPILE =
				((BaseKernelPolicy::VALID > 0) &&
				(REGS_ESTIMATE < 63) &&
				((TUNE_ARCH >= 200) || (BaseKernelPolicy::READ_MODIFIER == util::io::ld::NONE)) &&
				((TUNE_ARCH >= 200) || (BaseKernelPolicy::WRITE_MODIFIER == util::io::st::NONE)) &&
				(BaseKernelPolicy::LOG_THREADS <= B40C_LOG_CTA_THREADS(TUNE_ARCH)) &&
				(EST_REGS_OCCUPANCY > 0)),
		};

		static std::string TypeString()
		{
			char buffer[32];
			sprintf(buffer, "%d, %d, %d",
				KernelPolicy::LOG_THREADS,
				KernelPolicy::LOG_LOAD_VEC_SIZE,
				KernelPolicy::LOG_LOADS_PER_TILE);
			return buffer;
		}

		template <int VALID, int DUMMY = 0>
		struct GenKernel
		{
			static KernelPtr Kernel() {
				return consecutive_reduction::downsweep::Kernel<KernelPolicy>;
			}
		};

		template <int DUMMY>
		struct GenKernel<0, DUMMY>
		{
			static KernelPtr Kernel() {
				return NULL;
			}
		};

		static KernelPtr Kernel() {
			return GenKernel<VALID_COMPILE>::Kernel();
		}
	};


	/**
	 * Ranges for the tuning params
	 */
	template <typename ParamList, int PARAM> struct Ranges;

	// LOG_THREADS
	template <typename ParamList>
	struct Ranges<ParamList, LOG_THREADS> {
		enum {
			MIN = 5,	// 32
			MAX = 10	// 1024
		};
	};

	// LOG_LOAD_VEC_SIZE
	template <typename ParamList>
	struct Ranges<ParamList, LOG_LOAD_VEC_SIZE> {
		enum {
			MIN = 0,
			MAX = 2
		};
	};

	// LOG_LOADS_PER_TILE
	template <typename ParamList>
	struct Ranges<ParamList, LOG_LOADS_PER_TILE> {
		enum {
			MIN = 0,
			MAX = 2
		};
	};

	// CONSECUTIVE_SMEM_ASSIST
	template <typename ParamList>
	struct Ranges<ParamList, CONSECUTIVE_SMEM_ASSIST> {
		enum {
			MIN = 1,
			MAX = 1
		};
	};

	// LOG_SCHEDULE_GRANULARITY
	template <typename ParamList>
	struct Ranges<ParamList, LOG_SCHEDULE_GRANULARITY> {
		enum {
			MIN = util::Access<ParamList, LOG_THREADS>::VALUE +
				util::Access<ParamList, LOG_LOAD_VEC_SIZE>::VALUE +
				util::Access<ParamList, LOG_LOADS_PER_TILE>::VALUE,
			MAX = Ranges<ParamList, LOG_THREADS>::MAX +
				Ranges<ParamList, LOG_LOAD_VEC_SIZE>::MAX +
				Ranges<ParamList, LOG_LOADS_PER_TILE>::MAX
		};
	};
};


/******************************************************************************
 * General Tuning Parameter Enumerations and Ranges
 ******************************************************************************/

struct GeneralTuning
{
	enum Param
	{
		PARAM_BEGIN,
		PARAM_END,

		// Parameters below here are currently not part of the tuning sweep
		READ_MODIFIER,
		WRITE_MODIFIER,
		UNIFORM_SMEM_ALLOCATION,
		UNIFORM_GRID_SIZE,
		LOG_SCHEDULE_GRANULARITY,
	};


	/**
	 * Ranges for the tuning params
	 */
	template <typename ParamList, int PARAM> struct Ranges;

	// READ_MODIFIER
	template <typename ParamList>
	struct Ranges<ParamList, READ_MODIFIER> {
		enum {
			MIN = util::io::ld::NONE,
			MAX = util::io::ld::LIMIT - 1,
		};
	};

	// WRITE_MODIFIER
	template <typename ParamList>
	struct Ranges<ParamList, WRITE_MODIFIER> {
		enum {
			MIN = util::io::st::NONE,
			MAX = util::io::st::LIMIT - 1,
		};
	};

	// UNIFORM_SMEM_ALLOCATION
	template <typename ParamList>
	struct Ranges<ParamList, UNIFORM_SMEM_ALLOCATION> {
		enum {
			MIN = 0,
			MAX = 1
		};
	};

	// UNIFORM_GRID_SIZE
	template <typename ParamList>
	struct Ranges<ParamList, UNIFORM_GRID_SIZE> {
		enum {
			MIN = 0,
			MAX = 1
		};
	};
};


/******************************************************************************
 * Generators
 ******************************************************************************/



/**
 * Tuple callback generator
 */
template <
	typename ProblemType,
	typename Tuning,
	typename ConfigMap>
struct Callback
{
	typedef typename ConfigMap::mapped_type 	GrainMap;				// int -> LaunchDetails
	typedef typename ConfigMap::value_type 		ConfigMapPair;			// (string, GrainMap)
	typedef typename GrainMap::mapped_type 		LaunchDetails;			// (KernelDetails, kernel function ptr)
	typedef typename GrainMap::value_type 		GrainLaunchDetails;		// (int, LaunchDetails)


	ConfigMap *config_map;

	Callback(ConfigMap *config_map) : config_map(config_map) {}

	void Generate()
	{
		util::ParamListSweep<
			Tuning::BEGIN + 1,
			Tuning::END,
			Tuning::template Ranges>::template Invoke<util::EmptyTuple>(*this);
	}

	template <typename ParamList>
	void Invoke()
	{
		typedef typename Tuning::template KernelPolicy<
			ProblemType,
			ParamList> KernelPolicy;

		// Type string for this config family
		std::string typestring = KernelPolicy::TypeString();

		// Create pairing between kernel-details and kernel-pointer
		LaunchDetails launch_details(
			KernelDetails(KernelPolicy::THREADS, KernelPolicy::TILE_ELEMENTS),
			KernelPolicy::Kernel());

		// Create pairing between granularity and launch-details
		GrainLaunchDetails grain_launch_details(
			KernelPolicy::LOG_SCHEDULE_GRANULARITY,
			launch_details);

		// Check to see if we've started a grain list
		if (config_map->find(typestring) == config_map->end()) {

			// Not found.  Insert grain pair into new grain map, insert grain map into config map
			GrainMap grain_map;
			grain_map.insert(grain_launch_details);

			config_map->insert(ConfigMapPair(typestring, grain_map));

		} else {

			// Add this scheduling granularity to the config list
			config_map->find(typestring)->second.insert(grain_launch_details);
		}
	}
};



template <typename ProblemType>
struct Enactor : public util::EnactorBase
{
	typedef typename ProblemType::KeyType 			KeyType;
	typedef typename ProblemType::ValueType			ValueType;
	typedef typename ProblemType::SizeT 			SizeT;
	typedef typename ProblemType::ReductionOp 		ReductionOp;
	typedef typename ProblemType::EqualityOp		EqualityOp;
	typedef typename ProblemType::SpineSizeT		SpineSizeT;

	// Kernel pointer types
	typedef void (*UpsweepKernelPtr)(KeyType*, ValueType*, ValueType*, SizeT*, ReductionOp, EqualityOp, util::CtaWorkDistribution<SizeT>);
	typedef void (*SpineKernelPtr)(ValueType*, ValueType*, SizeT*, SizeT*, SpineSizeT, ReductionOp);
	typedef void (*DownsweepKernelPtr)(KeyType*, KeyType*, ValueType*, ValueType*, ValueType*,  SizeT*, SizeT*, ReductionOp, EqualityOp, util::CtaWorkDistribution<SizeT>);

	typedef std::pair<KernelDetails, UpsweepKernelPtr> 		UpsweepLaunchDetails;
	typedef std::pair<KernelDetails, SpineKernelPtr> 		SpineLaunchDetails;
	typedef std::pair<KernelDetails, DownsweepKernelPtr> 	DownsweepLaunchDetails;

	// Config grain-map types (LOG_GRANULARITY -> kernel pointer)
	typedef std::map<int, UpsweepLaunchDetails> 		UpsweepGrainMap;
	typedef std::map<int, SpineLaunchDetails> 			SpineGrainMap;
	typedef std::map<int, DownsweepLaunchDetails>		DownsweepGrainMap;

	// Config map types (tune-string -> grain map)
	typedef std::map<std::string, UpsweepGrainMap>		UpsweepMap;
	typedef std::map<std::string, SpineGrainMap> 		SpineMap;
	typedef std::map<std::string, DownsweepGrainMap>	DownsweepMap;

	// Configuration maps
	UpsweepMap 		upsweep_configs;
	SpineMap 		spine_configs;
	DownsweepMap 	downsweep_configs;

	// Temporary device storage needed for scanning value partials produced
	// by separate CTAs
	util::Spine partial_spine;

	// Temporary device storage needed for scanning flag partials produced
	// by separate CTAs
	util::Spine flag_spine;

	util::DoubleBuffer<KeyType, ValueType> 	d_problem_storage;
	SizeT 										num_elements;
	ReductionOp 								reduction_op;
	EqualityOp									equality_op;
	SizeT										*d_num_compacted;

	util::DoubleBuffer<KeyType, ValueType> 	h_problem_storage;	// host problem storage (selector points to input, but output contains reference result)
	SizeT 										h_num_compacted;		// number of elements in reference result

	/**
	 * Constructor
	 */
	Enactor(
		ReductionOp reduction_op,
		EqualityOp equality_op) :
			reduction_op(reduction_op),
			equality_op(equality_op),
			d_num_compacted(NULL)
	{
		// Pre-allocate our spines

		if (partial_spine.Setup<long long>(SmCount() * 8 * 8)) exit(1);
		if (flag_spine.Setup<long long>(SmCount() * 8 * 8)) exit(1);

		// Generate all config maps

		Callback<ProblemType, UpsweepTuning, UpsweepMap> 		upsweep_callback(&upsweep_configs);
		Callback<ProblemType, SpineTuning, SpineMap> 			spine_callback(&spine_configs);
		Callback<ProblemType, DownsweepTuning, DownsweepMap> 	downsweep_callback(&downsweep_configs);

		upsweep_callback.Generate();
		spine_callback.Generate();
		downsweep_callback.Generate();
	}


	/**
	 *
	 */
	hipError_t RunSample(
		int log_schedule_granularity,
		UpsweepLaunchDetails upsweep_details,
		SpineLaunchDetails spine_details,
		DownsweepLaunchDetails downsweep_details)
	{
		const bool OVERSUBSCRIBED_GRID_SIZE = true;
		const bool UNIFORM_SMEM_ALLOCATION = false;
		const bool UNIFORM_GRID_SIZE = false;

		hipError_t retval = hipSuccess;
		do {

			// Max CTA occupancy for the actual target device
			int max_cta_occupancy;
			if (retval = MaxCtaOccupancy(
				max_cta_occupancy,
				upsweep_details.second,
				upsweep_details.first.threads,
				downsweep_details.second,
				downsweep_details.first.threads)) break;

			// Compute sweep grid size
			int sweep_grid_size = GridSize(
				OVERSUBSCRIBED_GRID_SIZE,
				1 << log_schedule_granularity,
				max_cta_occupancy,
				num_elements,
				g_max_ctas);

			// Compute spine elements: one element per CTA, rounded
			// up to nearest spine tile size
			int spine_elements = ((sweep_grid_size + spine_details.first.tile_elements - 1) / spine_details.first.tile_elements) *
				spine_details.first.tile_elements;

			// Obtain a CTA work distribution
			util::CtaWorkDistribution<SizeT> work;
			work.Init(num_elements, sweep_grid_size, log_schedule_granularity);

			if (ENACTOR_DEBUG) {
				printf("Work: ");
				work.Print();
			}

			// Make sure our spine is big enough
			if (retval = partial_spine.Setup<ValueType>(spine_elements)) break;
			if (retval = flag_spine.Setup<SizeT>(spine_elements)) break;

			// If we're to output the compacted sizes to device memory, write out
			// compacted size to the last element of our flag spine instead
			d_num_compacted = ((SizeT*) flag_spine()) + spine_elements - 1;

			int dynamic_smem[3] = 	{0, 0, 0};
			int grid_size[3] = 		{work.grid_size, 1, work.grid_size};

			// Tuning option: make sure all kernels have the same overall smem allocation
			if (UNIFORM_SMEM_ALLOCATION) if (retval = PadUniformSmem(
				dynamic_smem,
				upsweep_details.second,
				spine_details.second,
				downsweep_details.second)) break;

			// Tuning option: make sure that all kernels launch the same number of CTAs)
			if (UNIFORM_GRID_SIZE) grid_size[1] = grid_size[0];

			if (ENACTOR_DEBUG) {
				printf("Upsweep<<<%d,%d,%d>>> Spine<<<%d,%d,%d>>> Downsweep<<<%d,%d,%d>>>\n",
					grid_size[0], upsweep_details.first.threads, dynamic_smem[0],
					grid_size[1], spine_details.first.threads, dynamic_smem[1],
					grid_size[2], downsweep_details.first.threads, dynamic_smem[2]);
			}

			// Upsweep scan into spine
			upsweep_details.second<<<grid_size[0], upsweep_details.first.threads, dynamic_smem[0]>>>(
				d_problem_storage.d_keys[d_problem_storage.selector],
				d_problem_storage.d_values[d_problem_storage.selector],
				(ValueType*) partial_spine(),
				(SizeT*) flag_spine(),
				reduction_op,
				equality_op,
				work);

			if (ENACTOR_DEBUG && (retval = util::B40CPerror(hipDeviceSynchronize(), "Enactor UpsweepKernel failed ", __FILE__, __LINE__, ENACTOR_DEBUG))) break;

			// Spine scan
			spine_details.second<<<grid_size[1], spine_details.first.threads, dynamic_smem[1]>>>(
				(ValueType*) partial_spine(),
				(ValueType*) partial_spine(),
				(SizeT*) flag_spine(),
				(SizeT*) flag_spine(),
				spine_elements,
				reduction_op);

			if (ENACTOR_DEBUG && (retval = util::B40CPerror(hipDeviceSynchronize(), "Enactor SpineKernel failed ", __FILE__, __LINE__, ENACTOR_DEBUG))) break;

			// Downsweep from spine
			downsweep_details.second<<<grid_size[2], downsweep_details.first.threads, dynamic_smem[2]>>>(
				d_problem_storage.d_keys[d_problem_storage.selector],
				d_problem_storage.d_keys[d_problem_storage.selector ^ 1],
				d_problem_storage.d_values[d_problem_storage.selector],
				d_problem_storage.d_values[d_problem_storage.selector ^ 1],
				(ValueType*) partial_spine(),
				(SizeT*) flag_spine(),
				d_num_compacted,
				reduction_op,
				equality_op,
				work);

			if (ENACTOR_DEBUG && (retval = util::B40CPerror(hipDeviceSynchronize(), "Enactor DownsweepKernel failed ", __FILE__, __LINE__, ENACTOR_DEBUG))) break;

		} while (0);

		return retval;
	}


	/**
	 *
	 */
	void TimeSample(
		int log_schedule_granularity,
		UpsweepLaunchDetails upsweep_details,
		SpineLaunchDetails spine_details,
		DownsweepLaunchDetails downsweep_details)
	{
		// Check if valid for dispatch
		if (!upsweep_details.second || !spine_details.second || !downsweep_details.second) {
			return;
		}

		// Invoke kernels (warmup)
		ENACTOR_DEBUG = g_verbose;
		if (RunSample(
			log_schedule_granularity,
			upsweep_details,
			spine_details,
			downsweep_details))
		{
			exit(1);
		}
		ENACTOR_DEBUG = false;

		// Perform the timed number of iterations
		GpuTimer timer;
		double elapsed = 0;
		for (int i = 0; i < g_iterations; i++) {

			// Start cuda timing record
			timer.Start();

			// Invoke kernels
			if (RunSample(
				log_schedule_granularity,
				upsweep_details,
				spine_details,
				downsweep_details))
			{
				exit(1);
			}

			// End cuda timing record
			timer.Stop();
			elapsed += timer.ElapsedMillis();

			// Flushes any stdio from the GPU
			if (util::B40CPerror(hipDeviceSynchronize(), "TimedCopy hipDeviceSynchronize failed: ", __FILE__, __LINE__)) {
				exit(1);
			}
		}

		// Display timing information
		long long bytes = ((num_elements * 2) + h_num_compacted) * (sizeof(KeyType) + sizeof(ValueType));
		double avg_runtime = elapsed / g_iterations;
		double throughput =  0.0;
		double bandwidth =  0.0;

		if (avg_runtime > 0.0) {
			throughput = ((double) num_elements) / avg_runtime / 1000.0 / 1000.0;
			bandwidth = bytes / avg_runtime / 1000.0 / 1000.0;
		}

		printf(", %f, %f, %f, ",
			avg_runtime, throughput, bandwidth);
		fflush(stdout);

		if (g_verify) {

			CompareDeviceResults(
				h_problem_storage.d_keys[1],
				d_problem_storage.d_keys[1],
				h_num_compacted);
			printf(", ");
			CompareDeviceResults(
				h_problem_storage.d_values[1],
				d_problem_storage.d_values[1],
				h_num_compacted);
			printf(", ");
			CompareDeviceResults(
				&h_num_compacted,
				d_num_compacted,
				1);
		}
	}


	/**
	 * Iterates over configuration space
	 */
	void IterateConfigSpace()
	{
		int config_id = 0;

		// Iterate upsweep configs
		for (typename UpsweepMap::iterator upsweep_config_itr = upsweep_configs.begin();
			upsweep_config_itr != upsweep_configs.end();
			upsweep_config_itr++)
		{
			std::string upsweep_string = upsweep_config_itr->first;

			// Iterate downsweep configs
			for (typename DownsweepMap::iterator downsweep_config_itr = downsweep_configs.begin();
				downsweep_config_itr != downsweep_configs.end();
				downsweep_config_itr++)
			{
				std::string downsweep_string = downsweep_config_itr->first;

				typename UpsweepGrainMap::iterator upsweep_grain_itr = upsweep_config_itr->second.begin();
				typename DownsweepGrainMap::iterator downsweep_grain_itr = downsweep_config_itr->second.begin();

				while (true) {

					if ((upsweep_grain_itr == upsweep_config_itr->second.end()) ||
						(downsweep_grain_itr == downsweep_config_itr->second.end()))
					{
						// Could not match grain

						printf("Could not match upsweep(%s) with downsweep(%s)\n",
							upsweep_string.c_str(),
							downsweep_string.c_str());

						exit(1);

					}
					else if (upsweep_grain_itr->first == downsweep_grain_itr->first)
					{
						// Matched grain

						// Iterate spine configs
						for (typename SpineMap::iterator spine_config_itr = spine_configs.begin();
							spine_config_itr != spine_configs.end();
							spine_config_itr++)
						{
							std::string spine_string = spine_config_itr->first;

							printf("%d, %d, %s, %s, %s",
								config_id,
								upsweep_grain_itr->first,
								upsweep_string.c_str(),
								spine_string.c_str(),
								downsweep_string.c_str());
							config_id++;

							TimeSample(
								upsweep_grain_itr->first,
								upsweep_grain_itr->second,
								spine_config_itr->second.begin()->second,
								downsweep_grain_itr->second);

							printf("\n");
							fflush(stdout);
						}

						break;

					} else if (upsweep_grain_itr->first < downsweep_grain_itr->first) {
						upsweep_grain_itr++;
					} else {
						downsweep_grain_itr++;
					}
				}
			}
		}
	}


	/**
	 * Creates an example problem and then dispatches the iterations
	 * to the GPU for the given number of iterations, displaying runtime information.
	 */
	void Test(SizeT num_elements)
	{
		this->num_elements = num_elements;

	    // Allocate the consecutive reduction problem on the host
		h_problem_storage.d_keys[0] = (KeyType*) malloc(num_elements * sizeof(KeyType));
		h_problem_storage.d_keys[1] = (KeyType*) malloc(num_elements * sizeof(KeyType));
		h_problem_storage.d_values[0] = (ValueType*) malloc(num_elements * sizeof(ValueType));
		h_problem_storage.d_values[1] = (ValueType*) malloc(num_elements * sizeof(ValueType));

		if (!h_problem_storage.d_keys[0] || !h_problem_storage.d_keys[1] || !h_problem_storage.d_values[0] || !h_problem_storage.d_values[1]){
			fprintf(stderr, "Host malloc of problem data failed\n");
			exit(1);
		}

		// Initialize problem
		if (g_verbose) printf("Input problem: \n");
		for (int i = 0; i < num_elements; i++) {
			h_problem_storage.d_keys[0][i] = (i / 7) & 1;							// Toggle every 7 elements
	//		util::RandomBits<T>(h_problem_storage.d_keys[0][i], 1, 1);				// Entropy-reduced random 0|1 values: roughly 26 / 64 elements toggled

			h_problem_storage.d_values[0][i] = 1;

/*
			if (g_verbose) {
				printf("(%lld, %lld), ",
					(long long) h_problem_storage.d_keys[0][i],
					(long long) h_problem_storage.d_values[0][i]);
			}
*/
		}
//		if (g_verbose) printf("\n");


		// Compute reference solution
		h_num_compacted = 0;
		h_problem_storage.d_keys[1][0] = h_problem_storage.d_keys[0][0];

		for (SizeT i = 0; i < num_elements; ++i) {

			if (h_problem_storage.d_keys[1][h_num_compacted] != h_problem_storage.d_keys[0][i]) {

				h_num_compacted++;
				h_problem_storage.d_keys[1][h_num_compacted] = h_problem_storage.d_keys[0][i];
				h_problem_storage.d_values[1][h_num_compacted] = h_problem_storage.d_values[0][i];

			} else {

				if (i == 0) {
					h_problem_storage.d_values[1][h_num_compacted] =
						h_problem_storage.d_values[0][i];
				} else {
					h_problem_storage.d_values[1][h_num_compacted] = reduction_op(
						h_problem_storage.d_values[1][h_num_compacted],
						h_problem_storage.d_values[0][i]);
				}
			}
		}
		h_num_compacted++;

		// Allocate device storage
		if (util::B40CPerror(hipMalloc((void**) &d_problem_storage.d_keys[0], sizeof(KeyType) * num_elements),
			"TimedConsecutiveReduction hipMalloc d_keys failed: ", __FILE__, __LINE__)) exit(1);
		if (util::B40CPerror(hipMalloc((void**) &d_problem_storage.d_keys[1], sizeof(KeyType) * num_elements),
			"TimedConsecutiveReduction hipMalloc d_keys failed: ", __FILE__, __LINE__)) exit(1);
		if (util::B40CPerror(hipMalloc((void**) &d_problem_storage.d_values[0], sizeof(ValueType) * num_elements),
			"TimedConsecutiveReduction hipMalloc d_values failed: ", __FILE__, __LINE__)) exit(1);
		if (util::B40CPerror(hipMalloc((void**) &d_problem_storage.d_values[1], sizeof(ValueType) * num_elements),
			"TimedConsecutiveReduction hipMalloc d_values failed: ", __FILE__, __LINE__)) exit(1);

		// Move a fresh copy of the problem into device storage
		if (util::B40CPerror(hipMemcpy(
				d_problem_storage.d_keys[0],
				h_problem_storage.d_keys[0],
				sizeof(KeyType) * num_elements,
				hipMemcpyHostToDevice),
			"TimedConsecutiveReduction hipMemcpy d_keys failed: ", __FILE__, __LINE__)) exit(1);
		if (util::B40CPerror(hipMemcpy(
				d_problem_storage.d_values[0],
				h_problem_storage.d_values[0],
				sizeof(ValueType) * num_elements,
				hipMemcpyHostToDevice),
			"TimedConsecutiveReduction hipMemcpy d_values failed: ", __FILE__, __LINE__)) exit(1);

		// Iterate configuration space
		IterateConfigSpace();

		// Free allocated memory
	    if (d_problem_storage.d_keys[0]) hipFree(d_problem_storage.d_keys[0]);
	    if (d_problem_storage.d_keys[1]) hipFree(d_problem_storage.d_keys[1]);
	    if (d_problem_storage.d_values[0]) hipFree(d_problem_storage.d_values[0]);
	    if (d_problem_storage.d_values[1]) hipFree(d_problem_storage.d_values[1]);

		// Free our allocated host memory
		if (h_problem_storage.d_keys[0]) free(h_problem_storage.d_keys[0]);
		if (h_problem_storage.d_keys[1]) free(h_problem_storage.d_keys[1]);
		if (h_problem_storage.d_values[0]) free(h_problem_storage.d_values[0]);
		if (h_problem_storage.d_values[1]) free(h_problem_storage.d_values[1]);
	}


};



/******************************************************************************
 * Test
 ******************************************************************************/



/**
 * Creates an example problem and then dispatches the iterations
 * to the GPU for the given number of iterations, displaying runtime information.
 */
template<
	typename KeyType,
	typename ValueType,
	typename SizeT,
	typename ReductionOp,
	typename EqualityOp>
void Test(
	SizeT num_elements,
	ReductionOp reduction_op,
	EqualityOp equality_op)
{
	typedef consecutive_reduction::ProblemType<
		KeyType,
		ValueType,
		SizeT,
		ReductionOp,
		EqualityOp> ProblemType;

	// Create enactor
	Enactor<ProblemType> enactor(
		reduction_op,
		equality_op);

	// Run test
	enactor.Test(num_elements);
}




/******************************************************************************
 * Main
 ******************************************************************************/

int main(int argc, char** argv)
{

	CommandLineArgs args(argc, argv);
	DeviceInit(args);

	// Seed random number generator
	srand(0);				// presently deterministic

	// Use 32-bit integer for array indexing
	typedef int SizeT;
	SizeT num_elements = 1024;

	// Parse command line arguments
    if (args.CheckCmdLineFlag("help")) {
		Usage();
		return 0;
	}
    args.GetCmdLineArgument("i", g_iterations);
    args.GetCmdLineArgument("n", num_elements);
    args.GetCmdLineArgument("max-ctas", g_max_ctas);
    g_verify = args.CheckCmdLineFlag("verify");
	g_verbose = args.CheckCmdLineFlag("v");

	util::CudaProperties cuda_props;

	printf("Test Scan: %d iterations, %lu elements", g_iterations, (unsigned long) num_elements);
	printf("\nCodeGen: \t[device_sm_version: %d, kernel_ptx_version: %d]\n\n",
		cuda_props.device_sm_version, cuda_props.kernel_ptx_version);

	printf(""
		"TuneID, "
		"SCHEDULING_GRANULARITY, "

		"UPSWEEP_LOG_THREADS, "
		"UPSWEEP_LOG_LOAD_VEC_SIZE, "
		"UPSWEEP_LOG_LOADS_PER_TILE, "

		"SPINE_LOG_THREADS, "
		"SPINE_LOG_LOAD_VEC_SIZE, "
		"SPINE_LOG_LOADS_PER_TILE, "

		"DOWNSWEEP_LOG_THREADS, "
		"DOWNSWEEP_LOG_LOAD_VEC_SIZE, "
		"DOWNSWEEP_LOG_LOADS_PER_TILE, "

		"elapsed time (ms), "
		"throughput (10^9 items/s), "
		"bandwidth (10^9 B/s)");
	if (g_verify) printf(", Correctness");
	printf("\n");


	// Execute test(s)
#if (TUNE_SIZE == 0) || (TUNE_SIZE == 1)
	{
		typedef unsigned char T;
		Sum<T> reduction_op;
		Equality<T> equality_op;

		Test<T, T>(num_elements * 4, reduction_op, equality_op);
	}
#endif
#if (TUNE_SIZE == 0) || (TUNE_SIZE == 2)
	{
		typedef unsigned short T;
		Sum<T> reduction_op;
		Equality<T> equality_op;
		Test<T, T>(num_elements * 2, reduction_op, equality_op);
	}
#endif
#if (TUNE_SIZE == 0) || (TUNE_SIZE == 4)
	{
		typedef unsigned int T, T;
		Sum<T> reduction_op;
		Equality<T> equality_op;
		Test<T, T>(num_elements, reduction_op, equality_op);
	}
#endif
#if (TUNE_SIZE == 0) || (TUNE_SIZE == 8)
	{
		typedef unsigned long long T;
		Sum<T> reduction_op;
		Equality<T> equality_op;
		Test<T, T>(num_elements / 2, reduction_op, equality_op);
	}
#endif

	return 0;
}



