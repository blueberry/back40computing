#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2010-2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2013, NVIDIA CORPORATION.  All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/


/******************************************************************************
 * Simple test driver program for consecutive reduction.
 ******************************************************************************/

#include <stdio.h> 

// Test utils
#include "b40c_test_util.h"
#include "test_consecutive_reduction.h"

#include <b40c/util/multiple_buffering.cuh>

using namespace b40c;

/******************************************************************************
 * Defines, constants, globals
 ******************************************************************************/

bool 	g_verbose 						= false;
bool 	g_sweep							= false;
int 	g_max_ctas 						= 0;
int 	g_iterations  					= 1;


/******************************************************************************
 * Utility Routines
 ******************************************************************************/

/**
 * Displays the commandline usage for this tool
 */
void Usage()
{
	printf("\ntest_consecutive_reduction [--device=<device index>] [--v] [--i=<num-iterations>] "
			"[--max-ctas=<max-thread-blocks>] [--n=<num-elements>] [--sweep]\n");
	printf("\n");
	printf("\t--v\tDisplays copied results to the console.\n");
	printf("\n");
	printf("\t--i\tPerforms the consecutive reduction operation <num-iterations> times\n");
	printf("\t\t\ton the device. Re-copies original input each time. Default = 1\n");
	printf("\n");
	printf("\t--n\tThe number of elements to comprise the sample problem\n");
	printf("\t\t\tDefault = 512\n");
	printf("\n");
}



/**
 * Creates an example consecutive reduction problem and then dispatches the problem
 * to the GPU for the given number of iterations, displaying runtime information.
 */
template<
	typename T,
	typename SizeT,
	typename ReductionOp>
void TestConsecutiveReduction(
	SizeT num_elements,
	ReductionOp scan_op)
{
    // Allocate the consecutive reduction problem on the host
	typedef util::DoubleBuffer<T, T> DoubleBuffer;
	DoubleBuffer h_problem_storage;

	h_problem_storage.d_keys[0] = (T*) malloc(num_elements * sizeof(T));
	h_problem_storage.d_keys[1] = (T*) malloc(num_elements * sizeof(T));
	h_problem_storage.d_values[0] = (T*) malloc(num_elements * sizeof(T));
	h_problem_storage.d_values[1] = (T*) malloc(num_elements * sizeof(T));

	if (!h_problem_storage.d_keys[0] || !h_problem_storage.d_keys[1] || !h_problem_storage.d_values[0] || !h_problem_storage.d_values[1]){
		fprintf(stderr, "Host malloc of problem data failed\n");
		exit(1);
	}

	// Initialize problem
	if (g_verbose) printf("Input problem: \n");
	for (int i = 0; i < num_elements; i++) {
		h_problem_storage.d_keys[0][i] = (i / 7) & 1;							// Toggle every 7 elements
//		util::RandomBits<T>(h_problem_storage.d_keys[0][i], 1, 1);				// Entropy-reduced random 0|1 values: roughly 26 / 64 elements toggled

		h_problem_storage.d_values[0][i] = 1;

		if (g_verbose) {
			printf("(%lld, %lld), ",
				(long long) h_problem_storage.d_keys[0][i],
				(long long) h_problem_storage.d_values[0][i]);
		}
	}
	if (g_verbose) printf("\n");

	// Compute reference solution
	SizeT num_compacted = 0;
	h_problem_storage.d_keys[1][0] = h_problem_storage.d_keys[0][0];

	for (SizeT i = 0; i < num_elements; ++i) {

		if (h_problem_storage.d_keys[1][num_compacted] != h_problem_storage.d_keys[0][i]) {

			num_compacted++;
			h_problem_storage.d_keys[1][num_compacted] = h_problem_storage.d_keys[0][i];
			h_problem_storage.d_values[1][num_compacted] = h_problem_storage.d_values[0][i];

		} else {

			if (i == 0) {
				h_problem_storage.d_values[1][num_compacted] =
					h_problem_storage.d_values[0][i];
			} else {
				h_problem_storage.d_values[1][num_compacted] = scan_op(
					h_problem_storage.d_values[1][num_compacted],
					h_problem_storage.d_values[0][i]);
			}
		}
	}
	num_compacted++;

	Equality<typename DoubleBuffer::KeyType> equality_op;

	// Execute test(s), optionally sweeping problem size downward
	SizeT orig_num_elements = num_elements;
	do {

		printf("\nLARGE config:\t");
		double large = TimedConsecutiveReduction<consecutive_reduction::LARGE_SIZE>(
			h_problem_storage,
			num_elements,
			num_compacted,
			scan_op,
			equality_op,
			g_max_ctas,
			g_verbose,
			g_iterations);

		printf("\nSMALL config:\t");
		double small = TimedConsecutiveReduction<consecutive_reduction::SMALL_SIZE>(
			h_problem_storage,
			num_elements,
			num_compacted,
			scan_op,
			equality_op,
			g_max_ctas,
			g_verbose,
			g_iterations);

		if (small > large) {
			printf("%lu-byte elements: Small faster at %lu elements\n", (unsigned long) sizeof(T), (unsigned long) num_elements);
		}

		num_elements -= 4096;

	} while (g_sweep && (num_elements < orig_num_elements ));

	// Free our allocated host memory
	if (h_problem_storage.d_keys[0]) free(h_problem_storage.d_keys[0]);
	if (h_problem_storage.d_keys[1]) free(h_problem_storage.d_keys[1]);
	if (h_problem_storage.d_values[0]) free(h_problem_storage.d_values[0]);
	if (h_problem_storage.d_values[1]) free(h_problem_storage.d_values[1]);
}




/******************************************************************************
 * Main
 ******************************************************************************/

int main(int argc, char** argv)
{
	// Initialize commandline args and device
	CommandLineArgs args(argc, argv);
	DeviceInit(args);

	// Seed random number generator
	srand(0);				// presently deterministic
	//srand(time(NULL));

	// Use 32-bit integer for array indexing
	typedef int SizeT;
	SizeT num_elements = 1024;

	// Parse command line arguments
    if (args.CheckCmdLineFlag("help")) {
		Usage();
		return 0;
	}
    g_sweep = args.CheckCmdLineFlag("sweep");
    args.GetCmdLineArgument("i", g_iterations);
    args.GetCmdLineArgument("n", num_elements);
    args.GetCmdLineArgument("max-ctas", g_max_ctas);
	g_verbose = args.CheckCmdLineFlag("v");

	{
		printf("\n-- UNSIGNED CHAR ----------------------------------------------\n");
		typedef unsigned char T;
		Sum<T> op;
		TestConsecutiveReduction<T>(num_elements * 4, op);
	}
	{
		printf("\n-- UNSIGNED SHORT ----------------------------------------------\n");
		typedef unsigned short T;
		Sum<T> op;
		TestConsecutiveReduction<T>(num_elements * 2, op);
	}
	{
		printf("\n-- UNSIGNED INT -----------------------------------------------\n");
		typedef unsigned int T;
		Sum<T> op;
		TestConsecutiveReduction<T>(num_elements, op);
	}
	{
		printf("\n-- UNSIGNED LONG LONG -----------------------------------------\n");
		typedef unsigned long long T;
		Sum<T> op;
		TestConsecutiveReduction<T>(num_elements / 2, op);
	}

	return 0;
}



