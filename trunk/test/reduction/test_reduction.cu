#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2010-2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2013, NVIDIA CORPORATION.  All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/


/******************************************************************************
 * Simple test driver program for reduction.
 ******************************************************************************/

#include <stdio.h> 

// Test utils
#include "b40c_test_util.h"
#include "test_reduction.h"

using namespace b40c;

/******************************************************************************
 * Defines, constants, globals
 ******************************************************************************/

bool 	g_verbose 						= false;
bool 	g_sweep							= false;
int 	g_max_ctas 						= 0;
int 	g_iterations  					= 1;



/******************************************************************************
 * Utility Routines
 ******************************************************************************/

/**
 * Displays the commandline usage for this tool
 */
void Usage()
{
	printf("\ntest_reduction [--device=<device index>] [--v] [--i=<num-iterations>] "
			"[--max-ctas=<max-thread-blocks>] [--n=<num-elements>] [--sweep]\n");
	printf("\n");
	printf("\t--v\tDisplays copied results to the console.\n");
	printf("\n");
	printf("\t--i\tPerforms the reduction operation <num-iterations> times\n");
	printf("\t\t\ton the device. Re-copies original input each time. Default = 1\n");
	printf("\n");
	printf("\t--n\tThe number of elements to comprise the sample problem\n");
	printf("\t\t\tDefault = 512\n");
	printf("\n");
}


/**
 * Creates an example reduction problem and then dispatches the problem
 * to the GPU for the given number of iterations, displaying runtime information.
 */
template<
	typename T,
	typename SizeT,
	typename ReductionOp>
void TestReduction(
	SizeT num_elements,
	ReductionOp reduction_op)
{
    // Allocate the reduction problem on the host and fill the keys with random bytes

	T *h_data 			= (T*) malloc(num_elements * sizeof(T));
	T *h_reference 		= (T*) malloc(sizeof(T));

	if ((h_data == NULL) || (h_reference == NULL)){
		fprintf(stderr, "Host malloc of problem data failed\n");
		exit(1);
	}

	for (size_t i = 0; i < num_elements; ++i) {
		// util::RandomBits<T>(h_data[i], 0);
		h_data[i] = i;
		h_reference[0] = (i == 0) ?
			h_data[i] :
			reduction_op(h_reference[0], h_data[i]);
	}

	//
    // Run the timing test(s)
	//


	// Execute test(s), optionally sweeping problem size downward
	size_t orig_num_elements = num_elements;
	do {
		printf("\nLARGE config:\t");
		double large = TimedReduction<reduction::LARGE_SIZE>(
			h_data, h_reference, num_elements, reduction_op, g_max_ctas, g_verbose, g_iterations);

		printf("\nSMALL config:\t");
		double small = TimedReduction<reduction::SMALL_SIZE>(
			h_data, h_reference, num_elements, reduction_op, g_max_ctas, g_verbose, g_iterations);

		if (small > large) {
			printf("%lu-byte elements: Small faster at %lu elements\n", (unsigned long) sizeof(T), (unsigned long) num_elements);
		}

		num_elements -= 4096;

	} while (g_sweep && (num_elements < orig_num_elements ));

	// Free our allocated host memory
	if (h_data) free(h_data);
    if (h_reference) free(h_reference);
}


/******************************************************************************
 * Main
 ******************************************************************************/

int main(int argc, char** argv)
{
	// Initialize commandline args and device
	CommandLineArgs args(argc, argv);
	DeviceInit(args);

	// Seed random number generator
	srand(0);				// presently deterministic
	//srand(time(NULL));

	// Use 32-bit integer for array indexing
	typedef int SizeT;
	SizeT num_elements = 1024;

	// Parse command line arguments
    if (args.CheckCmdLineFlag("help")) {
		Usage();
		return 0;
	}
    g_sweep = args.CheckCmdLineFlag("sweep");
    args.GetCmdLineArgument("i", g_iterations);
    args.GetCmdLineArgument("n", num_elements);
    args.GetCmdLineArgument("max-ctas", g_max_ctas);
	g_verbose = args.CheckCmdLineFlag("v");

	{
		printf("\n-- UNSIGNED CHAR ----------------------------------------------\n");
		typedef unsigned char T;
		Sum<T> reduction_op;
    	TestReduction<T>(num_elements * 4, reduction_op);
	}
	{
		printf("\n-- UNSIGNED SHORT ----------------------------------------------\n");
		typedef unsigned short T;
		Sum<T> reduction_op;
    	TestReduction<T>(num_elements * 2, reduction_op);
	}
	{
		printf("\n-- UNSIGNED INT -----------------------------------------------\n");
		typedef unsigned int T;
		Sum<T> reduction_op;
    	TestReduction<T>(num_elements, reduction_op);
	}
	{
		printf("\n-- UNSIGNED LONG LONG -----------------------------------------\n");
		typedef unsigned long long T;
		Sum<T> reduction_op;
    	TestReduction<T>(num_elements / 2, reduction_op);
	}

	return 0;
}



