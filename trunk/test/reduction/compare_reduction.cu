#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2010-2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2013, NVIDIA CORPORATION.  All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/


/******************************************************************************
 * Simple test driver program for reduction.
 ******************************************************************************/

#include <stdio.h> 

// Test utils
#include "b40c_test_util.h"
#include "test_reduction.h"

#include <thrust/device_vector.h>
#include <thrust/reduce.h>

using namespace b40c;


/******************************************************************************
 * Defines, constants, globals 
 ******************************************************************************/

bool 	g_verbose 						= false;
int 	g_max_ctas 						= 0;
int 	g_iterations  					= 1;



/******************************************************************************
 * Utility Routines
 ******************************************************************************/

/**
 * Displays the commandline usage for this tool
 */
void Usage() 
{
	printf("\ntest_reduction [--device=<device index>] [--v] [--i=<num-iterations>] "
			"[--max-ctas=<max-thread-blocks>] [--n=<num-elements>]\n");
	printf("\n");
	printf("\t--v\tDisplays copied results to the console.\n");
	printf("\n");
	printf("\t--i\tPerforms the reduction operation <num-iterations> times\n");
	printf("\t\t\ton the device. Re-copies original input each time. Default = 1\n");
	printf("\n");
	printf("\t--n\tThe number of elements to comprise the sample problem\n");
	printf("\t\t\tDefault = 512\n");
	printf("\n");
}


/**
 * Timed Thrust reduction.  Uses the GPU to copy the specified vector of elements for the given
 * number of iterations, displaying runtime information.
 */
template <
	typename T,
	typename SizeT,
	typename ReductionOp>
double TimedThrustReduction(
	T *h_data,
	T *h_reference,
	SizeT num_elements,
	ReductionOp reduction_op)
{
	using namespace b40c;

	T h_dest[1] = {0};
	
	// Allocate device storage  
	T *d_src, *d_dest;
	if (util::B40CPerror(hipMalloc((void**) &d_src, sizeof(T) * num_elements),
		"TimedReduction hipMalloc d_src failed: ", __FILE__, __LINE__)) exit(1);
	if (util::B40CPerror(hipMalloc((void**) &d_dest, sizeof(T)),
		"TimedReduction hipMalloc d_dest failed: ", __FILE__, __LINE__)) exit(1);

	// Move a fresh copy of the problem into device storage
	if (util::B40CPerror(hipMemcpy(d_src, h_data, sizeof(T) * num_elements, hipMemcpyHostToDevice),
		"TimedReduction hipMemcpy d_src failed: ", __FILE__, __LINE__)) exit(1);

	// Marker kernel in profiling stream
	util::FlushKernel<void><<<1,1>>>();
	
	// Perform a single iteration to allocate any memory if needed, prime code caches, etc.
	thrust::device_ptr<T> dev_ptr(d_src);		
	h_dest[0] = thrust::reduce(dev_ptr, dev_ptr + num_elements, (T) 0, reduction_op);
	
	// Perform the timed number of iterations
	GpuTimer timer;

	double elapsed = 0;
	for (int i = 0; i < g_iterations; i++) {

		// Marker kernel in profiling stream
		util::FlushKernel<void><<<1,1>>>();

		// Start timing record
		timer.Start();

		h_dest[0] = thrust::reduce(dev_ptr, dev_ptr + num_elements, (T) 0, reduction_op);
		
		// End timing record
		timer.Stop();
		elapsed += (double) timer.ElapsedMillis();
	}

	// Display timing information
	double avg_runtime = elapsed / g_iterations;
	double throughput = ((double) num_elements) / avg_runtime / 1000.0 / 1000.0;
	printf("\nThrust Reduction: %d iterations, %lu elements, ", g_iterations, (unsigned long) num_elements);
    printf("%f GPU ms, %f x10^9 elts/sec, %f x10^9 B/sec, ",
		avg_runtime, throughput, throughput * sizeof(T));
	
    // Free allocated memory
    if (d_src) hipFree(d_src);
    if (d_dest) hipFree(d_dest);

	// Flushes any stdio from the GPU
	hipDeviceSynchronize();

	// Display copied data
	if (g_verbose) {
		printf("Reduction: ");
		PrintValue(h_dest[0]);
		printf(", Reference: ");
		PrintValue(h_reference[0]);
		printf("\n\n");
	}

    // Verify solution
	CompareResults(h_dest, h_reference, 1, true);
	printf("\n");
	fflush(stdout);

	return throughput;
}


/**
 * Creates an example reduction problem and then dispatches the problem
 * to the GPU for the given number of iterations, displaying runtime information.
 */
template<
	typename T,
	typename SizeT,
	typename ReductionOp>
void TestReduction(
	SizeT num_elements,
	ReductionOp reduction_op)
{
    // Allocate the reduction problem on the host and fill the keys with random bytes

	T *h_data 			= (T*) malloc(num_elements * sizeof(T));
	T *h_reference 		= (T*) malloc(sizeof(T));

	if ((h_data == NULL) || (h_reference == NULL)){
		fprintf(stderr, "Host malloc of problem data failed\n");
		exit(1);
	}

	for (size_t i = 0; i < num_elements; ++i) {
		// util::RandomBits<T>(h_data[i], 0);
		h_data[i] = i;
		h_reference[0] = (i == 0) ?
			h_data[i] :
			reduction_op(h_reference[0], h_data[i]);
	}

	//
    // Run the timing test(s)
	//

	double b40c = TimedReduction<reduction::UNKNOWN_SIZE>(
		h_data, h_reference, num_elements, reduction_op, g_max_ctas, g_verbose, g_iterations);

	double thrust = TimedThrustReduction(
		h_data, h_reference, num_elements, reduction_op);

	printf("B40C speedup: %.2f\n", b40c/thrust);

	// Free our allocated host memory 
	if (h_data) free(h_data);
    if (h_reference) free(h_reference);
}


/******************************************************************************
 * Main
 ******************************************************************************/

int main(int argc, char** argv)
{
	// Initialize commandline args and device
	CommandLineArgs args(argc, argv);
	DeviceInit(args);

	// Seed random number generator
	srand(0);				// presently deterministic
	//srand(time(NULL));

	// Use 32-bit integer for array indexing
	typedef int SizeT;
	SizeT num_elements = 1024;

	// Parse command line arguments
    if (args.CheckCmdLineFlag("help")) {
		Usage();
		return 0;
	}
    args.GetCmdLineArgument("i", g_iterations);
    args.GetCmdLineArgument("n", num_elements);
    args.GetCmdLineArgument("max-ctas", g_max_ctas);
	g_verbose = args.CheckCmdLineFlag("v");

	// Execute test(s)
	{
		printf("\n-- UNSIGNED CHAR ----------------------------------------------\n");
		typedef unsigned char T;
		Sum<T> reduction_op;
    	TestReduction<T>(num_elements * 4, reduction_op);
	}
	{
		printf("\n-- UNSIGNED SHORT ----------------------------------------------\n");
		typedef unsigned short T;
		Sum<T> reduction_op;
    	TestReduction<T>(num_elements * 2, reduction_op);
	}
	{
		printf("\n-- UNSIGNED INT -----------------------------------------------\n");
		typedef unsigned int T;
		Sum<T> reduction_op;
    	TestReduction<T>(num_elements, reduction_op);
	}
	{
		printf("\n-- UNSIGNED LONG LONG -----------------------------------------\n");
		typedef unsigned long long T;
		Sum<T> reduction_op;
    	TestReduction<T>(num_elements / 2, reduction_op);
	}

	return 0;
}



