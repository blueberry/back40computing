#include "hip/hip_runtime.h"
/******************************************************************************
 * 
 * Copyright 2010-2012 Duane Merrill
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 ******************************************************************************/


/******************************************************************************
 * Simple test driver program for copy.
 ******************************************************************************/

#include <stdio.h> 

// Test utils
#include "b40c_test_util.h"
#include "test_copy.h"

using namespace b40c;

/******************************************************************************
 * Defines, constants, globals
 ******************************************************************************/

bool 	g_verbose;
bool 	g_sample;
bool 	g_from_host;
int 	g_max_ctas 						= 0;
int 	g_iterations  					= 1;
int 	g_num_elements 					= 1024;
int 	g_src_gpu						= -1;
int 	g_dest_gpu						= -1;


/******************************************************************************
 * Utility Routines
 ******************************************************************************/

/**
 * Displays the commandline usage for this tool
 */
void Usage()
{
	printf("\ntest_copy "
		"[--device=<device index>] "
		"[--v] "
		"[--i=<num-iterations>] "
		"[--max-ctas=<max-thread-blocks>] "
		"[--n=<num-bytes>] "
		"[--sample] "
		"[ [--src=<src-gpu> --dest=<dest-gpu>] | --from-host ]\n");
	printf("\n");
	printf("\t--v\tDisplays copied results to the console.\n");
	printf("\n");
	printf("\t--i\tPerforms the copy operation <num-iterations> times\n");
	printf("\t\t\ton the device. Re-copies original input each time. Default = %d\n", g_iterations);
	printf("\n");
	printf("\t--n\tThe number of bytes to comprise the sample problem\n");
	printf("\t\t\tDefault = %lu\n", (unsigned long) g_num_elements);
	printf("\n");
}


/**
 * Creates an example copy problem and then dispatches the problem
 * to the GPU for the given number of iterations, displaying runtime information.
 */
template <typename SizeT>
void TestCopy(SizeT num_elements)
{
	typedef unsigned char T;

	//
	// Allocate the copy problem on the host and fill the keys with random bytes
	//

	T *h_data = (T*) malloc(num_elements * sizeof(T));
	if (!h_data) {
		fprintf(stderr, "Host malloc of problem data failed\n");
		exit(1);
	}

	for (SizeT i = 0; i < num_elements; ++i) {
		// util::RandomBits<T>(h_data[i], 0);
		h_data[i] = i;
	}

	// Allocate device storage (and leave g_dest_gpu as current gpu)
	T *h_src = NULL;
	T *d_src = NULL;
	T *d_dest = NULL;

	bool same_device = (!g_from_host) && (g_src_gpu == g_dest_gpu);

	if (g_from_host) {
		int flags = hipHostMallocMapped;
		if (util::B40CPerror(hipHostAlloc((void**) &h_src, sizeof(T) * num_elements, flags),
			"TimedCopy hipHostAlloc d_src failed", __FILE__, __LINE__)) exit(1);

		// Map into GPU space
		if (util::B40CPerror(hipHostGetDevicePointer((void **)&d_src, (void *) h_src, 0),
			"TimedCopy hipHostGetDevicePointer h_src failed", __FILE__, __LINE__)) exit(1);

	} else {
		if (util::B40CPerror(hipSetDevice(g_src_gpu),
			"MultiGpuBfsEnactor hipSetDevice failed", __FILE__, __LINE__)) exit(1);
		if (util::B40CPerror(hipMalloc((void**) &d_src, sizeof(T) * num_elements),
			"TimedCopy hipMalloc d_src failed: ", __FILE__, __LINE__)) exit(1);
	}

	if (util::B40CPerror(hipSetDevice(g_dest_gpu),
		"MultiGpuBfsEnactor hipSetDevice failed", __FILE__, __LINE__)) exit(1);
	if (util::B40CPerror(hipMalloc((void**) &d_dest, sizeof(T) * num_elements),
		"TimedCopy hipMalloc d_dest failed: ", __FILE__, __LINE__)) exit(1);

	// Move a fresh copy of the problem into device storage
	if (util::B40CPerror(hipMemcpy(d_src, h_data, sizeof(T) * num_elements, hipMemcpyHostToDevice),
		"TimedCopy hipMemcpy d_src failed: ", __FILE__, __LINE__)) exit(1);

	//
    // Run the timing test(s)
	//

	b40c::GpuTimer gpu_timer;
	double max_exponent 		= log2(double(num_elements)) - 5.0;
	unsigned int max_int 		= (unsigned int) -1;

	if (g_sample) {
		// Sample problem sizes up to num_elements
		printf("ITERATION, ELEMENTS, BYTES, SAMPLES, AVG_MILLIS, BANDWIDTH, STATUS\n");

		for (int i = 0; i < g_iterations; i++) {

			// Sample a problem size
			unsigned int sample;
			b40c::util::RandomBits(sample);
			double scale = double(sample) / max_int;
			SizeT elements = (i < g_iterations / 2) ?
				(SizeT) pow(2.0, (max_exponent * scale) + 5.0) :		// log bias
				elements = scale * num_elements;						// uniform bias

			printf("%d, ", i);

			// One iteration at that problem size
			TimedCopy<copy::UNKNOWN_SIZE>(
				h_data,
				d_src,
				d_dest,
				elements,
				g_max_ctas,
				g_verbose,
				1,
				same_device,
				false);
		}
	} else {
		// Test large and small configs on num_elements
		printf("ELEMENTS, BYTES, SAMPLES, AVG_MILLIS, BANDWIDTH, STATUS\n\n");

		printf("Large-problem configuration:\n");
		TimedCopy<copy::LARGE_SIZE>(
			h_data,
			d_src,
			d_dest,
			num_elements,
			g_max_ctas,
			g_verbose,
			g_iterations,
			same_device);

		printf("\n");

		printf("Small-problem configuration:\n");
		TimedCopy<copy::SMALL_SIZE>(
			h_data,
			d_src,
			d_dest,
			num_elements,
			g_max_ctas,
			g_verbose,
			g_iterations,
			same_device);
	}

    // Free allocated memory
	if (h_data) free(h_data);
    if (h_src) {
		hipHostFree(h_src);
	} else {
		hipFree(d_src);
	}
    if (d_dest) hipFree(d_dest);
}


/******************************************************************************
 * Main
 ******************************************************************************/

int main(int argc, char** argv)
{

	CommandLineArgs args(argc, argv);
	DeviceInit(args);
	hipSetDeviceFlags(hipDeviceMapHost);

	//srand(time(NULL));
	srand(0);				// presently deterministic

	// Check command line arguments
    if (args.CheckCmdLineFlag("help")) {
		Usage();
		return 0;
	}
    args.GetCmdLineArgument("i", g_iterations);
    args.GetCmdLineArgument("n", g_num_elements);
    args.GetCmdLineArgument("src", g_src_gpu);
    args.GetCmdLineArgument("dest", g_dest_gpu);
    args.GetCmdLineArgument("max-ctas", g_max_ctas);
    g_from_host = args.CheckCmdLineFlag("from-host");
    g_sample = args.CheckCmdLineFlag("sample");
	g_verbose = args.CheckCmdLineFlag("v");

	if ((g_src_gpu > -1) && (g_dest_gpu > -1)) {

		printf("Inter-GPU copy.\n");

		// Set device
		if (util::B40CPerror(hipSetDevice(g_src_gpu),
			"MultiGpuBfsEnactor hipSetDevice failed", __FILE__, __LINE__)) exit(1);
		printf("Enabling peer access to GPU %d from GPU %d\n", g_src_gpu, g_dest_gpu);
		if (util::B40CPerror(hipDeviceEnablePeerAccess(g_dest_gpu, 0),
			"MultiGpuBfsEnactor hipDeviceEnablePeerAccess failed", __FILE__, __LINE__)) exit(1);

		// Set device
		if (util::B40CPerror(hipSetDevice(g_dest_gpu),
			"MultiGpuBfsEnactor hipSetDevice failed", __FILE__, __LINE__)) exit(1);
		printf("Enabling peer access to GPU %d from GPU %d\n", g_dest_gpu, g_src_gpu);
		if (util::B40CPerror(hipDeviceEnablePeerAccess(g_src_gpu, 0),
			"MultiGpuBfsEnactor hipDeviceEnablePeerAccess failed", __FILE__, __LINE__)) exit(1);

	} else {

		if (g_from_host) {
			printf("From pinned host memory.\n");
		}

		// Put current device as both src and dest
		if (util::B40CPerror(hipGetDevice(&g_src_gpu),
			"MultiGpuBfsEnactor hipGetDevice failed", __FILE__, __LINE__)) exit(1);
		if (util::B40CPerror(hipGetDevice(&g_dest_gpu),
			"MultiGpuBfsEnactor hipGetDevice failed", __FILE__, __LINE__)) exit(1);
	}
   	TestCopy(g_num_elements);

	return 0;
}



