/******************************************************************************
 * 
 * Copyright 2010-2012 Duane Merrill
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a scan of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 ******************************************************************************/


/******************************************************************************
 * Simple program for evaluating grid size
 ******************************************************************************/

#include <stdio.h> 
#include <algorithm>

// Sorting includes
#include <b40c/util/io/modified_load.cuh>
#include <b40c/util/io/modified_store.cuh>

#include <b40c/copy/policy.cuh>
#include <b40c/copy/enactor.cuh>

// Test utils
#include "b40c_test_util.h"


/******************************************************************************
 * Problem / Tuning Policy Types
 ******************************************************************************/

typedef int T;
typedef int SizeT;

typedef b40c::copy::Policy <
	T,
	SizeT,
	200, 					// CUDA_ARCH,
	10,						// LOG_SCHEDULE_GRANULARITY
	1,						// MIN_CTA_OCCUPANCY
	7, 						// LOG_THREADS
	2, 						// LOG_LOAD_VEC_SIZE
	0, 						// LOG_LOADS_PER_TILE
	b40c::util::io::ld::NONE,	// LOAD_MODIFIER
	b40c::util::io::st::NONE, 	// STORE_MODIFIER
	false,					// WORKSTEALING
	false> 					// OVERSUBSCRIBED
		Policy;


/******************************************************************************
 * Main
 ******************************************************************************/

int main(int argc, char** argv)
{
    // Initialize command line
    b40c::CommandLineArgs args(argc, argv);
    b40c::DeviceInit(args);

	// Usage/help
    if (args.CheckCmdLineFlag("help") || args.CheckCmdLineFlag("h")) {
    	printf("\ngrid_size [--device=<device index>] [--v] [--i=<samples>] [--n=<elements>]\n");
    	return 0;
    }

	// Parse commandline args
    SizeT num_elements = 1024 * 1024 * 64;			// 64 million items
    int samples = 10;								// 1 sample

    bool verbose = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("n", num_elements);
    args.GetCmdLineArgument("i", samples);

    // Allocate array of random grid sizes (1 - 65536)
    int *cta_sizes = new int[samples];
	for (int i = 0; i < samples; i++) {
		b40c::util::RandomBits(cta_sizes[i], 0, 16);
		if (cta_sizes[i] == 0) cta_sizes[i] = 1;
	}

	// Allocate and initialize host problem data
	T *h_data = new T[num_elements];
	for (SizeT i = 0; i < num_elements; ++i) {
		h_data[i] = i;
	}

	// Allocate device data.
	T *d_in;
	T *d_out;
	hipMalloc((void**) &d_in, sizeof(T) * num_elements);
	hipMalloc((void**) &d_out, sizeof(T) * num_elements);

	hipMemcpy(d_in, h_data, sizeof(T) * num_elements, hipMemcpyHostToDevice);

	//
	// Perform passes
	//

	// Create an enactor
	b40c::copy::Enactor enactor;
	enactor.ENACTOR_DEBUG = verbose;

	// Perform the timed number of iterations
	b40c::GpuTimer timer;

	printf("Sample, Items, CTAs, Elapsed, Throughput\n");
	for (int i = 0; i < samples; i++) {

		timer.Start();
		enactor.Copy<Policy>(d_out, d_in, num_elements, cta_sizes[i]);
		timer.Stop();

		float throughput = float(num_elements) / timer.ElapsedMillis() / 1000.0 / 1000.0;

		printf("%d, %d, %d, %f, %f\n",
			i,
			num_elements,
			cta_sizes[i],
			timer.ElapsedMillis(),
			throughput);
	}

	// Cleanup
	hipFree(d_in);
	hipFree(d_out);
	delete h_data;
	delete cta_sizes;

	return 0;
}

